#include "hip/hip_runtime.h"
#include "star2d3r-512-6-512_kernel.hu"
__device__ float __sbref_wrap(float *sb, size_t index) { return sb[index]; }

__global__ void kernel0_6(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c1Pad = (3);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c2Pad = (3);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 3;
    const AN5D_TYPE __halo2 = 3;
    const AN5D_TYPE __side0Len = 6;
    const AN5D_TYPE __side1Len = 512;
    const AN5D_TYPE __side2Len = 476;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0_0;
    float __reg_0_1;
    float __reg_0_2;
    float __reg_0_3;
    float __reg_0_4;
    float __reg_0_5;
    float __reg_0_6;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_1_5;
    float __reg_1_6;
    float __reg_2_0;
    float __reg_2_1;
    float __reg_2_2;
    float __reg_2_3;
    float __reg_2_4;
    float __reg_2_5;
    float __reg_2_6;
    float __reg_3_0;
    float __reg_3_1;
    float __reg_3_2;
    float __reg_3_3;
    float __reg_3_4;
    float __reg_3_5;
    float __reg_3_6;
    float __reg_4_0;
    float __reg_4_1;
    float __reg_4_2;
    float __reg_4_3;
    float __reg_4_4;
    float __reg_4_5;
    float __reg_4_6;
    float __reg_5_0;
    float __reg_5_1;
    float __reg_5_2;
    float __reg_5_3;
    float __reg_5_4;
    float __reg_5_5;
    float __reg_5_6;
    __shared__ float __d_sb_double[__blockSize * 2];
    float *__d_sb = __d_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __writeValid3 = __updateValid && __local_c2 >= (__halo2 * 3) && __local_c2 < __side2LenOl - (__halo2 * 3);
    const AN5D_TYPE __writeValid4 = __updateValid && __local_c2 >= (__halo2 * 4) && __local_c2 < __side2LenOl - (__halo2 * 4);
    const AN5D_TYPE __writeValid5 = __updateValid && __local_c2 >= (__halo2 * 5) && __local_c2 < __side2LenOl - (__halo2 * 5);
    const AN5D_TYPE __writeValid6 = __updateValid && __local_c2 >= (__halo2 * 6) && __local_c2 < __side2LenOl - (__halo2 * 6);
    const AN5D_TYPE __storeValid = __writeValid6;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR(__rn0, __a, __b, __c, __d, __e, __f, __g) do { __rn0 = (((((((((((((0.06251f * (__REGREF(__a, 0))) + (0.06255f * (__REGREF(__b, 0)))) + (0.06245f * (__REGREF(__c, 0)))) + (0.06252f * (__SBREF(__d_sb, -3)))) + (0.06249f * (__SBREF(__d_sb, -2)))) + (0.06244f * (__SBREF(__d_sb, -1)))) + (0.25002f * (__REGREF(__d, 0)))) + (0.06248f * (__SBREF(__d_sb, 1)))) + (0.06243f * (__SBREF(__d_sb, 2)))) + (0.06253f * (__SBREF(__d_sb, 3)))) + (0.06246f * (__REGREF(__e, 0)))) + (0.06242f * (__REGREF(__f, 0)))) + (0.06254f * (__REGREF(__g, 0)))); } while (0)
    #define __DB_SWITCH() do { __d_sb = &__d_sb_double[(__d_sb == __d_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a, b, c, d, e, f, g) do { __DB_SWITCH(); __d_sb[__tid] = d; __syncthreads(); } while (0)
    #define __CALC1(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid1) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __CALC2(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid2) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __CALC3(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid3) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __CALC4(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid4) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __CALC5(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid5) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __STORE(h, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __CALCEXPR(__DEST, reg0, reg1, reg2, reg3, reg4, reg5, reg6); } } while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_5_0, 0);
      __LOAD(__reg_5_1, 1);
      __LOAD(__reg_5_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __CALC1(__reg_1_3, __reg_5_0, __reg_5_1, __reg_5_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __LOAD(__reg_0_0, 7);
      __CALC1(__reg_1_4, __reg_5_1, __reg_5_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __LOAD(__reg_0_1, 8);
      __CALC1(__reg_1_5, __reg_5_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __LOAD(__reg_0_2, 9);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_5_0, __reg_5_1, __reg_5_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __LOAD(__reg_0_3, 10);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_5_1, __reg_5_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __LOAD(__reg_0_4, 11);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_5_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __LOAD(__reg_0_5, 12);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_5_0, __reg_5_1, __reg_5_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __LOAD(__reg_0_6, 13);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_5_1, __reg_5_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __LOAD(__reg_0_0, 14);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_5_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __LOAD(__reg_0_1, 15);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __CALC4(__reg_4_3, __reg_5_0, __reg_5_1, __reg_5_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
      __LOAD(__reg_0_2, 16);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __CALC4(__reg_4_4, __reg_5_1, __reg_5_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
      __LOAD(__reg_0_3, 17);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __CALC4(__reg_4_5, __reg_5_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
      __LOAD(__reg_0_4, 18);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
      __CALC5(__reg_5_3, __reg_5_0, __reg_5_1, __reg_5_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
      __LOAD(__reg_0_5, 19);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
      __CALC5(__reg_5_4, __reg_5_1, __reg_5_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
      __LOAD(__reg_0_6, 20);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
      __CALC5(__reg_5_5, __reg_5_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
      __LOAD(__reg_0_0, 21);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
      __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
      __STORE(3, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
      __LOAD(__reg_0_1, 22);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
      __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
      __STORE(4, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
      __LOAD(__reg_0_2, 23);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
      __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
      __STORE(5, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
      __LOAD(__reg_0_3, 24);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
      __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
      __STORE(6, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
      __LOAD(__reg_0_4, 25);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
      __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
      __STORE(7, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
      __LOAD(__reg_0_5, 26);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
      __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
      __STORE(8, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
      __LOAD(__reg_0_6, 27);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
      __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
      __STORE(9, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
      __LOAD(__reg_0_0, 28);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
      __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
      __STORE(10, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
      __LOAD(__reg_0_1, 29);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
      __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
      __STORE(11, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
      __LOAD(__reg_0_2, 30);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
      __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
      __STORE(12, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
      __LOAD(__reg_0_3, 31);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
      __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
      __STORE(13, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
      __LOAD(__reg_0_4, 32);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
      __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
      __STORE(14, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
      __LOAD(__reg_0_5, 33);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
      __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
      __STORE(15, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
      __LOAD(__reg_0_6, 34);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
      __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
      __STORE(16, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
      __LOAD(__reg_0_0, 35);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
      __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
      __STORE(17, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
      __LOAD(__reg_0_1, 36);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
      __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
      __STORE(18, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
    }
    else
    {
      __LOAD(__reg_0_0, 0);
      __LOAD(__reg_0_1, 1);
      __LOAD(__reg_0_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __LOAD(__reg_0_0, 7);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __LOAD(__reg_0_1, 8);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __LOAD(__reg_0_2, 9);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __LOAD(__reg_0_3, 10);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __LOAD(__reg_0_4, 11);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __LOAD(__reg_0_5, 12);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __LOAD(__reg_0_6, 13);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __LOAD(__reg_0_0, 14);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __LOAD(__reg_0_1, 15);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __LOAD(__reg_0_2, 16);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __LOAD(__reg_0_3, 17);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __LOAD(__reg_0_4, 18);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __LOAD(__reg_0_5, 19);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __LOAD(__reg_0_6, 20);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __LOAD(__reg_0_0, 21);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __LOAD(__reg_0_1, 22);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __LOAD(__reg_0_2, 23);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __LOAD(__reg_0_3, 24);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
      __LOAD(__reg_0_4, 25);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
      __LOAD(__reg_0_5, 26);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
      __LOAD(__reg_0_6, 27);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
      __LOAD(__reg_0_0, 28);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
      __LOAD(__reg_0_1, 29);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
      __LOAD(__reg_0_2, 30);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
      __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
      __LOAD(__reg_0_3, 31);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
      __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
      __LOAD(__reg_0_4, 32);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
      __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
      __LOAD(__reg_0_5, 33);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
      __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
      __LOAD(__reg_0_6, 34);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
      __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
      __LOAD(__reg_0_0, 35);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
      __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
      __LOAD(__reg_0_1, 36);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
      __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
      __STORE(18, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
      __DB_SWITCH(); __syncthreads();
    }
    __d_sb = __d_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 37; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 7;)
      {
        __LOAD(__reg_0_2, __h);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 18, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __h++;
        __LOAD(__reg_0_3, __h);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 18, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __h++;
        __LOAD(__reg_0_4, __h);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 18, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __h++;
        __LOAD(__reg_0_5, __h);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __STORE(__h - 18, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __h++;
        __LOAD(__reg_0_6, __h);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __STORE(__h - 18, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __h++;
        __LOAD(__reg_0_0, __h);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __STORE(__h - 18, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
        __h++;
        __LOAD(__reg_0_1, __h);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __STORE(__h - 18, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
        __h++;
      }
      if (0) {}
      else if (__h + 0 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 18, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6, __reg_0_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 17, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 16, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_0_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __STORE(__h - 15, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_0_6, __reg_0_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __STORE(__h - 14, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __STORE(__h - 13, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_0_6);
        __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __STORE(__h - 12, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_0_6, __reg_0_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 11, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 10, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_0_6);
        __STORE(__h - 9, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 8, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 7, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __STORE(__h - 6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_0_6);
        __STORE(__h - 5, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 4, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_0_6, __reg_0_0, __reg_0_1);
      }
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_2, __h + 0);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 18, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 17, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0, __reg_0_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 16, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __STORE(__h - 15, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_0_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __STORE(__h - 14, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_0_0, __reg_0_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __STORE(__h - 13, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __STORE(__h - 12, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_0_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 11, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_0_0, __reg_0_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 10, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 9, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_0_0);
        __STORE(__h - 8, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 7, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
        __STORE(__h - 5, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_0_0);
        __STORE(__h - 4, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 3, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_0_0, __reg_0_1, __reg_0_2);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_2, __h + 0);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 18, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __LOAD(__reg_0_3, __h + 1);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 17, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 16, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1, __reg_0_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __STORE(__h - 15, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __STORE(__h - 14, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_0_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __STORE(__h - 13, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_0_1, __reg_0_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __STORE(__h - 12, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 11, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_0_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 10, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_0_1, __reg_0_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 9, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __STORE(__h - 8, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_0_1);
        __STORE(__h - 7, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
        __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 5, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
        __STORE(__h - 4, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_0_1);
        __STORE(__h - 3, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 2, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_0_1, __reg_0_2, __reg_0_3);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_2, __h + 0);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 18, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __LOAD(__reg_0_3, __h + 1);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 17, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __LOAD(__reg_0_4, __h + 2);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 16, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __STORE(__h - 15, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2, __reg_0_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __STORE(__h - 14, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __STORE(__h - 13, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_0_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __STORE(__h - 12, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_0_2, __reg_0_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 11, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 10, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_0_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 9, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_0_2, __reg_0_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __STORE(__h - 8, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __STORE(__h - 7, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_0_2);
        __STORE(__h - 6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
        __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 5, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 4, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __STORE(__h - 3, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_0_2);
        __STORE(__h - 2, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 1, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_0_2, __reg_0_3, __reg_0_4);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_2, __h + 0);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 18, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __LOAD(__reg_0_3, __h + 1);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 17, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __LOAD(__reg_0_4, __h + 2);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 16, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __LOAD(__reg_0_5, __h + 3);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __STORE(__h - 15, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __STORE(__h - 14, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __STORE(__h - 13, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __STORE(__h - 12, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_0_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 11, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_0_3, __reg_0_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 10, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 9, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_0_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __STORE(__h - 8, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_0_3, __reg_0_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __STORE(__h - 7, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __STORE(__h - 6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
        __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_0_3);
        __STORE(__h - 5, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 4, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 3, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __STORE(__h - 2, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_0_3);
        __STORE(__h - 1, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_0_3, __reg_0_4);
        __STORE(__h + 0, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_0_3, __reg_0_4, __reg_0_5);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_2, __h + 0);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 18, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __LOAD(__reg_0_3, __h + 1);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 17, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __LOAD(__reg_0_4, __h + 2);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 16, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __LOAD(__reg_0_5, __h + 3);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __STORE(__h - 15, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __LOAD(__reg_0_6, __h + 4);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __STORE(__h - 14, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __STORE(__h - 13, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4, __reg_0_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __STORE(__h - 12, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 11, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_0_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 10, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_0_4, __reg_0_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 9, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __STORE(__h - 8, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_0_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __STORE(__h - 7, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_0_4, __reg_0_5);
        __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __STORE(__h - 6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __STORE(__h - 5, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_0_4);
        __STORE(__h - 4, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 3, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 2, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __STORE(__h - 1, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_0_4);
        __STORE(__h + 0, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_0_4, __reg_0_5);
        __STORE(__h + 1, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_0_4, __reg_0_5, __reg_0_6);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_2, __h + 0);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 18, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __LOAD(__reg_0_3, __h + 1);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 17, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __LOAD(__reg_0_4, __h + 2);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 16, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __LOAD(__reg_0_5, __h + 3);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __STORE(__h - 15, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __LOAD(__reg_0_6, __h + 4);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __STORE(__h - 14, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __LOAD(__reg_0_0, __h + 5);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __STORE(__h - 13, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __STORE(__h - 12, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5, __reg_0_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 11, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 10, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_0_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 9, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_0_5, __reg_0_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __STORE(__h - 8, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __STORE(__h - 7, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_0_5);
        __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __STORE(__h - 6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_0_5, __reg_0_6);
        __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __STORE(__h - 5, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 4, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_0_5);
        __STORE(__h - 3, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 2, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 1, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __STORE(__h + 0, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_0_5);
        __STORE(__h + 1, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_0_5, __reg_0_6);
        __STORE(__h + 2, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_0_5, __reg_0_6, __reg_0_0);
      }
    }
    else
    {
      for (__h = 37; __h <= __side1LenOl - 7;)
      {
        __LOAD(__reg_0_2, __h);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 18, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
        __h++;
        __LOAD(__reg_0_3, __h);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h - 18, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
        __h++;
        __LOAD(__reg_0_4, __h);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 18, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
        __h++;
        __LOAD(__reg_0_5, __h);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __STORE(__h - 18, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
        __h++;
        __LOAD(__reg_0_6, __h);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __STORE(__h - 18, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
        __h++;
        __LOAD(__reg_0_0, __h);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __STORE(__h - 18, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
        __h++;
        __LOAD(__reg_0_1, __h);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __STORE(__h - 18, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
        __h++;
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_2, __h);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
      __CALC5(__reg_5_1, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
      __STORE(__h - 18, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_3, __h);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
      __CALC5(__reg_5_2, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
      __STORE(__h - 18, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_4, __h);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
      __CALC5(__reg_5_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
      __STORE(__h - 18, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_5, __h);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
      __CALC5(__reg_5_4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
      __STORE(__h - 18, __reg_5_5, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_6, __h);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
      __CALC5(__reg_5_5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
      __STORE(__h - 18, __reg_5_6, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_0, __h);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
      __CALC5(__reg_5_6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
      __STORE(__h - 18, __reg_5_0, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_1, __h);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
      __CALC5(__reg_5_0, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
      __STORE(__h - 18, __reg_5_1, __reg_5_2, __reg_5_3, __reg_5_4, __reg_5_5, __reg_5_6, __reg_5_0);
      __h++;
    }
}
__global__ void kernel0_5(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c1Pad = (3);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c2Pad = (3);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 3;
    const AN5D_TYPE __halo2 = 3;
    const AN5D_TYPE __side0Len = 5;
    const AN5D_TYPE __side1Len = 512;
    const AN5D_TYPE __side2Len = 482;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0_0;
    float __reg_0_1;
    float __reg_0_2;
    float __reg_0_3;
    float __reg_0_4;
    float __reg_0_5;
    float __reg_0_6;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_1_5;
    float __reg_1_6;
    float __reg_2_0;
    float __reg_2_1;
    float __reg_2_2;
    float __reg_2_3;
    float __reg_2_4;
    float __reg_2_5;
    float __reg_2_6;
    float __reg_3_0;
    float __reg_3_1;
    float __reg_3_2;
    float __reg_3_3;
    float __reg_3_4;
    float __reg_3_5;
    float __reg_3_6;
    float __reg_4_0;
    float __reg_4_1;
    float __reg_4_2;
    float __reg_4_3;
    float __reg_4_4;
    float __reg_4_5;
    float __reg_4_6;
    __shared__ float __d_sb_double[__blockSize * 2];
    float *__d_sb = __d_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __writeValid3 = __updateValid && __local_c2 >= (__halo2 * 3) && __local_c2 < __side2LenOl - (__halo2 * 3);
    const AN5D_TYPE __writeValid4 = __updateValid && __local_c2 >= (__halo2 * 4) && __local_c2 < __side2LenOl - (__halo2 * 4);
    const AN5D_TYPE __writeValid5 = __updateValid && __local_c2 >= (__halo2 * 5) && __local_c2 < __side2LenOl - (__halo2 * 5);
    const AN5D_TYPE __storeValid = __writeValid5;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR(__rn0, __a, __b, __c, __d, __e, __f, __g) do { __rn0 = (((((((((((((0.06251f * (__REGREF(__a, 0))) + (0.06255f * (__REGREF(__b, 0)))) + (0.06245f * (__REGREF(__c, 0)))) + (0.06252f * (__SBREF(__d_sb, -3)))) + (0.06249f * (__SBREF(__d_sb, -2)))) + (0.06244f * (__SBREF(__d_sb, -1)))) + (0.25002f * (__REGREF(__d, 0)))) + (0.06248f * (__SBREF(__d_sb, 1)))) + (0.06243f * (__SBREF(__d_sb, 2)))) + (0.06253f * (__SBREF(__d_sb, 3)))) + (0.06246f * (__REGREF(__e, 0)))) + (0.06242f * (__REGREF(__f, 0)))) + (0.06254f * (__REGREF(__g, 0)))); } while (0)
    #define __DB_SWITCH() do { __d_sb = &__d_sb_double[(__d_sb == __d_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a, b, c, d, e, f, g) do { __DB_SWITCH(); __d_sb[__tid] = d; __syncthreads(); } while (0)
    #define __CALC1(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid1) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __CALC2(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid2) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __CALC3(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid3) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __CALC4(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid4) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __STORE(h, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __CALCEXPR(__DEST, reg0, reg1, reg2, reg3, reg4, reg5, reg6); } } while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_4_0, 0);
      __LOAD(__reg_4_1, 1);
      __LOAD(__reg_4_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __CALC1(__reg_1_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __LOAD(__reg_0_0, 7);
      __CALC1(__reg_1_4, __reg_4_1, __reg_4_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __LOAD(__reg_0_1, 8);
      __CALC1(__reg_1_5, __reg_4_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __LOAD(__reg_0_2, 9);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __LOAD(__reg_0_3, 10);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_4_1, __reg_4_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __LOAD(__reg_0_4, 11);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_4_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __LOAD(__reg_0_5, 12);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __LOAD(__reg_0_6, 13);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_4_1, __reg_4_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __LOAD(__reg_0_0, 14);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_4_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __LOAD(__reg_0_1, 15);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __CALC4(__reg_4_3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
      __LOAD(__reg_0_2, 16);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __CALC4(__reg_4_4, __reg_4_1, __reg_4_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
      __LOAD(__reg_0_3, 17);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __CALC4(__reg_4_5, __reg_4_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
      __LOAD(__reg_0_4, 18);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
      __STORE(3, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
      __LOAD(__reg_0_5, 19);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
      __STORE(4, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
      __LOAD(__reg_0_6, 20);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
      __STORE(5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
      __LOAD(__reg_0_0, 21);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
      __STORE(6, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
      __LOAD(__reg_0_1, 22);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
      __STORE(7, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
      __LOAD(__reg_0_2, 23);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
      __STORE(8, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
      __LOAD(__reg_0_3, 24);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
      __STORE(9, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
      __LOAD(__reg_0_4, 25);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
      __STORE(10, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
      __LOAD(__reg_0_5, 26);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
      __STORE(11, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
      __LOAD(__reg_0_6, 27);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
      __STORE(12, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
      __LOAD(__reg_0_0, 28);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
      __STORE(13, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
      __LOAD(__reg_0_1, 29);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
      __STORE(14, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
      __LOAD(__reg_0_2, 30);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
      __STORE(15, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
    }
    else
    {
      __LOAD(__reg_0_0, 0);
      __LOAD(__reg_0_1, 1);
      __LOAD(__reg_0_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __LOAD(__reg_0_0, 7);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __LOAD(__reg_0_1, 8);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __LOAD(__reg_0_2, 9);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __LOAD(__reg_0_3, 10);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __LOAD(__reg_0_4, 11);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __LOAD(__reg_0_5, 12);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __LOAD(__reg_0_6, 13);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __LOAD(__reg_0_0, 14);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __LOAD(__reg_0_1, 15);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __LOAD(__reg_0_2, 16);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __LOAD(__reg_0_3, 17);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __LOAD(__reg_0_4, 18);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __LOAD(__reg_0_5, 19);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __LOAD(__reg_0_6, 20);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __LOAD(__reg_0_0, 21);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __LOAD(__reg_0_1, 22);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __LOAD(__reg_0_2, 23);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __LOAD(__reg_0_3, 24);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
      __LOAD(__reg_0_4, 25);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
      __LOAD(__reg_0_5, 26);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
      __LOAD(__reg_0_6, 27);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
      __LOAD(__reg_0_0, 28);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
      __LOAD(__reg_0_1, 29);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
      __LOAD(__reg_0_2, 30);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
      __STORE(15, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
    }
    __d_sb = __d_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 31; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 7;)
      {
        __LOAD(__reg_0_3, __h);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __STORE(__h - 15, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __h++;
        __LOAD(__reg_0_4, __h);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __STORE(__h - 15, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __h++;
        __LOAD(__reg_0_5, __h);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __STORE(__h - 15, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __h++;
        __LOAD(__reg_0_6, __h);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __STORE(__h - 15, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __h++;
        __LOAD(__reg_0_0, __h);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __STORE(__h - 15, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __h++;
        __LOAD(__reg_0_1, __h);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __STORE(__h - 15, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __h++;
        __LOAD(__reg_0_2, __h);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __STORE(__h - 15, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 0 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __STORE(__h - 15, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0, __reg_0_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __STORE(__h - 14, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __STORE(__h - 13, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_0_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __STORE(__h - 12, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_0_0, __reg_0_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __STORE(__h - 11, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __STORE(__h - 10, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_0_0);
        __STORE(__h - 9, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 8, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 7, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __STORE(__h - 6, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_0_0);
        __STORE(__h - 5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 4, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_0_0, __reg_0_1, __reg_0_2);
      }
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_3, __h + 0);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __STORE(__h - 15, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __STORE(__h - 14, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1, __reg_0_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __STORE(__h - 13, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __STORE(__h - 12, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_0_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __STORE(__h - 11, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_0_1, __reg_0_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __STORE(__h - 10, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __STORE(__h - 9, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_0_1);
        __STORE(__h - 8, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 7, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 6, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __STORE(__h - 5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_0_1);
        __STORE(__h - 4, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_0_1, __reg_0_2, __reg_0_3);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_3, __h + 0);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __STORE(__h - 15, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __LOAD(__reg_0_4, __h + 1);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __STORE(__h - 14, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __STORE(__h - 13, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2, __reg_0_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __STORE(__h - 12, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __STORE(__h - 11, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_0_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __STORE(__h - 10, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_0_2, __reg_0_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __STORE(__h - 9, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __STORE(__h - 8, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_0_2);
        __STORE(__h - 7, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 6, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __STORE(__h - 4, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_0_2);
        __STORE(__h - 3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 2, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_0_2, __reg_0_3, __reg_0_4);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_3, __h + 0);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __STORE(__h - 15, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __LOAD(__reg_0_4, __h + 1);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __STORE(__h - 14, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __LOAD(__reg_0_5, __h + 2);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __STORE(__h - 13, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __STORE(__h - 12, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __STORE(__h - 11, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __STORE(__h - 10, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_0_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __STORE(__h - 9, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_0_3, __reg_0_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __STORE(__h - 8, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __STORE(__h - 7, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_0_3);
        __STORE(__h - 6, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 4, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __STORE(__h - 3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_0_3);
        __STORE(__h - 2, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 1, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_0_3, __reg_0_4, __reg_0_5);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_3, __h + 0);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __STORE(__h - 15, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __LOAD(__reg_0_4, __h + 1);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __STORE(__h - 14, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __LOAD(__reg_0_5, __h + 2);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __STORE(__h - 13, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __LOAD(__reg_0_6, __h + 3);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __STORE(__h - 12, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __STORE(__h - 11, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4, __reg_0_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __STORE(__h - 10, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __STORE(__h - 9, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_0_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __STORE(__h - 8, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_0_4, __reg_0_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __STORE(__h - 7, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __STORE(__h - 6, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_0_4);
        __STORE(__h - 5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 4, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __STORE(__h - 2, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_0_4);
        __STORE(__h - 1, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_0_4, __reg_0_5);
        __STORE(__h + 0, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_0_4, __reg_0_5, __reg_0_6);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_3, __h + 0);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __STORE(__h - 15, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __LOAD(__reg_0_4, __h + 1);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __STORE(__h - 14, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __LOAD(__reg_0_5, __h + 2);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __STORE(__h - 13, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __LOAD(__reg_0_6, __h + 3);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __STORE(__h - 12, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __LOAD(__reg_0_0, __h + 4);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __STORE(__h - 11, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __STORE(__h - 10, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5, __reg_0_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __STORE(__h - 9, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __STORE(__h - 8, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_0_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __STORE(__h - 7, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_0_5, __reg_0_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __STORE(__h - 6, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __STORE(__h - 5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_0_5);
        __STORE(__h - 4, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 2, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __STORE(__h - 1, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_0_5);
        __STORE(__h + 0, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_0_5, __reg_0_6);
        __STORE(__h + 1, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_0_5, __reg_0_6, __reg_0_0);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_3, __h + 0);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __STORE(__h - 15, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __LOAD(__reg_0_4, __h + 1);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __STORE(__h - 14, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __LOAD(__reg_0_5, __h + 2);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __STORE(__h - 13, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __LOAD(__reg_0_6, __h + 3);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __STORE(__h - 12, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __LOAD(__reg_0_0, __h + 4);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __STORE(__h - 11, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __LOAD(__reg_0_1, __h + 5);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __STORE(__h - 10, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __STORE(__h - 9, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6, __reg_0_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __STORE(__h - 8, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __STORE(__h - 7, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_0_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __STORE(__h - 6, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_0_6, __reg_0_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __STORE(__h - 5, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __STORE(__h - 4, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_0_6);
        __STORE(__h - 3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 2, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 1, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __STORE(__h + 0, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_0_6);
        __STORE(__h + 1, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_0_6, __reg_0_0);
        __STORE(__h + 2, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_0_6, __reg_0_0, __reg_0_1);
      }
    }
    else
    {
      for (__h = 31; __h <= __side1LenOl - 7;)
      {
        __LOAD(__reg_0_3, __h);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __STORE(__h - 15, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
        __h++;
        __LOAD(__reg_0_4, __h);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __STORE(__h - 15, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
        __h++;
        __LOAD(__reg_0_5, __h);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __STORE(__h - 15, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
        __h++;
        __LOAD(__reg_0_6, __h);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __STORE(__h - 15, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
        __h++;
        __LOAD(__reg_0_0, __h);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __STORE(__h - 15, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
        __h++;
        __LOAD(__reg_0_1, __h);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __STORE(__h - 15, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
        __h++;
        __LOAD(__reg_0_2, __h);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __STORE(__h - 15, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
        __h++;
        __DB_SWITCH();  __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_3, __h);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __CALC4(__reg_4_5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
      __STORE(__h - 15, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_4, __h);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __CALC4(__reg_4_6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
      __STORE(__h - 15, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_5, __h);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __CALC4(__reg_4_0, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
      __STORE(__h - 15, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_6, __h);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __CALC4(__reg_4_1, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
      __STORE(__h - 15, __reg_4_2, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_0, __h);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __CALC4(__reg_4_2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
      __STORE(__h - 15, __reg_4_3, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_1, __h);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __CALC4(__reg_4_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
      __STORE(__h - 15, __reg_4_4, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_2, __h);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __CALC4(__reg_4_4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
      __STORE(__h - 15, __reg_4_5, __reg_4_6, __reg_4_0, __reg_4_1, __reg_4_2, __reg_4_3, __reg_4_4);
      __h++;
    }
}
__global__ void kernel0_4(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c1Pad = (3);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c2Pad = (3);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 3;
    const AN5D_TYPE __halo2 = 3;
    const AN5D_TYPE __side0Len = 4;
    const AN5D_TYPE __side1Len = 512;
    const AN5D_TYPE __side2Len = 488;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0_0;
    float __reg_0_1;
    float __reg_0_2;
    float __reg_0_3;
    float __reg_0_4;
    float __reg_0_5;
    float __reg_0_6;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_1_5;
    float __reg_1_6;
    float __reg_2_0;
    float __reg_2_1;
    float __reg_2_2;
    float __reg_2_3;
    float __reg_2_4;
    float __reg_2_5;
    float __reg_2_6;
    float __reg_3_0;
    float __reg_3_1;
    float __reg_3_2;
    float __reg_3_3;
    float __reg_3_4;
    float __reg_3_5;
    float __reg_3_6;
    __shared__ float __d_sb_double[__blockSize * 2];
    float *__d_sb = __d_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __writeValid3 = __updateValid && __local_c2 >= (__halo2 * 3) && __local_c2 < __side2LenOl - (__halo2 * 3);
    const AN5D_TYPE __writeValid4 = __updateValid && __local_c2 >= (__halo2 * 4) && __local_c2 < __side2LenOl - (__halo2 * 4);
    const AN5D_TYPE __storeValid = __writeValid4;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR(__rn0, __a, __b, __c, __d, __e, __f, __g) do { __rn0 = (((((((((((((0.06251f * (__REGREF(__a, 0))) + (0.06255f * (__REGREF(__b, 0)))) + (0.06245f * (__REGREF(__c, 0)))) + (0.06252f * (__SBREF(__d_sb, -3)))) + (0.06249f * (__SBREF(__d_sb, -2)))) + (0.06244f * (__SBREF(__d_sb, -1)))) + (0.25002f * (__REGREF(__d, 0)))) + (0.06248f * (__SBREF(__d_sb, 1)))) + (0.06243f * (__SBREF(__d_sb, 2)))) + (0.06253f * (__SBREF(__d_sb, 3)))) + (0.06246f * (__REGREF(__e, 0)))) + (0.06242f * (__REGREF(__f, 0)))) + (0.06254f * (__REGREF(__g, 0)))); } while (0)
    #define __DB_SWITCH() do { __d_sb = &__d_sb_double[(__d_sb == __d_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a, b, c, d, e, f, g) do { __DB_SWITCH(); __d_sb[__tid] = d; __syncthreads(); } while (0)
    #define __CALC1(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid1) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __CALC2(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid2) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __CALC3(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid3) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __STORE(h, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __CALCEXPR(__DEST, reg0, reg1, reg2, reg3, reg4, reg5, reg6); } } while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_3_0, 0);
      __LOAD(__reg_3_1, 1);
      __LOAD(__reg_3_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __CALC1(__reg_1_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __LOAD(__reg_0_0, 7);
      __CALC1(__reg_1_4, __reg_3_1, __reg_3_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __LOAD(__reg_0_1, 8);
      __CALC1(__reg_1_5, __reg_3_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __LOAD(__reg_0_2, 9);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __LOAD(__reg_0_3, 10);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_3_1, __reg_3_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __LOAD(__reg_0_4, 11);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_3_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __LOAD(__reg_0_5, 12);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __LOAD(__reg_0_6, 13);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_3_1, __reg_3_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __LOAD(__reg_0_0, 14);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_3_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __LOAD(__reg_0_1, 15);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __STORE(3, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
      __LOAD(__reg_0_2, 16);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __STORE(4, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
      __LOAD(__reg_0_3, 17);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __STORE(5, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
      __LOAD(__reg_0_4, 18);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __STORE(6, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
      __LOAD(__reg_0_5, 19);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __STORE(7, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
      __LOAD(__reg_0_6, 20);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __STORE(8, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
      __LOAD(__reg_0_0, 21);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __STORE(9, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
      __LOAD(__reg_0_1, 22);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __STORE(10, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
      __LOAD(__reg_0_2, 23);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __STORE(11, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
      __LOAD(__reg_0_3, 24);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __STORE(12, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
    }
    else
    {
      __LOAD(__reg_0_0, 0);
      __LOAD(__reg_0_1, 1);
      __LOAD(__reg_0_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __LOAD(__reg_0_0, 7);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __LOAD(__reg_0_1, 8);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __LOAD(__reg_0_2, 9);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __LOAD(__reg_0_3, 10);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __LOAD(__reg_0_4, 11);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __LOAD(__reg_0_5, 12);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __LOAD(__reg_0_6, 13);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __LOAD(__reg_0_0, 14);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __LOAD(__reg_0_1, 15);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __LOAD(__reg_0_2, 16);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __LOAD(__reg_0_3, 17);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __LOAD(__reg_0_4, 18);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __LOAD(__reg_0_5, 19);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __LOAD(__reg_0_6, 20);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __LOAD(__reg_0_0, 21);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __LOAD(__reg_0_1, 22);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __LOAD(__reg_0_2, 23);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __LOAD(__reg_0_3, 24);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __STORE(12, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
    }
    __d_sb = __d_sb_double + __blockSize * 0;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 25; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 7;)
      {
        __LOAD(__reg_0_4, __h);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __STORE(__h - 12, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __h++;
        __LOAD(__reg_0_5, __h);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __STORE(__h - 12, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __h++;
        __LOAD(__reg_0_6, __h);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __STORE(__h - 12, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __h++;
        __LOAD(__reg_0_0, __h);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __STORE(__h - 12, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __h++;
        __LOAD(__reg_0_1, __h);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __STORE(__h - 12, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __h++;
        __LOAD(__reg_0_2, __h);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __STORE(__h - 12, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __h++;
        __LOAD(__reg_0_3, __h);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __STORE(__h - 12, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __h++;
      }
      if (0) {}
      else if (__h + 0 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __STORE(__h - 12, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1, __reg_0_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __STORE(__h - 11, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __STORE(__h - 10, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_0_1);
        __STORE(__h - 9, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 8, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 7, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __STORE(__h - 6, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_0_1);
        __STORE(__h - 5, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 4, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_0_1, __reg_0_2, __reg_0_3);
      }
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_4, __h + 0);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __STORE(__h - 12, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __STORE(__h - 11, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2, __reg_0_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __STORE(__h - 10, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __STORE(__h - 9, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_0_2);
        __STORE(__h - 8, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 7, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 6, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __STORE(__h - 5, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_0_2);
        __STORE(__h - 4, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 3, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_0_2, __reg_0_3, __reg_0_4);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_4, __h + 0);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __STORE(__h - 12, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __LOAD(__reg_0_5, __h + 1);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __STORE(__h - 11, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __STORE(__h - 10, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __STORE(__h - 9, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __STORE(__h - 8, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_0_3);
        __STORE(__h - 7, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 6, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 5, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __STORE(__h - 4, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_0_3);
        __STORE(__h - 3, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_0_3, __reg_0_4, __reg_0_5);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_4, __h + 0);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __STORE(__h - 12, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __LOAD(__reg_0_5, __h + 1);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __STORE(__h - 11, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __LOAD(__reg_0_6, __h + 2);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __STORE(__h - 10, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __STORE(__h - 9, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4, __reg_0_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __STORE(__h - 8, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __STORE(__h - 7, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_0_4);
        __STORE(__h - 6, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 5, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 4, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __STORE(__h - 3, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_0_4);
        __STORE(__h - 2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 1, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_0_4, __reg_0_5, __reg_0_6);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_4, __h + 0);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __STORE(__h - 12, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __LOAD(__reg_0_5, __h + 1);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __STORE(__h - 11, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __LOAD(__reg_0_6, __h + 2);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __STORE(__h - 10, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __LOAD(__reg_0_0, __h + 3);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __STORE(__h - 9, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __STORE(__h - 8, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5, __reg_0_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __STORE(__h - 7, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __STORE(__h - 6, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_0_5);
        __STORE(__h - 5, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 4, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 3, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __STORE(__h - 2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_0_5);
        __STORE(__h - 1, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_0_5, __reg_0_6);
        __STORE(__h + 0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_0_5, __reg_0_6, __reg_0_0);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_4, __h + 0);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __STORE(__h - 12, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __LOAD(__reg_0_5, __h + 1);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __STORE(__h - 11, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __LOAD(__reg_0_6, __h + 2);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __STORE(__h - 10, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __LOAD(__reg_0_0, __h + 3);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __STORE(__h - 9, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __LOAD(__reg_0_1, __h + 4);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __STORE(__h - 8, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __STORE(__h - 7, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6, __reg_0_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __STORE(__h - 6, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __STORE(__h - 5, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_0_6);
        __STORE(__h - 4, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 3, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __STORE(__h - 1, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_0_6);
        __STORE(__h + 0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_0_6, __reg_0_0);
        __STORE(__h + 1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_0_6, __reg_0_0, __reg_0_1);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_4, __h + 0);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __STORE(__h - 12, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __LOAD(__reg_0_5, __h + 1);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __STORE(__h - 11, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __LOAD(__reg_0_6, __h + 2);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __STORE(__h - 10, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __LOAD(__reg_0_0, __h + 3);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __STORE(__h - 9, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __LOAD(__reg_0_1, __h + 4);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __STORE(__h - 8, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __LOAD(__reg_0_2, __h + 5);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __STORE(__h - 7, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __STORE(__h - 6, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0, __reg_0_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __STORE(__h - 5, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __STORE(__h - 4, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_0_0);
        __STORE(__h - 3, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 2, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 1, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __STORE(__h + 0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_0_0);
        __STORE(__h + 1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_0_0, __reg_0_1);
        __STORE(__h + 2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_0_0, __reg_0_1, __reg_0_2);
      }
    }
    else
    {
      for (__h = 25; __h <= __side1LenOl - 7;)
      {
        __LOAD(__reg_0_4, __h);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __STORE(__h - 12, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
        __h++;
        __LOAD(__reg_0_5, __h);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __STORE(__h - 12, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
        __h++;
        __LOAD(__reg_0_6, __h);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __STORE(__h - 12, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
        __h++;
        __LOAD(__reg_0_0, __h);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __STORE(__h - 12, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
        __h++;
        __LOAD(__reg_0_1, __h);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __STORE(__h - 12, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
        __h++;
        __LOAD(__reg_0_2, __h);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __STORE(__h - 12, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
        __h++;
        __LOAD(__reg_0_3, __h);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __STORE(__h - 12, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
        __h++;
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_4, __h);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __CALC3(__reg_3_2, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __STORE(__h - 12, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_5, __h);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __CALC3(__reg_3_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __STORE(__h - 12, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_6, __h);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __CALC3(__reg_3_4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __STORE(__h - 12, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_0, __h);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __CALC3(__reg_3_5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __STORE(__h - 12, __reg_3_6, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_1, __h);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __CALC3(__reg_3_6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __STORE(__h - 12, __reg_3_0, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_2, __h);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __CALC3(__reg_3_0, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __STORE(__h - 12, __reg_3_1, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_3, __h);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __CALC3(__reg_3_1, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __STORE(__h - 12, __reg_3_2, __reg_3_3, __reg_3_4, __reg_3_5, __reg_3_6, __reg_3_0, __reg_3_1);
      __h++;
    }
}
__global__ void kernel0_3(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c1Pad = (3);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c2Pad = (3);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 3;
    const AN5D_TYPE __halo2 = 3;
    const AN5D_TYPE __side0Len = 3;
    const AN5D_TYPE __side1Len = 512;
    const AN5D_TYPE __side2Len = 494;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0_0;
    float __reg_0_1;
    float __reg_0_2;
    float __reg_0_3;
    float __reg_0_4;
    float __reg_0_5;
    float __reg_0_6;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_1_5;
    float __reg_1_6;
    float __reg_2_0;
    float __reg_2_1;
    float __reg_2_2;
    float __reg_2_3;
    float __reg_2_4;
    float __reg_2_5;
    float __reg_2_6;
    __shared__ float __d_sb_double[__blockSize * 2];
    float *__d_sb = __d_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __writeValid3 = __updateValid && __local_c2 >= (__halo2 * 3) && __local_c2 < __side2LenOl - (__halo2 * 3);
    const AN5D_TYPE __storeValid = __writeValid3;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR(__rn0, __a, __b, __c, __d, __e, __f, __g) do { __rn0 = (((((((((((((0.06251f * (__REGREF(__a, 0))) + (0.06255f * (__REGREF(__b, 0)))) + (0.06245f * (__REGREF(__c, 0)))) + (0.06252f * (__SBREF(__d_sb, -3)))) + (0.06249f * (__SBREF(__d_sb, -2)))) + (0.06244f * (__SBREF(__d_sb, -1)))) + (0.25002f * (__REGREF(__d, 0)))) + (0.06248f * (__SBREF(__d_sb, 1)))) + (0.06243f * (__SBREF(__d_sb, 2)))) + (0.06253f * (__SBREF(__d_sb, 3)))) + (0.06246f * (__REGREF(__e, 0)))) + (0.06242f * (__REGREF(__f, 0)))) + (0.06254f * (__REGREF(__g, 0)))); } while (0)
    #define __DB_SWITCH() do { __d_sb = &__d_sb_double[(__d_sb == __d_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a, b, c, d, e, f, g) do { __DB_SWITCH(); __d_sb[__tid] = d; __syncthreads(); } while (0)
    #define __CALC1(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid1) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __CALC2(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid2) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __STORE(h, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __CALCEXPR(__DEST, reg0, reg1, reg2, reg3, reg4, reg5, reg6); } } while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_2_0, 0);
      __LOAD(__reg_2_1, 1);
      __LOAD(__reg_2_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __CALC1(__reg_1_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __LOAD(__reg_0_0, 7);
      __CALC1(__reg_1_4, __reg_2_1, __reg_2_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __LOAD(__reg_0_1, 8);
      __CALC1(__reg_1_5, __reg_2_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __LOAD(__reg_0_2, 9);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __LOAD(__reg_0_3, 10);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_2_1, __reg_2_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __LOAD(__reg_0_4, 11);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_2_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __LOAD(__reg_0_5, 12);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __STORE(3, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __LOAD(__reg_0_6, 13);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __STORE(4, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __LOAD(__reg_0_0, 14);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __STORE(5, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __LOAD(__reg_0_1, 15);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __STORE(6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __LOAD(__reg_0_2, 16);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __STORE(7, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __LOAD(__reg_0_3, 17);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __STORE(8, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __LOAD(__reg_0_4, 18);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __STORE(9, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
    }
    else
    {
      __LOAD(__reg_0_0, 0);
      __LOAD(__reg_0_1, 1);
      __LOAD(__reg_0_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __LOAD(__reg_0_0, 7);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __LOAD(__reg_0_1, 8);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __LOAD(__reg_0_2, 9);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __LOAD(__reg_0_3, 10);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __LOAD(__reg_0_4, 11);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __LOAD(__reg_0_5, 12);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __LOAD(__reg_0_6, 13);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __LOAD(__reg_0_0, 14);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __LOAD(__reg_0_1, 15);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __LOAD(__reg_0_2, 16);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __LOAD(__reg_0_3, 17);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __LOAD(__reg_0_4, 18);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __STORE(9, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __DB_SWITCH(); __syncthreads();
    }
    __d_sb = __d_sb_double + __blockSize * 0;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 19; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 7;)
      {
        __LOAD(__reg_0_5, __h);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __STORE(__h - 9, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __h++;
        __LOAD(__reg_0_6, __h);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __STORE(__h - 9, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __h++;
        __LOAD(__reg_0_0, __h);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __STORE(__h - 9, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __h++;
        __LOAD(__reg_0_1, __h);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __STORE(__h - 9, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __h++;
        __LOAD(__reg_0_2, __h);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __STORE(__h - 9, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __h++;
        __LOAD(__reg_0_3, __h);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __STORE(__h - 9, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __h++;
        __LOAD(__reg_0_4, __h);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __STORE(__h - 9, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 0 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2);
        __STORE(__h - 9, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 8, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 7, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __STORE(__h - 6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_0_2);
        __STORE(__h - 5, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_0_2, __reg_0_3, __reg_0_4);
      }
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_5, __h + 0);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __STORE(__h - 9, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3);
        __STORE(__h - 8, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 7, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __STORE(__h - 5, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_0_3);
        __STORE(__h - 4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 3, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_0_3, __reg_0_4, __reg_0_5);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_5, __h + 0);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __STORE(__h - 9, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __LOAD(__reg_0_6, __h + 1);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __STORE(__h - 8, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4);
        __STORE(__h - 7, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 5, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __STORE(__h - 4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_0_4);
        __STORE(__h - 3, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 2, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_0_4, __reg_0_5, __reg_0_6);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_5, __h + 0);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __STORE(__h - 9, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __LOAD(__reg_0_6, __h + 1);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __STORE(__h - 8, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __LOAD(__reg_0_0, __h + 2);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __STORE(__h - 7, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5);
        __STORE(__h - 6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 5, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __STORE(__h - 3, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_0_5);
        __STORE(__h - 2, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 1, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_0_5, __reg_0_6, __reg_0_0);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_5, __h + 0);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __STORE(__h - 9, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __LOAD(__reg_0_6, __h + 1);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __STORE(__h - 8, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __LOAD(__reg_0_0, __h + 2);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __STORE(__h - 7, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __LOAD(__reg_0_1, __h + 3);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __STORE(__h - 6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6);
        __STORE(__h - 5, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 3, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __STORE(__h - 2, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_0_6);
        __STORE(__h - 1, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_0_6, __reg_0_0);
        __STORE(__h + 0, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_0_6, __reg_0_0, __reg_0_1);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_5, __h + 0);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __STORE(__h - 9, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __LOAD(__reg_0_6, __h + 1);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __STORE(__h - 8, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __LOAD(__reg_0_0, __h + 2);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __STORE(__h - 7, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __LOAD(__reg_0_1, __h + 3);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __STORE(__h - 6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __LOAD(__reg_0_2, __h + 4);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __STORE(__h - 5, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0);
        __STORE(__h - 4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 3, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 2, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __STORE(__h - 1, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_0_0);
        __STORE(__h + 0, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_0_0, __reg_0_1);
        __STORE(__h + 1, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_0_0, __reg_0_1, __reg_0_2);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_5, __h + 0);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __STORE(__h - 9, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __LOAD(__reg_0_6, __h + 1);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __STORE(__h - 8, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __LOAD(__reg_0_0, __h + 2);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __STORE(__h - 7, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __LOAD(__reg_0_1, __h + 3);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __STORE(__h - 6, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __LOAD(__reg_0_2, __h + 4);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __STORE(__h - 5, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __LOAD(__reg_0_3, __h + 5);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __STORE(__h - 4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1);
        __STORE(__h - 3, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 2, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 1, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __STORE(__h + 0, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_0_1);
        __STORE(__h + 1, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_0_1, __reg_0_2);
        __STORE(__h + 2, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_0_1, __reg_0_2, __reg_0_3);
      }
    }
    else
    {
      for (__h = 19; __h <= __side1LenOl - 7;)
      {
        __LOAD(__reg_0_5, __h);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __STORE(__h - 9, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
        __h++;
        __LOAD(__reg_0_6, __h);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __STORE(__h - 9, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
        __h++;
        __LOAD(__reg_0_0, __h);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __STORE(__h - 9, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
        __h++;
        __LOAD(__reg_0_1, __h);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __STORE(__h - 9, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
        __h++;
        __LOAD(__reg_0_2, __h);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __STORE(__h - 9, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
        __h++;
        __LOAD(__reg_0_3, __h);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __STORE(__h - 9, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
        __h++;
        __LOAD(__reg_0_4, __h);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __STORE(__h - 9, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
        __h++;
        __DB_SWITCH();  __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_5, __h);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __CALC2(__reg_2_6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __STORE(__h - 9, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_6, __h);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __CALC2(__reg_2_0, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __STORE(__h - 9, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_0, __h);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __CALC2(__reg_2_1, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __STORE(__h - 9, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_1, __h);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __CALC2(__reg_2_2, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __STORE(__h - 9, __reg_2_3, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_2, __h);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __CALC2(__reg_2_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __STORE(__h - 9, __reg_2_4, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_3, __h);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __CALC2(__reg_2_4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __STORE(__h - 9, __reg_2_5, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_4, __h);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __CALC2(__reg_2_5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __STORE(__h - 9, __reg_2_6, __reg_2_0, __reg_2_1, __reg_2_2, __reg_2_3, __reg_2_4, __reg_2_5);
      __h++;
    }
}
__global__ void kernel0_2(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c1Pad = (3);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c2Pad = (3);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 3;
    const AN5D_TYPE __halo2 = 3;
    const AN5D_TYPE __side0Len = 2;
    const AN5D_TYPE __side1Len = 512;
    const AN5D_TYPE __side2Len = 500;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0_0;
    float __reg_0_1;
    float __reg_0_2;
    float __reg_0_3;
    float __reg_0_4;
    float __reg_0_5;
    float __reg_0_6;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_1_5;
    float __reg_1_6;
    __shared__ float __d_sb_double[__blockSize * 2];
    float *__d_sb = __d_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __storeValid = __writeValid2;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR(__rn0, __a, __b, __c, __d, __e, __f, __g) do { __rn0 = (((((((((((((0.06251f * (__REGREF(__a, 0))) + (0.06255f * (__REGREF(__b, 0)))) + (0.06245f * (__REGREF(__c, 0)))) + (0.06252f * (__SBREF(__d_sb, -3)))) + (0.06249f * (__SBREF(__d_sb, -2)))) + (0.06244f * (__SBREF(__d_sb, -1)))) + (0.25002f * (__REGREF(__d, 0)))) + (0.06248f * (__SBREF(__d_sb, 1)))) + (0.06243f * (__SBREF(__d_sb, 2)))) + (0.06253f * (__SBREF(__d_sb, 3)))) + (0.06246f * (__REGREF(__e, 0)))) + (0.06242f * (__REGREF(__f, 0)))) + (0.06254f * (__REGREF(__g, 0)))); } while (0)
    #define __DB_SWITCH() do { __d_sb = &__d_sb_double[(__d_sb == __d_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a, b, c, d, e, f, g) do { __DB_SWITCH(); __d_sb[__tid] = d; __syncthreads(); } while (0)
    #define __CALC1(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid1) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __STORE(h, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __CALCEXPR(__DEST, reg0, reg1, reg2, reg3, reg4, reg5, reg6); } } while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_1_0, 0);
      __LOAD(__reg_1_1, 1);
      __LOAD(__reg_1_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __CALC1(__reg_1_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __LOAD(__reg_0_0, 7);
      __CALC1(__reg_1_4, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __LOAD(__reg_0_1, 8);
      __CALC1(__reg_1_5, __reg_1_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __LOAD(__reg_0_2, 9);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __STORE(3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __LOAD(__reg_0_3, 10);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __STORE(4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __LOAD(__reg_0_4, 11);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __STORE(5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __LOAD(__reg_0_5, 12);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __STORE(6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
    }
    else
    {
      __LOAD(__reg_0_0, 0);
      __LOAD(__reg_0_1, 1);
      __LOAD(__reg_0_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __LOAD(__reg_0_0, 7);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __LOAD(__reg_0_1, 8);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __LOAD(__reg_0_2, 9);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __LOAD(__reg_0_3, 10);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __LOAD(__reg_0_4, 11);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __LOAD(__reg_0_5, 12);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __STORE(6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __DB_SWITCH(); __syncthreads();
    }
    __d_sb = __d_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 13; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 7;)
      {
        __LOAD(__reg_0_6, __h);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __h++;
        __LOAD(__reg_0_0, __h);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 6, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __h++;
        __LOAD(__reg_0_1, __h);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 6, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __h++;
        __LOAD(__reg_0_2, __h);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __h++;
        __LOAD(__reg_0_3, __h);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 6, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __h++;
        __LOAD(__reg_0_4, __h);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 6, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __h++;
        __LOAD(__reg_0_5, __h);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __h++;
      }
      if (0) {}
      else if (__h + 0 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3);
        __STORE(__h - 5, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 4, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4, __reg_0_5);
      }
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_6, __h + 0);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __STORE(__h - 5, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4);
        __STORE(__h - 4, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4, __reg_0_5, __reg_0_6);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_6, __h + 0);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __LOAD(__reg_0_0, __h + 1);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 5, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __STORE(__h - 4, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5);
        __STORE(__h - 3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 2, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5, __reg_0_6, __reg_0_0);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_6, __h + 0);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __LOAD(__reg_0_0, __h + 1);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 5, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __LOAD(__reg_0_1, __h + 2);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 4, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __STORE(__h - 3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6);
        __STORE(__h - 2, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6, __reg_0_0, __reg_0_1);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_6, __h + 0);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __LOAD(__reg_0_0, __h + 1);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 5, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __LOAD(__reg_0_1, __h + 2);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 4, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __LOAD(__reg_0_2, __h + 3);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __STORE(__h - 2, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0);
        __STORE(__h - 1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0, __reg_0_1);
        __STORE(__h + 0, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0, __reg_0_1, __reg_0_2);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_6, __h + 0);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __LOAD(__reg_0_0, __h + 1);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 5, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __LOAD(__reg_0_1, __h + 2);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 4, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __LOAD(__reg_0_2, __h + 3);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __LOAD(__reg_0_3, __h + 4);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 2, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __STORE(__h - 1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1);
        __STORE(__h + 0, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1, __reg_0_2);
        __STORE(__h + 1, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1, __reg_0_2, __reg_0_3);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_6, __h + 0);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __LOAD(__reg_0_0, __h + 1);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 5, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __LOAD(__reg_0_1, __h + 2);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 4, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __LOAD(__reg_0_2, __h + 3);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __LOAD(__reg_0_3, __h + 4);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 2, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __LOAD(__reg_0_4, __h + 5);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __STORE(__h + 0, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2);
        __STORE(__h + 1, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2, __reg_0_3);
        __STORE(__h + 2, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2, __reg_0_3, __reg_0_4);
      }
    }
    else
    {
      for (__h = 13; __h <= __side1LenOl - 7;)
      {
        __LOAD(__reg_0_6, __h);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __h++;
        __LOAD(__reg_0_0, __h);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 6, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __h++;
        __LOAD(__reg_0_1, __h);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 6, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __h++;
        __LOAD(__reg_0_2, __h);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __h++;
        __LOAD(__reg_0_3, __h);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 6, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __h++;
        __LOAD(__reg_0_4, __h);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 6, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __h++;
        __LOAD(__reg_0_5, __h);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __h++;
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_6, __h);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_0, __h);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __STORE(__h - 6, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_1, __h);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __STORE(__h - 6, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_2, __h);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __STORE(__h - 6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_3, __h);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __STORE(__h - 6, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_4, __h);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __STORE(__h - 6, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_5, __h);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __STORE(__h - 6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __h++;
    }
}
__global__ void kernel0_1(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c1Pad = (3);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c2Pad = (3);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 3;
    const AN5D_TYPE __halo2 = 3;
    const AN5D_TYPE __side0Len = 1;
    const AN5D_TYPE __side1Len = 512;
    const AN5D_TYPE __side2Len = 506;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0_0;
    float __reg_0_1;
    float __reg_0_2;
    float __reg_0_3;
    float __reg_0_4;
    float __reg_0_5;
    float __reg_0_6;
    __shared__ float __d_sb_double[__blockSize * 2];
    float *__d_sb = __d_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __storeValid = __writeValid1;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR(__rn0, __a, __b, __c, __d, __e, __f, __g) do { __rn0 = (((((((((((((0.06251f * (__REGREF(__a, 0))) + (0.06255f * (__REGREF(__b, 0)))) + (0.06245f * (__REGREF(__c, 0)))) + (0.06252f * (__SBREF(__d_sb, -3)))) + (0.06249f * (__SBREF(__d_sb, -2)))) + (0.06244f * (__SBREF(__d_sb, -1)))) + (0.25002f * (__REGREF(__d, 0)))) + (0.06248f * (__SBREF(__d_sb, 1)))) + (0.06243f * (__SBREF(__d_sb, 2)))) + (0.06253f * (__SBREF(__d_sb, 3)))) + (0.06246f * (__REGREF(__e, 0)))) + (0.06242f * (__REGREF(__f, 0)))) + (0.06254f * (__REGREF(__g, 0)))); } while (0)
    #define __DB_SWITCH() do { __d_sb = &__d_sb_double[(__d_sb == __d_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a, b, c, d, e, f, g) do { __DB_SWITCH(); __d_sb[__tid] = d; __syncthreads(); } while (0)
    #define __STORE(h, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __CALCEXPR(__DEST, reg0, reg1, reg2, reg3, reg4, reg5, reg6); } } while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0_0, 0);
      __LOAD(__reg_0_1, 1);
      __LOAD(__reg_0_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __STORE(3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
    }
    else
    {
      __LOAD(__reg_0_0, 0);
      __LOAD(__reg_0_1, 1);
      __LOAD(__reg_0_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __STORE(3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
    }
    __d_sb = __d_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 7; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 7;)
      {
        __LOAD(__reg_0_0, __h);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __h++;
        __LOAD(__reg_0_1, __h);
        __STORE(__h - 3, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __h++;
        __LOAD(__reg_0_2, __h);
        __STORE(__h - 3, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __h++;
        __LOAD(__reg_0_3, __h);
        __STORE(__h - 3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __h++;
        __LOAD(__reg_0_4, __h);
        __STORE(__h - 3, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __h++;
        __LOAD(__reg_0_5, __h);
        __STORE(__h - 3, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __h++;
        __LOAD(__reg_0_6, __h);
        __STORE(__h - 3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 0 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
      }
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 2, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 2, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __LOAD(__reg_0_2, __h + 2);
        __STORE(__h - 1, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 2, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __LOAD(__reg_0_2, __h + 2);
        __STORE(__h - 1, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __LOAD(__reg_0_3, __h + 3);
        __STORE(__h + 0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 2, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __LOAD(__reg_0_2, __h + 2);
        __STORE(__h - 1, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __LOAD(__reg_0_3, __h + 3);
        __STORE(__h + 0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __LOAD(__reg_0_4, __h + 4);
        __STORE(__h + 1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 2, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __LOAD(__reg_0_2, __h + 2);
        __STORE(__h - 1, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __LOAD(__reg_0_3, __h + 3);
        __STORE(__h + 0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __LOAD(__reg_0_4, __h + 4);
        __STORE(__h + 1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __LOAD(__reg_0_5, __h + 5);
        __STORE(__h + 2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      }
    }
    else
    {
      for (__h = 7; __h <= __side1LenOl - 7;)
      {
        __LOAD(__reg_0_0, __h);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __h++;
        __LOAD(__reg_0_1, __h);
        __STORE(__h - 3, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __h++;
        __LOAD(__reg_0_2, __h);
        __STORE(__h - 3, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __h++;
        __LOAD(__reg_0_3, __h);
        __STORE(__h - 3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __h++;
        __LOAD(__reg_0_4, __h);
        __STORE(__h - 3, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __h++;
        __LOAD(__reg_0_5, __h);
        __STORE(__h - 3, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __h++;
        __LOAD(__reg_0_6, __h);
        __STORE(__h - 3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __h++;
        __DB_SWITCH();  __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_0, __h);
      __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_1, __h);
      __STORE(__h - 3, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_2, __h);
      __STORE(__h - 3, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_3, __h);
      __STORE(__h - 3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_4, __h);
      __STORE(__h - 3, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_5, __h);
      __STORE(__h - 3, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_6, __h);
      __STORE(__h - 3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __h++;
    }
}
