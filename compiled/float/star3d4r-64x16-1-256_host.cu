#include <assert.h>
#include <stdio.h>
#include "star3d4r-64x16-1-256_kernel.hu"
#define BENCH_DIM 3
#define BENCH_FPP 49
#define BENCH_RAD 4

#include "common.h"

double kernel_stencil(SB_TYPE *A1, int compsize, int timestep, bool scop)
{
  double start_time = sb_time(), end_time = 0.0;
  int dimsize = compsize + BENCH_RAD * 2;
  SB_TYPE (*A)[dimsize][dimsize][dimsize]
    = (SB_TYPE (*)[dimsize][dimsize][dimsize])A1;

  if (scop) {
    if (dimsize >= 9 && timestep >= 1) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

      float *dev_A;
      
      cudaCheckReturn(hipMalloc((void **) &dev_A, (size_t)(2) * (size_t)(dimsize) * (size_t)(dimsize) * (size_t)(dimsize) * sizeof(float)));
      
{
      cudaCheckReturn(hipMemcpy(dev_A, A, (size_t)(2) * (size_t)(dimsize) * (size_t)(dimsize) * (size_t)(dimsize) * sizeof(float), hipMemcpyHostToDevice));
#ifdef STENCILBENCH
hipDeviceSynchronize();
SB_START_INSTRUMENTS;
#endif
}
    {
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
      const AN5D_TYPE __c0Len = (timestep - 0);
      const AN5D_TYPE __c0Pad = (0);
      #define __c0 c0
      const AN5D_TYPE __c1Len = (dimsize - 4 - 4);
      const AN5D_TYPE __c1Pad = (4);
      #define __c1 c1
      const AN5D_TYPE __c2Len = (dimsize - 4 - 4);
      const AN5D_TYPE __c2Pad = (4);
      #define __c2 c2
      const AN5D_TYPE __c3Len = (dimsize - 4 - 4);
      const AN5D_TYPE __c3Pad = (4);
      #define __c3 c3
      const AN5D_TYPE __halo1 = 4;
      const AN5D_TYPE __halo2 = 4;
      const AN5D_TYPE __halo3 = 4;
      AN5D_TYPE c0;
      AN5D_TYPE __side0LenMax;
      {
        const AN5D_TYPE __side0Len = 1;
        const AN5D_TYPE __side1Len = 256;
        const AN5D_TYPE __side2Len = 8;
        const AN5D_TYPE __side3Len = 56;
        const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
        const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
        const AN5D_TYPE __OlLen3 = (__halo3 * __side0Len);
        const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
        const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
        const AN5D_TYPE __side3LenOl = (__side3Len + 2 * __OlLen3);
        const AN5D_TYPE __blockSize = 1 * __side2LenOl * __side3LenOl;
        assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
        dim3 k0_dimBlock(__blockSize, 1, 1);
        dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len) * ((__c3Len + __side3Len - 1) / __side3Len), 1, 1);
        AN5D_TYPE __c0Padr = (__c0Len % 2) != (((__c0Len + __side0Len - 1) / __side0Len) % 2) && __c0Len % __side0Len < 2 ? 1 : 0;
        __side0LenMax = __side0Len;
        for (c0 = __c0Pad; c0 < __c0Pad + __c0Len / __side0Len - __c0Padr; c0 += 1)
        {
          kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
      }
      if ((__c0Len % 2) != (((__c0Len + __side0LenMax - 1) / __side0LenMax) % 2))
      {
        if (__c0Len % __side0LenMax == 0)
        {
        }
      }
      else if (__c0Len % __side0LenMax)
      {
      }
    }
    cudaCheckKernel();
{
#ifdef STENCILBENCH
hipDeviceSynchronize();
SB_STOP_INSTRUMENTS;
#endif
      cudaCheckReturn(hipMemcpy(A, dev_A, (size_t)(2) * (size_t)(dimsize) * (size_t)(dimsize) * (size_t)(dimsize) * sizeof(float), hipMemcpyDeviceToHost));
}
      cudaCheckReturn(hipFree(dev_A));
    }
  }
  else {
    for (int t = 0; t < timestep; t++)
#pragma omp parallel for
      for (int i = BENCH_RAD; i < dimsize - BENCH_RAD; i++)
        for (int j = BENCH_RAD; j < dimsize - BENCH_RAD; j++)
          for (int k = BENCH_RAD; k < dimsize - BENCH_RAD; k++)
            A[(t+1)%2][i][j][k] =
              0.25000f * A[t%2][i][j][k]
              + 0.03228f * A[t%2][i][j][k-4] + 0.03138f * A[t%2][i][j][k-3]
              + 0.03118f * A[t%2][i][j][k-2] + 0.03027f * A[t%2][i][j][k-1]
              + 0.03022f * A[t%2][i][j][k+1] + 0.03112f * A[t%2][i][j][k+2]
              + 0.03132f * A[t%2][i][j][k+3] + 0.03222f * A[t%2][i][j][k+4]

              + 0.03026f * A[t%2][i-1][j][k] + 0.03024f * A[t%2][i+1][j][k]
              + 0.03027f * A[t%2][i][j-1][k] + 0.03023f * A[t%2][i][j+1][k]

              + 0.03116f * A[t%2][i-2][j][k] + 0.03114f * A[t%2][i+2][j][k]
              + 0.03117f * A[t%2][i][j-2][k] + 0.03113f * A[t%2][i][j+2][k]

              + 0.03136f * A[t%2][i-3][j][k] + 0.03134f * A[t%2][i+3][j][k]
              + 0.03137f * A[t%2][i][j-3][k] + 0.03133f * A[t%2][i][j+3][k]

              + 0.03226f * A[t%2][i-4][j][k] + 0.03224f * A[t%2][i+4][j][k]
              + 0.03227f * A[t%2][i][j-4][k] + 0.03223f * A[t%2][i][j+4][k];
  }

  return (((end_time != 0.0) ? end_time : sb_time()) - start_time);
}
