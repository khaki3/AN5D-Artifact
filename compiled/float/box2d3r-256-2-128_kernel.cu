#include "hip/hip_runtime.h"
#include "box2d3r-256-2-128_kernel.hu"
__device__ float __sbref_wrap(float *sb, size_t index) { return sb[index]; }

__global__ void kernel0_2(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c1Pad = (3);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c2Pad = (3);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 3;
    const AN5D_TYPE __halo2 = 3;
    const AN5D_TYPE __side0Len = 2;
    const AN5D_TYPE __side1Len = 128;
    const AN5D_TYPE __side2Len = 244;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_1_5;
    float __reg_1_6;
    float __reg_2_0;
    float __reg_2_1;
    float __reg_2_2;
    float __reg_2_3;
    float __reg_2_4;
    float __reg_2_5;
    float __reg_2_6;
    __shared__ float __a_sb_double[__blockSize * 2];
    float *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __storeValid = __writeValid2;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((0.01530f * (__SBREF(__a_sb, -3))) + (0.01531f * (__SBREF(__a_sb, -2)))) + (0.01532f * (__SBREF(__a_sb, -1)))) + (0.01533f * (__REGREF(__a, 0)))) + (0.01534f * (__SBREF(__a_sb, 1)))) + (0.01535f * (__SBREF(__a_sb, 2)))) + (0.01536f * (__SBREF(__a_sb, 3)))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((0.01537f * (__SBREF(__a_sb, -3)))) + (0.01538f * (__SBREF(__a_sb, -2)))) + (0.01539f * (__SBREF(__a_sb, -1)))) + (0.01540f * (__REGREF(__a, 0)))) + (0.01541f * (__SBREF(__a_sb, 1)))) + (0.01542f * (__SBREF(__a_sb, 2)))) + (0.01543f * (__SBREF(__a_sb, 3))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { float etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((0.01544f * (__SBREF(__a_sb, -3)))) + (0.01545f * (__SBREF(__a_sb, -2)))) + (0.01546f * (__SBREF(__a_sb, -1)))) + (0.01546f * (__REGREF(__a, 0)))) + (0.01547f * (__SBREF(__a_sb, 1)))) + (0.01548f * (__SBREF(__a_sb, 2)))) + (0.01549f * (__SBREF(__a_sb, 3)))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { float etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_3_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((0.01550f * (__SBREF(__a_sb, -3)))) + (0.01551f * (__SBREF(__a_sb, -2)))) + (0.01552f * (__SBREF(__a_sb, -1)))) + (0.25424f * (__REGREF(__a, 0)))) + (0.01554f * (__SBREF(__a_sb, 1)))) + (0.01555f * (__SBREF(__a_sb, 2)))) + (0.01556f * (__SBREF(__a_sb, 3))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_3(out, a) do { float etmp; __CALCEXPR_3_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_4_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((0.01557f * (__SBREF(__a_sb, -3)))) + (0.01558f * (__SBREF(__a_sb, -2)))) + (0.01559f * (__SBREF(__a_sb, -1)))) + (0.01560f * (__REGREF(__a, 0)))) + (0.01561f * (__SBREF(__a_sb, 1)))) + (0.01562f * (__SBREF(__a_sb, 2)))) + (0.01564f * (__SBREF(__a_sb, 3)))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_4(out, a) do { float etmp; __CALCEXPR_4_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_5_wrap(__rn0, __a) do { __rn0 = (((((((((((((((0.01565f * (__SBREF(__a_sb, -3)))) + (0.01566f * (__SBREF(__a_sb, -2)))) + (0.01567f * (__SBREF(__a_sb, -1)))) + (0.01568f * (__REGREF(__a, 0)))) + (0.01569f * (__SBREF(__a_sb, 1)))) + (0.01570f * (__SBREF(__a_sb, 2)))) + (0.01571f * (__SBREF(__a_sb, 3))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_5(out, a) do { float etmp; __CALCEXPR_5_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_6_wrap(__rn0, __a) do { __rn0 = ((((((((0.01572f * (__SBREF(__a_sb, -3)))) + (0.01573f * (__SBREF(__a_sb, -2)))) + (0.01574f * (__SBREF(__a_sb, -1)))) + (0.01575f * (__REGREF(__a, 0)))) + (0.01576f * (__SBREF(__a_sb, 1)))) + (0.01577f * (__SBREF(__a_sb, 2)))) + (0.01578f * (__SBREF(__a_sb, 3)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_6(out, a) do { float etmp; __CALCEXPR_6_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); __CALCEXPR_3(out3, reg); __CALCEXPR_4(out4, reg); __CALCEXPR_5(out5, reg); __CALCEXPR_6(out6, reg); } while (0);
    #define __CALC1(out0, out1, out2, out3, out4, out5, out6, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, reg); } else out3 = reg; } while (0)
    #define __CALC2(out0, out1, out2, out3, out4, out5, out6, reg) do { __CALCSETUP(reg); if (__writeValid2) { __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, reg); } else out3 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_3);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_4);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_5);
      __LOAD(__reg_0, 9);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_6);
      __STORE(3, __reg_2_3);
      __LOAD(__reg_0, 10);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_0);
      __STORE(4, __reg_2_4);
      __LOAD(__reg_0, 11);
      __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_1_1);
      __STORE(5, __reg_2_5);
      __LOAD(__reg_0, 12);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_1_2);
      __STORE(6, __reg_2_6);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_3);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_4);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_5);
      __LOAD(__reg_0, 9);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_6);
      __LOAD(__reg_0, 10);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_0);
      __LOAD(__reg_0, 11);
      __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_1_1);
      __LOAD(__reg_0, 12);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_1_2);
      __STORE(6, __reg_2_6);
      __DB_SWITCH(); __syncthreads();
    }
    __a_sb = __a_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 13; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 10;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_3);
        __STORE(__h - 6, __reg_2_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_4);
        __STORE(__h - 6, __reg_2_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_5);
        __STORE(__h - 6, __reg_2_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_6);
        __STORE(__h - 6, __reg_2_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_0);
        __STORE(__h - 6, __reg_2_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_1_1);
        __STORE(__h - 6, __reg_2_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_1_2);
        __STORE(__h - 6, __reg_2_6);
        __h++;
      }
      if (0) {}
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_3);
        __STORE(__h - 6, __reg_2_0);
        __reg_1_6 = __reg_0;
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_4);
        __STORE(__h - 5, __reg_2_1);
        __reg_1_0 = __reg_0;
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_5, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_5);
        __STORE(__h - 4, __reg_2_2);
        __reg_1_1 = __reg_0;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_6);
        __STORE(__h - 3, __reg_2_3);
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_5, __reg_2_4, __reg_1_0);
        __STORE(__h - 2, __reg_2_4);
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_5, __reg_1_1);
        __STORE(__h - 1, __reg_2_5);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_3);
        __STORE(__h - 6, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_4);
        __STORE(__h - 5, __reg_2_1);
        __reg_1_0 = __reg_0;
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_5);
        __STORE(__h - 4, __reg_2_2);
        __reg_1_1 = __reg_0;
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_6, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_6);
        __STORE(__h - 3, __reg_2_3);
        __reg_1_2 = __reg_0;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_0);
        __STORE(__h - 2, __reg_2_4);
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_6, __reg_2_5, __reg_1_1);
        __STORE(__h - 1, __reg_2_5);
        __CALC2(__reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_6, __reg_1_2);
        __STORE(__h + 0, __reg_2_6);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_3);
        __STORE(__h - 6, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_4);
        __STORE(__h - 5, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_5);
        __STORE(__h - 4, __reg_2_2);
        __reg_1_1 = __reg_0;
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_0, __reg_1_6, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_6);
        __STORE(__h - 3, __reg_2_3);
        __reg_1_2 = __reg_0;
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_0, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_0);
        __STORE(__h - 2, __reg_2_4);
        __reg_1_3 = __reg_0;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_0, __reg_2_6, __reg_2_5, __reg_1_1);
        __STORE(__h - 1, __reg_2_5);
        __CALC2(__reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_0, __reg_2_6, __reg_1_2);
        __STORE(__h + 0, __reg_2_6);
        __CALC2(__reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_0, __reg_1_3);
        __STORE(__h + 1, __reg_2_0);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_3);
        __STORE(__h - 6, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_4);
        __STORE(__h - 5, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_5);
        __STORE(__h - 4, __reg_2_2);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_6);
        __STORE(__h - 3, __reg_2_3);
        __reg_1_2 = __reg_0;
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_0);
        __STORE(__h - 2, __reg_2_4);
        __reg_1_3 = __reg_0;
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_1_1);
        __STORE(__h - 1, __reg_2_5);
        __reg_1_4 = __reg_0;
        __CALC2(__reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_1, __reg_2_0, __reg_2_6, __reg_1_2);
        __STORE(__h + 0, __reg_2_6);
        __CALC2(__reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_1, __reg_2_0, __reg_1_3);
        __STORE(__h + 1, __reg_2_0);
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_1, __reg_1_4);
        __STORE(__h + 2, __reg_2_1);
      }
      else if (__h + 7 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_3);
        __STORE(__h - 6, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_4);
        __STORE(__h - 5, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_5);
        __STORE(__h - 4, __reg_2_2);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_6);
        __STORE(__h - 3, __reg_2_3);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_0);
        __STORE(__h - 2, __reg_2_4);
        __reg_1_3 = __reg_0;
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_1_1);
        __STORE(__h - 1, __reg_2_5);
        __reg_1_4 = __reg_0;
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_1_2);
        __STORE(__h + 0, __reg_2_6);
        __reg_1_5 = __reg_0;
        __CALC2(__reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_3);
        __STORE(__h + 1, __reg_2_0);
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_2, __reg_2_1, __reg_1_4);
        __STORE(__h + 2, __reg_2_1);
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_2, __reg_1_5);
        __STORE(__h + 3, __reg_2_2);
      }
      else if (__h + 8 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_3);
        __STORE(__h - 6, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_4);
        __STORE(__h - 5, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_5);
        __STORE(__h - 4, __reg_2_2);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_6);
        __STORE(__h - 3, __reg_2_3);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_0);
        __STORE(__h - 2, __reg_2_4);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_1_1);
        __STORE(__h - 1, __reg_2_5);
        __reg_1_4 = __reg_0;
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_1_2);
        __STORE(__h + 0, __reg_2_6);
        __reg_1_5 = __reg_0;
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_3, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_3);
        __STORE(__h + 1, __reg_2_0);
        __reg_1_6 = __reg_0;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_4);
        __STORE(__h + 2, __reg_2_1);
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_3, __reg_2_2, __reg_1_5);
        __STORE(__h + 3, __reg_2_2);
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_3, __reg_1_6);
        __STORE(__h + 4, __reg_2_3);
      }
      else if (__h + 9 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_3);
        __STORE(__h - 6, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_4);
        __STORE(__h - 5, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_5);
        __STORE(__h - 4, __reg_2_2);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_6);
        __STORE(__h - 3, __reg_2_3);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_0);
        __STORE(__h - 2, __reg_2_4);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_1_1);
        __STORE(__h - 1, __reg_2_5);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_1_2);
        __STORE(__h + 0, __reg_2_6);
        __reg_1_5 = __reg_0;
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_3);
        __STORE(__h + 1, __reg_2_0);
        __reg_1_6 = __reg_0;
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_4, __reg_0);
        __CALC2(__reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_4);
        __STORE(__h + 2, __reg_2_1);
        __reg_1_0 = __reg_0;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_5);
        __STORE(__h + 3, __reg_2_2);
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_4, __reg_2_3, __reg_1_6);
        __STORE(__h + 4, __reg_2_3);
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_4, __reg_1_0);
        __STORE(__h + 5, __reg_2_4);
      }
    }
    else
    {
      for (__h = 13; __h <= __side1LenOl - 7;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_3);
        __STORE(__h - 6, __reg_2_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_4);
        __STORE(__h - 6, __reg_2_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_5);
        __STORE(__h - 6, __reg_2_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_6);
        __STORE(__h - 6, __reg_2_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_0);
        __STORE(__h - 6, __reg_2_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_1_1);
        __STORE(__h - 6, __reg_2_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_1_2);
        __STORE(__h - 6, __reg_2_6);
        __h++;
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_3);
      __STORE(__h - 6, __reg_2_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_4);
      __STORE(__h - 6, __reg_2_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_5);
      __STORE(__h - 6, __reg_2_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_6);
      __STORE(__h - 6, __reg_2_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_0);
      __STORE(__h - 6, __reg_2_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_2_5, __reg_1_1);
      __STORE(__h - 6, __reg_2_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_6, __reg_1_2);
      __STORE(__h - 6, __reg_2_6);
      __h++;
    }
}
__global__ void kernel0_1(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c1Pad = (3);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c2Pad = (3);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 3;
    const AN5D_TYPE __halo2 = 3;
    const AN5D_TYPE __side0Len = 1;
    const AN5D_TYPE __side1Len = 128;
    const AN5D_TYPE __side2Len = 250;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_1_5;
    float __reg_1_6;
    __shared__ float __a_sb_double[__blockSize * 2];
    float *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __storeValid = __writeValid1;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((0.01530f * (__SBREF(__a_sb, -3))) + (0.01531f * (__SBREF(__a_sb, -2)))) + (0.01532f * (__SBREF(__a_sb, -1)))) + (0.01533f * (__REGREF(__a, 0)))) + (0.01534f * (__SBREF(__a_sb, 1)))) + (0.01535f * (__SBREF(__a_sb, 2)))) + (0.01536f * (__SBREF(__a_sb, 3)))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((0.01537f * (__SBREF(__a_sb, -3)))) + (0.01538f * (__SBREF(__a_sb, -2)))) + (0.01539f * (__SBREF(__a_sb, -1)))) + (0.01540f * (__REGREF(__a, 0)))) + (0.01541f * (__SBREF(__a_sb, 1)))) + (0.01542f * (__SBREF(__a_sb, 2)))) + (0.01543f * (__SBREF(__a_sb, 3))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { float etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((0.01544f * (__SBREF(__a_sb, -3)))) + (0.01545f * (__SBREF(__a_sb, -2)))) + (0.01546f * (__SBREF(__a_sb, -1)))) + (0.01546f * (__REGREF(__a, 0)))) + (0.01547f * (__SBREF(__a_sb, 1)))) + (0.01548f * (__SBREF(__a_sb, 2)))) + (0.01549f * (__SBREF(__a_sb, 3)))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { float etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_3_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((0.01550f * (__SBREF(__a_sb, -3)))) + (0.01551f * (__SBREF(__a_sb, -2)))) + (0.01552f * (__SBREF(__a_sb, -1)))) + (0.25424f * (__REGREF(__a, 0)))) + (0.01554f * (__SBREF(__a_sb, 1)))) + (0.01555f * (__SBREF(__a_sb, 2)))) + (0.01556f * (__SBREF(__a_sb, 3))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_3(out, a) do { float etmp; __CALCEXPR_3_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_4_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((0.01557f * (__SBREF(__a_sb, -3)))) + (0.01558f * (__SBREF(__a_sb, -2)))) + (0.01559f * (__SBREF(__a_sb, -1)))) + (0.01560f * (__REGREF(__a, 0)))) + (0.01561f * (__SBREF(__a_sb, 1)))) + (0.01562f * (__SBREF(__a_sb, 2)))) + (0.01564f * (__SBREF(__a_sb, 3)))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_4(out, a) do { float etmp; __CALCEXPR_4_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_5_wrap(__rn0, __a) do { __rn0 = (((((((((((((((0.01565f * (__SBREF(__a_sb, -3)))) + (0.01566f * (__SBREF(__a_sb, -2)))) + (0.01567f * (__SBREF(__a_sb, -1)))) + (0.01568f * (__REGREF(__a, 0)))) + (0.01569f * (__SBREF(__a_sb, 1)))) + (0.01570f * (__SBREF(__a_sb, 2)))) + (0.01571f * (__SBREF(__a_sb, 3))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_5(out, a) do { float etmp; __CALCEXPR_5_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_6_wrap(__rn0, __a) do { __rn0 = ((((((((0.01572f * (__SBREF(__a_sb, -3)))) + (0.01573f * (__SBREF(__a_sb, -2)))) + (0.01574f * (__SBREF(__a_sb, -1)))) + (0.01575f * (__REGREF(__a, 0)))) + (0.01576f * (__SBREF(__a_sb, 1)))) + (0.01577f * (__SBREF(__a_sb, 2)))) + (0.01578f * (__SBREF(__a_sb, 3)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_6(out, a) do { float etmp; __CALCEXPR_6_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); __CALCEXPR_3(out3, reg); __CALCEXPR_4(out4, reg); __CALCEXPR_5(out5, reg); __CALCEXPR_6(out6, reg); } while (0);
    #define __CALC1(out0, out1, out2, out3, out4, out5, out6, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, reg); } else out3 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __STORE(3, __reg_1_3);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __STORE(3, __reg_1_3);
    }
    __a_sb = __a_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 7; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 10;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 3, __reg_1_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 3, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 3, __reg_1_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h - 3, __reg_1_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 3, __reg_1_3);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 2, __reg_1_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_6, __reg_0);
        __STORE(__h - 1, __reg_1_6);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 2, __reg_1_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 1, __reg_1_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 2, __reg_1_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 1, __reg_1_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 2, __reg_1_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 1, __reg_1_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
      }
      else if (__h + 7 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 2, __reg_1_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 1, __reg_1_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
      }
      else if (__h + 8 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 2, __reg_1_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 1, __reg_1_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_4, __reg_0);
        __STORE(__h + 4, __reg_1_4);
      }
      else if (__h + 9 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 2, __reg_1_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 1, __reg_1_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h + 4, __reg_1_4);
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_5, __reg_0);
        __STORE(__h + 5, __reg_1_5);
      }
    }
    else
    {
      for (__h = 7; __h <= __side1LenOl - 7;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 3, __reg_1_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 3, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 3, __reg_1_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h - 3, __reg_1_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 3, __reg_1_3);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __STORE(__h - 3, __reg_1_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
      __STORE(__h - 3, __reg_1_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
      __STORE(__h - 3, __reg_1_6);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __STORE(__h - 3, __reg_1_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __STORE(__h - 3, __reg_1_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __STORE(__h - 3, __reg_1_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __STORE(__h - 3, __reg_1_3);
      __h++;
    }
}
