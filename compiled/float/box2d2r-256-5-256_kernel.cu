#include "hip/hip_runtime.h"
#include "box2d2r-256-5-256_kernel.hu"
__device__ float __sbref_wrap(float *sb, size_t index) { return sb[index]; }

__global__ void kernel0_5(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 2 - 2);
    const AN5D_TYPE __c1Pad = (2);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 2 - 2);
    const AN5D_TYPE __c2Pad = (2);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 2;
    const AN5D_TYPE __halo2 = 2;
    const AN5D_TYPE __side0Len = 5;
    const AN5D_TYPE __side1Len = 256;
    const AN5D_TYPE __side2Len = 236;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_2_0;
    float __reg_2_1;
    float __reg_2_2;
    float __reg_2_3;
    float __reg_2_4;
    float __reg_3_0;
    float __reg_3_1;
    float __reg_3_2;
    float __reg_3_3;
    float __reg_3_4;
    float __reg_4_0;
    float __reg_4_1;
    float __reg_4_2;
    float __reg_4_3;
    float __reg_4_4;
    float __reg_5_0;
    float __reg_5_1;
    float __reg_5_2;
    float __reg_5_3;
    float __reg_5_4;
    __shared__ float __a_sb_double[__blockSize * 2];
    float *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __writeValid3 = __updateValid && __local_c2 >= (__halo2 * 3) && __local_c2 < __side2LenOl - (__halo2 * 3);
    const AN5D_TYPE __writeValid4 = __updateValid && __local_c2 >= (__halo2 * 4) && __local_c2 < __side2LenOl - (__halo2 * 4);
    const AN5D_TYPE __writeValid5 = __updateValid && __local_c2 >= (__halo2 * 5) && __local_c2 < __side2LenOl - (__halo2 * 5);
    const AN5D_TYPE __storeValid = __writeValid5;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((0.03125f * (__SBREF(__a_sb, -2))) + (0.03126f * (__SBREF(__a_sb, -1)))) + (0.03127f * (__REGREF(__a, 0)))) + (0.03128f * (__SBREF(__a_sb, 1)))) + (0.03129f * (__SBREF(__a_sb, 2)))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((0.03130f * (__SBREF(__a_sb, -2)))) + (0.03131f * (__SBREF(__a_sb, -1)))) + (0.03132f * (__REGREF(__a, 0)))) + (0.03133f * (__SBREF(__a_sb, 1)))) + (0.03134f * (__SBREF(__a_sb, 2))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { float etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((0.03135f * (__SBREF(__a_sb, -2)))) + (0.03136f * (__SBREF(__a_sb, -1)))) + (0.24712f * (__REGREF(__a, 0)))) + (0.03138f * (__SBREF(__a_sb, 1)))) + (0.03139f * (__SBREF(__a_sb, 2)))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { float etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_3_wrap(__rn0, __a) do { __rn0 = (((((((((((0.03140f * (__SBREF(__a_sb, -2)))) + (0.03141f * (__SBREF(__a_sb, -1)))) + (0.03142f * (__REGREF(__a, 0)))) + (0.03143f * (__SBREF(__a_sb, 1)))) + (0.03144f * (__SBREF(__a_sb, 2))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_3(out, a) do { float etmp; __CALCEXPR_3_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_4_wrap(__rn0, __a) do { __rn0 = ((((((0.03145f * (__SBREF(__a_sb, -2)))) + (0.03146f * (__SBREF(__a_sb, -1)))) + (0.03147f * (__REGREF(__a, 0)))) + (0.03148f * (__SBREF(__a_sb, 1)))) + (0.03149f * (__SBREF(__a_sb, 2)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_4(out, a) do { float etmp; __CALCEXPR_4_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, out3, out4, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); __CALCEXPR_3(out3, reg); __CALCEXPR_4(out4, reg); } while (0);
    #define __CALC1(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __CALC2(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid2) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __CALC3(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid3) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __CALC4(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid4) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __CALC5(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid5) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_0);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_0);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_0);
      __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_0);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_0);
      __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_0);
      __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
      __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
      __LOAD(__reg_0, 9);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
      __LOAD(__reg_0, 10);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
      __CALC5(__reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_4_2);
      __LOAD(__reg_0, 11);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
      __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
      __LOAD(__reg_0, 12);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
      __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
      __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
      __STORE(2, __reg_5_2);
      __LOAD(__reg_0, 13);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
      __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
      __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
      __STORE(3, __reg_5_3);
      __LOAD(__reg_0, 14);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
      __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_4_1);
      __STORE(4, __reg_5_4);
      __LOAD(__reg_0, 15);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
      __CALC5(__reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_4_2);
      __STORE(5, __reg_5_0);
      __LOAD(__reg_0, 16);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
      __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
      __STORE(6, __reg_5_1);
      __LOAD(__reg_0, 17);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
      __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
      __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
      __STORE(7, __reg_5_2);
      __LOAD(__reg_0, 18);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
      __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
      __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
      __STORE(8, __reg_5_3);
      __LOAD(__reg_0, 19);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
      __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_4_1);
      __STORE(9, __reg_5_4);
      __LOAD(__reg_0, 20);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
      __CALC5(__reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_4_2);
      __STORE(10, __reg_5_0);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
      __LOAD(__reg_0, 9);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
      __LOAD(__reg_0, 10);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
      __LOAD(__reg_0, 11);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __LOAD(__reg_0, 12);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
      __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
      __LOAD(__reg_0, 13);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
      __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
      __LOAD(__reg_0, 14);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
      __LOAD(__reg_0, 15);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
      __LOAD(__reg_0, 16);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
      __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
      __LOAD(__reg_0, 17);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
      __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
      __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
      __LOAD(__reg_0, 18);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
      __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
      __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
      __LOAD(__reg_0, 19);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
      __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_4_1);
      __LOAD(__reg_0, 20);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
      __CALC5(__reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_4_2);
      __STORE(10, __reg_5_0);
    }
    __a_sb = __a_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 21; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 7;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
        __STORE(__h - 10, __reg_5_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
        __STORE(__h - 10, __reg_5_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
        __STORE(__h - 10, __reg_5_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_4_1);
        __STORE(__h - 10, __reg_5_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __CALC5(__reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_4_2);
        __STORE(__h - 10, __reg_5_0);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
        __STORE(__h - 10, __reg_5_1);
        __reg_1_1 = __reg_0;
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
        __STORE(__h - 9, __reg_5_2);
        __reg_1_2 = __reg_0;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
        __STORE(__h - 8, __reg_5_3);
        __reg_2_1 = __reg_1_1;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_4_1);
        __STORE(__h - 7, __reg_5_4);
        __reg_2_2 = __reg_1_2;
        __CALC3(__reg_3_3, __reg_3_3, __reg_3_3, __reg_3_0, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __CALC5(__reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_4_2);
        __STORE(__h - 6, __reg_5_0);
        __reg_3_1 = __reg_2_1;
        __CALC3(__reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
        __STORE(__h - 5, __reg_5_1);
        __reg_3_2 = __reg_2_2;
        __CALC4(__reg_4_3, __reg_4_3, __reg_4_3, __reg_4_0, __reg_4_4, __reg_3_1);
        __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
        __STORE(__h - 4, __reg_5_2);
        __reg_4_1 = __reg_3_1;
        __CALC4(__reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_0, __reg_3_2);
        __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
        __STORE(__h - 3, __reg_5_3);
        __reg_4_2 = __reg_3_2;
        __CALC5(__reg_5_3, __reg_5_3, __reg_5_3, __reg_5_0, __reg_5_4, __reg_4_1);
        __STORE(__h - 2, __reg_5_4);
        __CALC5(__reg_5_4, __reg_5_4, __reg_5_4, __reg_5_4, __reg_5_0, __reg_4_2);
        __STORE(__h - 1, __reg_5_0);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
        __STORE(__h - 10, __reg_5_1);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
        __STORE(__h - 9, __reg_5_2);
        __reg_1_2 = __reg_0;
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
        __STORE(__h - 8, __reg_5_3);
        __reg_1_3 = __reg_0;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_4_1);
        __STORE(__h - 7, __reg_5_4);
        __reg_2_2 = __reg_1_2;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __CALC5(__reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_4_2);
        __STORE(__h - 6, __reg_5_0);
        __reg_2_3 = __reg_1_3;
        __CALC3(__reg_3_4, __reg_3_4, __reg_3_4, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
        __STORE(__h - 5, __reg_5_1);
        __reg_3_2 = __reg_2_2;
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
        __STORE(__h - 4, __reg_5_2);
        __reg_3_3 = __reg_2_3;
        __CALC4(__reg_4_4, __reg_4_4, __reg_4_4, __reg_4_1, __reg_4_0, __reg_3_2);
        __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
        __STORE(__h - 3, __reg_5_3);
        __reg_4_2 = __reg_3_2;
        __CALC4(__reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_1, __reg_3_3);
        __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_4_1);
        __STORE(__h - 2, __reg_5_4);
        __reg_4_3 = __reg_3_3;
        __CALC5(__reg_5_4, __reg_5_4, __reg_5_4, __reg_5_1, __reg_5_0, __reg_4_2);
        __STORE(__h - 1, __reg_5_0);
        __CALC5(__reg_5_0, __reg_5_0, __reg_5_0, __reg_5_0, __reg_5_1, __reg_4_3);
        __STORE(__h + 0, __reg_5_1);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
        __STORE(__h - 10, __reg_5_1);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
        __STORE(__h - 9, __reg_5_2);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
        __STORE(__h - 8, __reg_5_3);
        __reg_1_3 = __reg_0;
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_4_1);
        __STORE(__h - 7, __reg_5_4);
        __reg_1_4 = __reg_0;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __CALC5(__reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_4_2);
        __STORE(__h - 6, __reg_5_0);
        __reg_2_3 = __reg_1_3;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
        __STORE(__h - 5, __reg_5_1);
        __reg_2_4 = __reg_1_4;
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_0, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
        __STORE(__h - 4, __reg_5_2);
        __reg_3_3 = __reg_2_3;
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
        __STORE(__h - 3, __reg_5_3);
        __reg_3_4 = __reg_2_4;
        __CALC4(__reg_4_0, __reg_4_0, __reg_4_0, __reg_4_2, __reg_4_1, __reg_3_3);
        __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_4_1);
        __STORE(__h - 2, __reg_5_4);
        __reg_4_3 = __reg_3_3;
        __CALC4(__reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_2, __reg_3_4);
        __CALC5(__reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_4_2);
        __STORE(__h - 1, __reg_5_0);
        __reg_4_4 = __reg_3_4;
        __CALC5(__reg_5_0, __reg_5_0, __reg_5_0, __reg_5_2, __reg_5_1, __reg_4_3);
        __STORE(__h + 0, __reg_5_1);
        __CALC5(__reg_5_1, __reg_5_1, __reg_5_1, __reg_5_1, __reg_5_2, __reg_4_4);
        __STORE(__h + 1, __reg_5_2);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
        __STORE(__h - 10, __reg_5_1);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
        __STORE(__h - 9, __reg_5_2);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
        __STORE(__h - 8, __reg_5_3);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_4_1);
        __STORE(__h - 7, __reg_5_4);
        __reg_1_4 = __reg_0;
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __CALC5(__reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_4_2);
        __STORE(__h - 6, __reg_5_0);
        __reg_1_0 = __reg_0;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
        __STORE(__h - 5, __reg_5_1);
        __reg_2_4 = __reg_1_4;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
        __STORE(__h - 4, __reg_5_2);
        __reg_2_0 = __reg_1_0;
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_1, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
        __STORE(__h - 3, __reg_5_3);
        __reg_3_4 = __reg_2_4;
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_4_1);
        __STORE(__h - 2, __reg_5_4);
        __reg_3_0 = __reg_2_0;
        __CALC4(__reg_4_1, __reg_4_1, __reg_4_1, __reg_4_3, __reg_4_2, __reg_3_4);
        __CALC5(__reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_4_2);
        __STORE(__h - 1, __reg_5_0);
        __reg_4_4 = __reg_3_4;
        __CALC4(__reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_3, __reg_3_0);
        __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
        __STORE(__h + 0, __reg_5_1);
        __reg_4_0 = __reg_3_0;
        __CALC5(__reg_5_1, __reg_5_1, __reg_5_1, __reg_5_3, __reg_5_2, __reg_4_4);
        __STORE(__h + 1, __reg_5_2);
        __CALC5(__reg_5_2, __reg_5_2, __reg_5_2, __reg_5_2, __reg_5_3, __reg_4_0);
        __STORE(__h + 2, __reg_5_3);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
        __STORE(__h - 10, __reg_5_1);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
        __STORE(__h - 9, __reg_5_2);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
        __STORE(__h - 8, __reg_5_3);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_4_1);
        __STORE(__h - 7, __reg_5_4);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __CALC5(__reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_4_2);
        __STORE(__h - 6, __reg_5_0);
        __reg_1_0 = __reg_0;
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
        __STORE(__h - 5, __reg_5_1);
        __reg_1_1 = __reg_0;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
        __STORE(__h - 4, __reg_5_2);
        __reg_2_0 = __reg_1_0;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
        __STORE(__h - 3, __reg_5_3);
        __reg_2_1 = __reg_1_1;
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_2, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_4_1);
        __STORE(__h - 2, __reg_5_4);
        __reg_3_0 = __reg_2_0;
        __CALC3(__reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __CALC5(__reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_4_2);
        __STORE(__h - 1, __reg_5_0);
        __reg_3_1 = __reg_2_1;
        __CALC4(__reg_4_2, __reg_4_2, __reg_4_2, __reg_4_4, __reg_4_3, __reg_3_0);
        __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
        __STORE(__h + 0, __reg_5_1);
        __reg_4_0 = __reg_3_0;
        __CALC4(__reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_4, __reg_3_1);
        __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
        __STORE(__h + 1, __reg_5_2);
        __reg_4_1 = __reg_3_1;
        __CALC5(__reg_5_2, __reg_5_2, __reg_5_2, __reg_5_4, __reg_5_3, __reg_4_0);
        __STORE(__h + 2, __reg_5_3);
        __CALC5(__reg_5_3, __reg_5_3, __reg_5_3, __reg_5_3, __reg_5_4, __reg_4_1);
        __STORE(__h + 3, __reg_5_4);
      }
    }
    else
    {
      for (__h = 21; __h <= __side1LenOl - 5;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
        __STORE(__h - 10, __reg_5_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
        __STORE(__h - 10, __reg_5_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
        __STORE(__h - 10, __reg_5_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_4_1);
        __STORE(__h - 10, __reg_5_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __CALC5(__reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_4_2);
        __STORE(__h - 10, __reg_5_0);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
      __CALC5(__reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_4_3);
      __STORE(__h - 10, __reg_5_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
      __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
      __CALC5(__reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_5_2, __reg_4_4);
      __STORE(__h - 10, __reg_5_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
      __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
      __CALC5(__reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_5_3, __reg_4_0);
      __STORE(__h - 10, __reg_5_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
      __CALC5(__reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_5_4, __reg_4_1);
      __STORE(__h - 10, __reg_5_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
      __CALC5(__reg_5_4, __reg_5_3, __reg_5_2, __reg_5_1, __reg_5_0, __reg_4_2);
      __STORE(__h - 10, __reg_5_0);
      __h++;
    }
}
__global__ void kernel0_4(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 2 - 2);
    const AN5D_TYPE __c1Pad = (2);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 2 - 2);
    const AN5D_TYPE __c2Pad = (2);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 2;
    const AN5D_TYPE __halo2 = 2;
    const AN5D_TYPE __side0Len = 4;
    const AN5D_TYPE __side1Len = 256;
    const AN5D_TYPE __side2Len = 240;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_2_0;
    float __reg_2_1;
    float __reg_2_2;
    float __reg_2_3;
    float __reg_2_4;
    float __reg_3_0;
    float __reg_3_1;
    float __reg_3_2;
    float __reg_3_3;
    float __reg_3_4;
    float __reg_4_0;
    float __reg_4_1;
    float __reg_4_2;
    float __reg_4_3;
    float __reg_4_4;
    __shared__ float __a_sb_double[__blockSize * 2];
    float *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __writeValid3 = __updateValid && __local_c2 >= (__halo2 * 3) && __local_c2 < __side2LenOl - (__halo2 * 3);
    const AN5D_TYPE __writeValid4 = __updateValid && __local_c2 >= (__halo2 * 4) && __local_c2 < __side2LenOl - (__halo2 * 4);
    const AN5D_TYPE __storeValid = __writeValid4;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((0.03125f * (__SBREF(__a_sb, -2))) + (0.03126f * (__SBREF(__a_sb, -1)))) + (0.03127f * (__REGREF(__a, 0)))) + (0.03128f * (__SBREF(__a_sb, 1)))) + (0.03129f * (__SBREF(__a_sb, 2)))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((0.03130f * (__SBREF(__a_sb, -2)))) + (0.03131f * (__SBREF(__a_sb, -1)))) + (0.03132f * (__REGREF(__a, 0)))) + (0.03133f * (__SBREF(__a_sb, 1)))) + (0.03134f * (__SBREF(__a_sb, 2))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { float etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((0.03135f * (__SBREF(__a_sb, -2)))) + (0.03136f * (__SBREF(__a_sb, -1)))) + (0.24712f * (__REGREF(__a, 0)))) + (0.03138f * (__SBREF(__a_sb, 1)))) + (0.03139f * (__SBREF(__a_sb, 2)))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { float etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_3_wrap(__rn0, __a) do { __rn0 = (((((((((((0.03140f * (__SBREF(__a_sb, -2)))) + (0.03141f * (__SBREF(__a_sb, -1)))) + (0.03142f * (__REGREF(__a, 0)))) + (0.03143f * (__SBREF(__a_sb, 1)))) + (0.03144f * (__SBREF(__a_sb, 2))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_3(out, a) do { float etmp; __CALCEXPR_3_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_4_wrap(__rn0, __a) do { __rn0 = ((((((0.03145f * (__SBREF(__a_sb, -2)))) + (0.03146f * (__SBREF(__a_sb, -1)))) + (0.03147f * (__REGREF(__a, 0)))) + (0.03148f * (__SBREF(__a_sb, 1)))) + (0.03149f * (__SBREF(__a_sb, 2)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_4(out, a) do { float etmp; __CALCEXPR_4_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, out3, out4, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); __CALCEXPR_3(out3, reg); __CALCEXPR_4(out4, reg); } while (0);
    #define __CALC1(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __CALC2(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid2) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __CALC3(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid3) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __CALC4(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid4) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_0);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_0);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_0);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_0);
      __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
      __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
      __LOAD(__reg_0, 9);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
      __LOAD(__reg_0, 10);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
      __STORE(2, __reg_4_2);
      __LOAD(__reg_0, 11);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
      __STORE(3, __reg_4_3);
      __LOAD(__reg_0, 12);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
      __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
      __STORE(4, __reg_4_4);
      __LOAD(__reg_0, 13);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
      __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
      __STORE(5, __reg_4_0);
      __LOAD(__reg_0, 14);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
      __STORE(6, __reg_4_1);
      __LOAD(__reg_0, 15);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
      __STORE(7, __reg_4_2);
      __LOAD(__reg_0, 16);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
      __STORE(8, __reg_4_3);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
      __LOAD(__reg_0, 9);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
      __LOAD(__reg_0, 10);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
      __LOAD(__reg_0, 11);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __LOAD(__reg_0, 12);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
      __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
      __LOAD(__reg_0, 13);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
      __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
      __LOAD(__reg_0, 14);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
      __LOAD(__reg_0, 15);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
      __LOAD(__reg_0, 16);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
      __STORE(8, __reg_4_3);
    }
    __a_sb = __a_sb_double + __blockSize * 0;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 17; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 7;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __STORE(__h - 8, __reg_4_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __STORE(__h - 8, __reg_4_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __STORE(__h - 8, __reg_4_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __STORE(__h - 8, __reg_4_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __STORE(__h - 8, __reg_4_3);
        __h++;
      }
      if (0) {}
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __STORE(__h - 8, __reg_4_4);
        __reg_1_2 = __reg_0;
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __STORE(__h - 7, __reg_4_0);
        __reg_1_3 = __reg_0;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __STORE(__h - 6, __reg_4_1);
        __reg_2_2 = __reg_1_2;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __STORE(__h - 5, __reg_4_2);
        __reg_2_3 = __reg_1_3;
        __CALC3(__reg_3_4, __reg_3_4, __reg_3_4, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __STORE(__h - 4, __reg_4_3);
        __reg_3_2 = __reg_2_2;
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __STORE(__h - 3, __reg_4_4);
        __reg_3_3 = __reg_2_3;
        __CALC4(__reg_4_4, __reg_4_4, __reg_4_4, __reg_4_1, __reg_4_0, __reg_3_2);
        __STORE(__h - 2, __reg_4_0);
        __CALC4(__reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_1, __reg_3_3);
        __STORE(__h - 1, __reg_4_1);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __STORE(__h - 8, __reg_4_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __STORE(__h - 7, __reg_4_0);
        __reg_1_3 = __reg_0;
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __STORE(__h - 6, __reg_4_1);
        __reg_1_4 = __reg_0;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __STORE(__h - 5, __reg_4_2);
        __reg_2_3 = __reg_1_3;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __STORE(__h - 4, __reg_4_3);
        __reg_2_4 = __reg_1_4;
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_0, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __STORE(__h - 3, __reg_4_4);
        __reg_3_3 = __reg_2_3;
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __STORE(__h - 2, __reg_4_0);
        __reg_3_4 = __reg_2_4;
        __CALC4(__reg_4_0, __reg_4_0, __reg_4_0, __reg_4_2, __reg_4_1, __reg_3_3);
        __STORE(__h - 1, __reg_4_1);
        __CALC4(__reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_2, __reg_3_4);
        __STORE(__h + 0, __reg_4_2);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __STORE(__h - 8, __reg_4_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __STORE(__h - 7, __reg_4_0);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __STORE(__h - 6, __reg_4_1);
        __reg_1_4 = __reg_0;
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __STORE(__h - 5, __reg_4_2);
        __reg_1_0 = __reg_0;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __STORE(__h - 4, __reg_4_3);
        __reg_2_4 = __reg_1_4;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __STORE(__h - 3, __reg_4_4);
        __reg_2_0 = __reg_1_0;
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_1, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __STORE(__h - 2, __reg_4_0);
        __reg_3_4 = __reg_2_4;
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __STORE(__h - 1, __reg_4_1);
        __reg_3_0 = __reg_2_0;
        __CALC4(__reg_4_1, __reg_4_1, __reg_4_1, __reg_4_3, __reg_4_2, __reg_3_4);
        __STORE(__h + 0, __reg_4_2);
        __CALC4(__reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_3, __reg_3_0);
        __STORE(__h + 1, __reg_4_3);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __STORE(__h - 8, __reg_4_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __STORE(__h - 7, __reg_4_0);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __STORE(__h - 6, __reg_4_1);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __STORE(__h - 5, __reg_4_2);
        __reg_1_0 = __reg_0;
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __STORE(__h - 4, __reg_4_3);
        __reg_1_1 = __reg_0;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __STORE(__h - 3, __reg_4_4);
        __reg_2_0 = __reg_1_0;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __STORE(__h - 2, __reg_4_0);
        __reg_2_1 = __reg_1_1;
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_2, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __STORE(__h - 1, __reg_4_1);
        __reg_3_0 = __reg_2_0;
        __CALC3(__reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __STORE(__h + 0, __reg_4_2);
        __reg_3_1 = __reg_2_1;
        __CALC4(__reg_4_2, __reg_4_2, __reg_4_2, __reg_4_4, __reg_4_3, __reg_3_0);
        __STORE(__h + 1, __reg_4_3);
        __CALC4(__reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_4, __reg_3_1);
        __STORE(__h + 2, __reg_4_4);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __STORE(__h - 8, __reg_4_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __STORE(__h - 7, __reg_4_0);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __STORE(__h - 6, __reg_4_1);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __STORE(__h - 5, __reg_4_2);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __STORE(__h - 4, __reg_4_3);
        __reg_1_1 = __reg_0;
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __STORE(__h - 3, __reg_4_4);
        __reg_1_2 = __reg_0;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __STORE(__h - 2, __reg_4_0);
        __reg_2_1 = __reg_1_1;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __STORE(__h - 1, __reg_4_1);
        __reg_2_2 = __reg_1_2;
        __CALC3(__reg_3_3, __reg_3_3, __reg_3_3, __reg_3_0, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __STORE(__h + 0, __reg_4_2);
        __reg_3_1 = __reg_2_1;
        __CALC3(__reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __STORE(__h + 1, __reg_4_3);
        __reg_3_2 = __reg_2_2;
        __CALC4(__reg_4_3, __reg_4_3, __reg_4_3, __reg_4_0, __reg_4_4, __reg_3_1);
        __STORE(__h + 2, __reg_4_4);
        __CALC4(__reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_0, __reg_3_2);
        __STORE(__h + 3, __reg_4_0);
      }
    }
    else
    {
      for (__h = 17; __h <= __side1LenOl - 5;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
        __STORE(__h - 8, __reg_4_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
        __STORE(__h - 8, __reg_4_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
        __STORE(__h - 8, __reg_4_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
        __STORE(__h - 8, __reg_4_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
        __STORE(__h - 8, __reg_4_3);
        __h++;
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
      __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_3_1);
      __STORE(__h - 8, __reg_4_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
      __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_2);
      __STORE(__h - 8, __reg_4_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_3);
      __STORE(__h - 8, __reg_4_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_4);
      __STORE(__h - 8, __reg_4_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_4, __reg_4_3, __reg_3_0);
      __STORE(__h - 8, __reg_4_3);
      __h++;
    }
}
__global__ void kernel0_3(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 2 - 2);
    const AN5D_TYPE __c1Pad = (2);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 2 - 2);
    const AN5D_TYPE __c2Pad = (2);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 2;
    const AN5D_TYPE __halo2 = 2;
    const AN5D_TYPE __side0Len = 3;
    const AN5D_TYPE __side1Len = 256;
    const AN5D_TYPE __side2Len = 244;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_2_0;
    float __reg_2_1;
    float __reg_2_2;
    float __reg_2_3;
    float __reg_2_4;
    float __reg_3_0;
    float __reg_3_1;
    float __reg_3_2;
    float __reg_3_3;
    float __reg_3_4;
    __shared__ float __a_sb_double[__blockSize * 2];
    float *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __writeValid3 = __updateValid && __local_c2 >= (__halo2 * 3) && __local_c2 < __side2LenOl - (__halo2 * 3);
    const AN5D_TYPE __storeValid = __writeValid3;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((0.03125f * (__SBREF(__a_sb, -2))) + (0.03126f * (__SBREF(__a_sb, -1)))) + (0.03127f * (__REGREF(__a, 0)))) + (0.03128f * (__SBREF(__a_sb, 1)))) + (0.03129f * (__SBREF(__a_sb, 2)))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((0.03130f * (__SBREF(__a_sb, -2)))) + (0.03131f * (__SBREF(__a_sb, -1)))) + (0.03132f * (__REGREF(__a, 0)))) + (0.03133f * (__SBREF(__a_sb, 1)))) + (0.03134f * (__SBREF(__a_sb, 2))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { float etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((0.03135f * (__SBREF(__a_sb, -2)))) + (0.03136f * (__SBREF(__a_sb, -1)))) + (0.24712f * (__REGREF(__a, 0)))) + (0.03138f * (__SBREF(__a_sb, 1)))) + (0.03139f * (__SBREF(__a_sb, 2)))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { float etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_3_wrap(__rn0, __a) do { __rn0 = (((((((((((0.03140f * (__SBREF(__a_sb, -2)))) + (0.03141f * (__SBREF(__a_sb, -1)))) + (0.03142f * (__REGREF(__a, 0)))) + (0.03143f * (__SBREF(__a_sb, 1)))) + (0.03144f * (__SBREF(__a_sb, 2))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_3(out, a) do { float etmp; __CALCEXPR_3_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_4_wrap(__rn0, __a) do { __rn0 = ((((((0.03145f * (__SBREF(__a_sb, -2)))) + (0.03146f * (__SBREF(__a_sb, -1)))) + (0.03147f * (__REGREF(__a, 0)))) + (0.03148f * (__SBREF(__a_sb, 1)))) + (0.03149f * (__SBREF(__a_sb, 2)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_4(out, a) do { float etmp; __CALCEXPR_4_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, out3, out4, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); __CALCEXPR_3(out3, reg); __CALCEXPR_4(out4, reg); } while (0);
    #define __CALC1(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __CALC2(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid2) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __CALC3(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid3) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_0);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_0);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
      __STORE(2, __reg_3_2);
      __LOAD(__reg_0, 9);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
      __STORE(3, __reg_3_3);
      __LOAD(__reg_0, 10);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
      __STORE(4, __reg_3_4);
      __LOAD(__reg_0, 11);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __STORE(5, __reg_3_0);
      __LOAD(__reg_0, 12);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
      __STORE(6, __reg_3_1);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
      __LOAD(__reg_0, 9);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
      __LOAD(__reg_0, 10);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
      __LOAD(__reg_0, 11);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __LOAD(__reg_0, 12);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
      __STORE(6, __reg_3_1);
    }
    __a_sb = __a_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 13; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 7;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __STORE(__h - 6, __reg_3_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __STORE(__h - 6, __reg_3_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __STORE(__h - 6, __reg_3_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __STORE(__h - 6, __reg_3_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __STORE(__h - 6, __reg_3_1);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __STORE(__h - 6, __reg_3_2);
        __reg_1_3 = __reg_0;
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __STORE(__h - 5, __reg_3_3);
        __reg_1_4 = __reg_0;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __STORE(__h - 4, __reg_3_4);
        __reg_2_3 = __reg_1_3;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __STORE(__h - 3, __reg_3_0);
        __reg_2_4 = __reg_1_4;
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_0, __reg_3_2, __reg_3_1, __reg_2_3);
        __STORE(__h - 2, __reg_3_1);
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_2, __reg_2_4);
        __STORE(__h - 1, __reg_3_2);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __STORE(__h - 6, __reg_3_2);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __STORE(__h - 5, __reg_3_3);
        __reg_1_4 = __reg_0;
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __STORE(__h - 4, __reg_3_4);
        __reg_1_0 = __reg_0;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __STORE(__h - 3, __reg_3_0);
        __reg_2_4 = __reg_1_4;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __STORE(__h - 2, __reg_3_1);
        __reg_2_0 = __reg_1_0;
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_1, __reg_3_3, __reg_3_2, __reg_2_4);
        __STORE(__h - 1, __reg_3_2);
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_3, __reg_2_0);
        __STORE(__h + 0, __reg_3_3);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __STORE(__h - 6, __reg_3_2);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __STORE(__h - 5, __reg_3_3);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __STORE(__h - 4, __reg_3_4);
        __reg_1_0 = __reg_0;
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __STORE(__h - 3, __reg_3_0);
        __reg_1_1 = __reg_0;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __STORE(__h - 2, __reg_3_1);
        __reg_2_0 = __reg_1_0;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __STORE(__h - 1, __reg_3_2);
        __reg_2_1 = __reg_1_1;
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_2, __reg_3_4, __reg_3_3, __reg_2_0);
        __STORE(__h + 0, __reg_3_3);
        __CALC3(__reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_4, __reg_2_1);
        __STORE(__h + 1, __reg_3_4);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __STORE(__h - 6, __reg_3_2);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __STORE(__h - 5, __reg_3_3);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __STORE(__h - 4, __reg_3_4);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __STORE(__h - 3, __reg_3_0);
        __reg_1_1 = __reg_0;
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __STORE(__h - 2, __reg_3_1);
        __reg_1_2 = __reg_0;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __STORE(__h - 1, __reg_3_2);
        __reg_2_1 = __reg_1_1;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __STORE(__h + 0, __reg_3_3);
        __reg_2_2 = __reg_1_2;
        __CALC3(__reg_3_3, __reg_3_3, __reg_3_3, __reg_3_0, __reg_3_4, __reg_2_1);
        __STORE(__h + 1, __reg_3_4);
        __CALC3(__reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_0, __reg_2_2);
        __STORE(__h + 2, __reg_3_0);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __STORE(__h - 6, __reg_3_2);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __STORE(__h - 5, __reg_3_3);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __STORE(__h - 4, __reg_3_4);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __STORE(__h - 3, __reg_3_0);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __STORE(__h - 2, __reg_3_1);
        __reg_1_2 = __reg_0;
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __STORE(__h - 1, __reg_3_2);
        __reg_1_3 = __reg_0;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __STORE(__h + 0, __reg_3_3);
        __reg_2_2 = __reg_1_2;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __STORE(__h + 1, __reg_3_4);
        __reg_2_3 = __reg_1_3;
        __CALC3(__reg_3_4, __reg_3_4, __reg_3_4, __reg_3_1, __reg_3_0, __reg_2_2);
        __STORE(__h + 2, __reg_3_0);
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_1, __reg_2_3);
        __STORE(__h + 3, __reg_3_1);
      }
    }
    else
    {
      for (__h = 13; __h <= __side1LenOl - 5;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
        __STORE(__h - 6, __reg_3_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
        __STORE(__h - 6, __reg_3_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
        __STORE(__h - 6, __reg_3_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
        __STORE(__h - 6, __reg_3_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
        __STORE(__h - 6, __reg_3_1);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_4);
      __STORE(__h - 6, __reg_3_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_3_3, __reg_2_0);
      __STORE(__h - 6, __reg_3_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_4, __reg_2_1);
      __STORE(__h - 6, __reg_3_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_2);
      __STORE(__h - 6, __reg_3_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_3);
      __STORE(__h - 6, __reg_3_1);
      __h++;
    }
}
__global__ void kernel0_2(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 2 - 2);
    const AN5D_TYPE __c1Pad = (2);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 2 - 2);
    const AN5D_TYPE __c2Pad = (2);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 2;
    const AN5D_TYPE __halo2 = 2;
    const AN5D_TYPE __side0Len = 2;
    const AN5D_TYPE __side1Len = 256;
    const AN5D_TYPE __side2Len = 248;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_2_0;
    float __reg_2_1;
    float __reg_2_2;
    float __reg_2_3;
    float __reg_2_4;
    __shared__ float __a_sb_double[__blockSize * 2];
    float *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __storeValid = __writeValid2;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((0.03125f * (__SBREF(__a_sb, -2))) + (0.03126f * (__SBREF(__a_sb, -1)))) + (0.03127f * (__REGREF(__a, 0)))) + (0.03128f * (__SBREF(__a_sb, 1)))) + (0.03129f * (__SBREF(__a_sb, 2)))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((0.03130f * (__SBREF(__a_sb, -2)))) + (0.03131f * (__SBREF(__a_sb, -1)))) + (0.03132f * (__REGREF(__a, 0)))) + (0.03133f * (__SBREF(__a_sb, 1)))) + (0.03134f * (__SBREF(__a_sb, 2))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { float etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((0.03135f * (__SBREF(__a_sb, -2)))) + (0.03136f * (__SBREF(__a_sb, -1)))) + (0.24712f * (__REGREF(__a, 0)))) + (0.03138f * (__SBREF(__a_sb, 1)))) + (0.03139f * (__SBREF(__a_sb, 2)))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { float etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_3_wrap(__rn0, __a) do { __rn0 = (((((((((((0.03140f * (__SBREF(__a_sb, -2)))) + (0.03141f * (__SBREF(__a_sb, -1)))) + (0.03142f * (__REGREF(__a, 0)))) + (0.03143f * (__SBREF(__a_sb, 1)))) + (0.03144f * (__SBREF(__a_sb, 2))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_3(out, a) do { float etmp; __CALCEXPR_3_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_4_wrap(__rn0, __a) do { __rn0 = ((((((0.03145f * (__SBREF(__a_sb, -2)))) + (0.03146f * (__SBREF(__a_sb, -1)))) + (0.03147f * (__REGREF(__a, 0)))) + (0.03148f * (__SBREF(__a_sb, 1)))) + (0.03149f * (__SBREF(__a_sb, 2)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_4(out, a) do { float etmp; __CALCEXPR_4_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, out3, out4, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); __CALCEXPR_3(out3, reg); __CALCEXPR_4(out4, reg); } while (0);
    #define __CALC1(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __CALC2(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid2) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __STORE(2, __reg_2_2);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __STORE(3, __reg_2_3);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __STORE(4, __reg_2_4);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __STORE(4, __reg_2_4);
    }
    __a_sb = __a_sb_double + __blockSize * 0;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 9; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 7;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __STORE(__h - 4, __reg_2_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __STORE(__h - 4, __reg_2_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __STORE(__h - 4, __reg_2_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __STORE(__h - 4, __reg_2_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __STORE(__h - 4, __reg_2_4);
        __h++;
      }
      if (0) {}
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __STORE(__h - 4, __reg_2_0);
        __reg_1_4 = __reg_0;
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __STORE(__h - 3, __reg_2_1);
        __reg_1_0 = __reg_0;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_3, __reg_2_2, __reg_1_4);
        __STORE(__h - 2, __reg_2_2);
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_3, __reg_1_0);
        __STORE(__h - 1, __reg_2_3);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __STORE(__h - 4, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __STORE(__h - 3, __reg_2_1);
        __reg_1_0 = __reg_0;
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __STORE(__h - 2, __reg_2_2);
        __reg_1_1 = __reg_0;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_4, __reg_2_3, __reg_1_0);
        __STORE(__h - 1, __reg_2_3);
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_4, __reg_1_1);
        __STORE(__h + 0, __reg_2_4);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __STORE(__h - 4, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __STORE(__h - 3, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __STORE(__h - 2, __reg_2_2);
        __reg_1_1 = __reg_0;
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __STORE(__h - 1, __reg_2_3);
        __reg_1_2 = __reg_0;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_0, __reg_2_4, __reg_1_1);
        __STORE(__h + 0, __reg_2_4);
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_0, __reg_1_2);
        __STORE(__h + 1, __reg_2_0);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __STORE(__h - 4, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __STORE(__h - 3, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __STORE(__h - 2, __reg_2_2);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __STORE(__h - 1, __reg_2_3);
        __reg_1_2 = __reg_0;
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __STORE(__h + 0, __reg_2_4);
        __reg_1_3 = __reg_0;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_1, __reg_2_0, __reg_1_2);
        __STORE(__h + 1, __reg_2_0);
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_1, __reg_1_3);
        __STORE(__h + 2, __reg_2_1);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __STORE(__h - 4, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __STORE(__h - 3, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __STORE(__h - 2, __reg_2_2);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __STORE(__h - 1, __reg_2_3);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __STORE(__h + 0, __reg_2_4);
        __reg_1_3 = __reg_0;
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __STORE(__h + 1, __reg_2_0);
        __reg_1_4 = __reg_0;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_2, __reg_2_1, __reg_1_3);
        __STORE(__h + 2, __reg_2_1);
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_2, __reg_1_4);
        __STORE(__h + 3, __reg_2_2);
      }
    }
    else
    {
      for (__h = 9; __h <= __side1LenOl - 5;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
        __STORE(__h - 4, __reg_2_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
        __STORE(__h - 4, __reg_2_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
        __STORE(__h - 4, __reg_2_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
        __STORE(__h - 4, __reg_2_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
        __STORE(__h - 4, __reg_2_4);
        __h++;
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_2);
      __STORE(__h - 4, __reg_2_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_3);
      __STORE(__h - 4, __reg_2_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_4);
      __STORE(__h - 4, __reg_2_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_2_3, __reg_1_0);
      __STORE(__h - 4, __reg_2_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_4, __reg_1_1);
      __STORE(__h - 4, __reg_2_4);
      __h++;
    }
}
__global__ void kernel0_1(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 2 - 2);
    const AN5D_TYPE __c1Pad = (2);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 2 - 2);
    const AN5D_TYPE __c2Pad = (2);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 2;
    const AN5D_TYPE __halo2 = 2;
    const AN5D_TYPE __side0Len = 1;
    const AN5D_TYPE __side1Len = 256;
    const AN5D_TYPE __side2Len = 252;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    __shared__ float __a_sb_double[__blockSize * 2];
    float *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __storeValid = __writeValid1;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((0.03125f * (__SBREF(__a_sb, -2))) + (0.03126f * (__SBREF(__a_sb, -1)))) + (0.03127f * (__REGREF(__a, 0)))) + (0.03128f * (__SBREF(__a_sb, 1)))) + (0.03129f * (__SBREF(__a_sb, 2)))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((0.03130f * (__SBREF(__a_sb, -2)))) + (0.03131f * (__SBREF(__a_sb, -1)))) + (0.03132f * (__REGREF(__a, 0)))) + (0.03133f * (__SBREF(__a_sb, 1)))) + (0.03134f * (__SBREF(__a_sb, 2))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { float etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((0.03135f * (__SBREF(__a_sb, -2)))) + (0.03136f * (__SBREF(__a_sb, -1)))) + (0.24712f * (__REGREF(__a, 0)))) + (0.03138f * (__SBREF(__a_sb, 1)))) + (0.03139f * (__SBREF(__a_sb, 2)))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { float etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_3_wrap(__rn0, __a) do { __rn0 = (((((((((((0.03140f * (__SBREF(__a_sb, -2)))) + (0.03141f * (__SBREF(__a_sb, -1)))) + (0.03142f * (__REGREF(__a, 0)))) + (0.03143f * (__SBREF(__a_sb, 1)))) + (0.03144f * (__SBREF(__a_sb, 2))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_3(out, a) do { float etmp; __CALCEXPR_3_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_4_wrap(__rn0, __a) do { __rn0 = ((((((0.03145f * (__SBREF(__a_sb, -2)))) + (0.03146f * (__SBREF(__a_sb, -1)))) + (0.03147f * (__REGREF(__a, 0)))) + (0.03148f * (__SBREF(__a_sb, 1)))) + (0.03149f * (__SBREF(__a_sb, 2)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_4(out, a) do { float etmp; __CALCEXPR_4_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, out3, out4, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); __CALCEXPR_3(out3, reg); __CALCEXPR_4(out4, reg); } while (0);
    #define __CALC1(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __STORE(2, __reg_1_2);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __STORE(2, __reg_1_2);
    }
    __a_sb = __a_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 5; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 7;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 2, __reg_1_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __STORE(__h - 2, __reg_1_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 2, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 2, __reg_1_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h - 2, __reg_1_2);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 2, __reg_1_3);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_4, __reg_0);
        __STORE(__h - 1, __reg_1_4);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 2, __reg_1_3);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_0, __reg_1_4, __reg_0);
        __STORE(__h - 1, __reg_1_4);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 2, __reg_1_3);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __STORE(__h - 1, __reg_1_4);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 2, __reg_1_3);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __STORE(__h - 1, __reg_1_4);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 2, __reg_1_3);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __STORE(__h - 1, __reg_1_4);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
      }
    }
    else
    {
      for (__h = 5; __h <= __side1LenOl - 5;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 2, __reg_1_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __STORE(__h - 2, __reg_1_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 2, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 2, __reg_1_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h - 2, __reg_1_2);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __STORE(__h - 2, __reg_1_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __STORE(__h - 2, __reg_1_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __STORE(__h - 2, __reg_1_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __STORE(__h - 2, __reg_1_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __STORE(__h - 2, __reg_1_2);
      __h++;
    }
}
