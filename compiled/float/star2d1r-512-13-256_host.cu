#include <assert.h>
#include <stdio.h>
#include "star2d1r-512-13-256_kernel.hu"
#define BENCH_DIM 2
#define BENCH_FPP 9
#define BENCH_RAD 1

#include "common.h"

double kernel_stencil(SB_TYPE *A1, int compsize, int timestep, bool scop)
{
  double start_time = sb_time(), end_time = 0.0;
  int dimsize = compsize + BENCH_RAD * 2;
  SB_TYPE (*A)[dimsize][dimsize] = (SB_TYPE (*)[dimsize][dimsize])A1;

  if (scop) {
    if (dimsize >= 3 && timestep >= 1) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

      float *dev_A;
      
      cudaCheckReturn(hipMalloc((void **) &dev_A, (size_t)(2) * (size_t)(dimsize) * (size_t)(dimsize) * sizeof(float)));
      
{
      cudaCheckReturn(hipMemcpy(dev_A, A, (size_t)(2) * (size_t)(dimsize) * (size_t)(dimsize) * sizeof(float), hipMemcpyHostToDevice));
#ifdef STENCILBENCH
hipDeviceSynchronize();
SB_START_INSTRUMENTS;
#endif
}
    {
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
      const AN5D_TYPE __c0Len = (timestep - 0);
      const AN5D_TYPE __c0Pad = (0);
      #define __c0 c0
      const AN5D_TYPE __c1Len = (dimsize - 1 - 1);
      const AN5D_TYPE __c1Pad = (1);
      #define __c1 c1
      const AN5D_TYPE __c2Len = (dimsize - 1 - 1);
      const AN5D_TYPE __c2Pad = (1);
      #define __c2 c2
      const AN5D_TYPE __halo1 = 1;
      const AN5D_TYPE __halo2 = 1;
      AN5D_TYPE c0;
      AN5D_TYPE __side0LenMax;
      {
        const AN5D_TYPE __side0Len = 13;
        const AN5D_TYPE __side1Len = 256;
        const AN5D_TYPE __side2Len = 486;
        const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
        const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
        const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
        const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
        const AN5D_TYPE __blockSize = 1 * __side2LenOl;
        assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
        dim3 k0_dimBlock(__blockSize, 1, 1);
        dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
        AN5D_TYPE __c0Padr = (__c0Len % 2) != (((__c0Len + __side0Len - 1) / __side0Len) % 2) && __c0Len % __side0Len < 2 ? 1 : 0;
        __side0LenMax = __side0Len;
        for (c0 = __c0Pad; c0 < __c0Pad + __c0Len / __side0Len - __c0Padr; c0 += 1)
        {
          kernel0_13<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
      }
      if ((__c0Len % 2) != (((__c0Len + __side0LenMax - 1) / __side0LenMax) % 2))
      {
        if (__c0Len % __side0LenMax == 0)
        {
          {
            const AN5D_TYPE __side0Len = 7;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 498;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_7<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 6;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 500;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_6<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 1)
        {
          {
            const AN5D_TYPE __side0Len = 6;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 500;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_6<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 4;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 504;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_4<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 4;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 504;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_4<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 2)
        {
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 510;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 510;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 3)
        {
          {
            const AN5D_TYPE __side0Len = 2;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 508;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 510;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 4)
        {
          {
            const AN5D_TYPE __side0Len = 2;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 508;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 2;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 508;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 5)
        {
          {
            const AN5D_TYPE __side0Len = 3;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 506;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 2;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 508;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 6)
        {
          {
            const AN5D_TYPE __side0Len = 3;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 506;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 3;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 506;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 7)
        {
          {
            const AN5D_TYPE __side0Len = 4;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 504;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_4<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 3;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 506;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 8)
        {
          {
            const AN5D_TYPE __side0Len = 4;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 504;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_4<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 4;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 504;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_4<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 9)
        {
          {
            const AN5D_TYPE __side0Len = 5;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 502;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_5<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 4;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 504;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_4<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 10)
        {
          {
            const AN5D_TYPE __side0Len = 5;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 502;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_5<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 5;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 502;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_5<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 11)
        {
          {
            const AN5D_TYPE __side0Len = 6;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 500;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_6<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 5;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 502;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_5<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 12)
        {
          {
            const AN5D_TYPE __side0Len = 6;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 500;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_6<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 6;
            const AN5D_TYPE __side1Len = 256;
            const AN5D_TYPE __side2Len = 500;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_6<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
      }
      else if (__c0Len % __side0LenMax)
      {
        if (__c0Len % __side0LenMax == 1)
        {
          const AN5D_TYPE __side0Len = 1;
          const AN5D_TYPE __side1Len = 256;
          const AN5D_TYPE __side2Len = 510;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 2)
        {
          const AN5D_TYPE __side0Len = 2;
          const AN5D_TYPE __side1Len = 256;
          const AN5D_TYPE __side2Len = 508;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 3)
        {
          const AN5D_TYPE __side0Len = 3;
          const AN5D_TYPE __side1Len = 256;
          const AN5D_TYPE __side2Len = 506;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 4)
        {
          const AN5D_TYPE __side0Len = 4;
          const AN5D_TYPE __side1Len = 256;
          const AN5D_TYPE __side2Len = 504;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_4<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 5)
        {
          const AN5D_TYPE __side0Len = 5;
          const AN5D_TYPE __side1Len = 256;
          const AN5D_TYPE __side2Len = 502;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_5<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 6)
        {
          const AN5D_TYPE __side0Len = 6;
          const AN5D_TYPE __side1Len = 256;
          const AN5D_TYPE __side2Len = 500;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_6<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 7)
        {
          const AN5D_TYPE __side0Len = 7;
          const AN5D_TYPE __side1Len = 256;
          const AN5D_TYPE __side2Len = 498;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_7<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 8)
        {
          const AN5D_TYPE __side0Len = 8;
          const AN5D_TYPE __side1Len = 256;
          const AN5D_TYPE __side2Len = 496;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_8<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 9)
        {
          const AN5D_TYPE __side0Len = 9;
          const AN5D_TYPE __side1Len = 256;
          const AN5D_TYPE __side2Len = 494;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_9<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 10)
        {
          const AN5D_TYPE __side0Len = 10;
          const AN5D_TYPE __side1Len = 256;
          const AN5D_TYPE __side2Len = 492;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_10<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 11)
        {
          const AN5D_TYPE __side0Len = 11;
          const AN5D_TYPE __side1Len = 256;
          const AN5D_TYPE __side2Len = 490;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_11<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 12)
        {
          const AN5D_TYPE __side0Len = 12;
          const AN5D_TYPE __side1Len = 256;
          const AN5D_TYPE __side2Len = 488;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_12<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
      }
    }
    cudaCheckKernel();
{
#ifdef STENCILBENCH
hipDeviceSynchronize();
SB_STOP_INSTRUMENTS;
#endif
      cudaCheckReturn(hipMemcpy(A, dev_A, (size_t)(2) * (size_t)(dimsize) * (size_t)(dimsize) * sizeof(float), hipMemcpyDeviceToHost));
}
      cudaCheckReturn(hipFree(dev_A));
    }
  }
  else {
    for (int t = 0; t < timestep; t++)
#pragma omp parallel for
      for (int i = BENCH_RAD; i < dimsize - BENCH_RAD; i++)
        for (int j = BENCH_RAD; j < dimsize - BENCH_RAD; j++)
          A[(t+1)%2][i][j] =
            0.1873f * A[t%2][i-1][j]
            + 0.1876f * A[t%2][i][j-1]
            + 0.2500f * A[t%2][i][j]
            + 0.1877f * A[t%2][i][j+1]
            + 0.1874f * A[t%2][i+1][j];
  }

  return (((end_time != 0.0) ? end_time : sb_time()) - start_time);
}
