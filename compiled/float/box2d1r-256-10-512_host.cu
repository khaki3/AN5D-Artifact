#include <assert.h>
#include <stdio.h>
#include "box2d1r-256-10-512_kernel.hu"
#define BENCH_DIM 2
#define BENCH_FPP 17
#define BENCH_RAD 1

#include "common.h"

double kernel_stencil(SB_TYPE *A1, int compsize, int timestep, bool scop)
{
  double start_time = sb_time(), end_time = 0.0;
  int dimsize = compsize + BENCH_RAD * 2;
  SB_TYPE (*A)[dimsize][dimsize] = (SB_TYPE (*)[dimsize][dimsize])A1;

  if (scop) {
    if (dimsize >= 3 && timestep >= 1) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

      float *dev_A;
      
      cudaCheckReturn(hipMalloc((void **) &dev_A, (size_t)(2) * (size_t)(dimsize) * (size_t)(dimsize) * sizeof(float)));
      
{
      cudaCheckReturn(hipMemcpy(dev_A, A, (size_t)(2) * (size_t)(dimsize) * (size_t)(dimsize) * sizeof(float), hipMemcpyHostToDevice));
#ifdef STENCILBENCH
hipDeviceSynchronize();
SB_START_INSTRUMENTS;
#endif
}
    {
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
      const AN5D_TYPE __c0Len = (timestep - 0);
      const AN5D_TYPE __c0Pad = (0);
      #define __c0 c0
      const AN5D_TYPE __c1Len = (dimsize - 1 - 1);
      const AN5D_TYPE __c1Pad = (1);
      #define __c1 c1
      const AN5D_TYPE __c2Len = (dimsize - 1 - 1);
      const AN5D_TYPE __c2Pad = (1);
      #define __c2 c2
      const AN5D_TYPE __halo1 = 1;
      const AN5D_TYPE __halo2 = 1;
      AN5D_TYPE c0;
      AN5D_TYPE __side0LenMax;
      {
        const AN5D_TYPE __side0Len = 10;
        const AN5D_TYPE __side1Len = 512;
        const AN5D_TYPE __side2Len = 236;
        const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
        const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
        const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
        const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
        const AN5D_TYPE __blockSize = 1 * __side2LenOl;
        assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
        dim3 k0_dimBlock(__blockSize, 1, 1);
        dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
        AN5D_TYPE __c0Padr = (__c0Len % 2) != (((__c0Len + __side0Len - 1) / __side0Len) % 2) && __c0Len % __side0Len < 2 ? 1 : 0;
        __side0LenMax = __side0Len;
        for (c0 = __c0Pad; c0 < __c0Pad + __c0Len / __side0Len - __c0Padr; c0 += 1)
        {
          kernel0_10<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
      }
      if ((__c0Len % 2) != (((__c0Len + __side0LenMax - 1) / __side0LenMax) % 2))
      {
        if (__c0Len % __side0LenMax == 0)
        {
          {
            const AN5D_TYPE __side0Len = 5;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 246;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_5<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 5;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 246;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_5<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 1)
        {
          {
            const AN5D_TYPE __side0Len = 5;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 246;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_5<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 3;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 250;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 3;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 250;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 2)
        {
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 254;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 254;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 3)
        {
          {
            const AN5D_TYPE __side0Len = 2;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 252;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 254;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 4)
        {
          {
            const AN5D_TYPE __side0Len = 2;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 252;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 2;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 252;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 5)
        {
          {
            const AN5D_TYPE __side0Len = 3;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 250;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 2;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 252;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 6)
        {
          {
            const AN5D_TYPE __side0Len = 3;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 250;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 3;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 250;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 7)
        {
          {
            const AN5D_TYPE __side0Len = 4;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 248;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_4<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 3;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 250;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 8)
        {
          {
            const AN5D_TYPE __side0Len = 4;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 248;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_4<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 4;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 248;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_4<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 9)
        {
          {
            const AN5D_TYPE __side0Len = 5;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 246;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_5<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 4;
            const AN5D_TYPE __side1Len = 512;
            const AN5D_TYPE __side2Len = 248;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_4<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
          }
        }
      }
      else if (__c0Len % __side0LenMax)
      {
        if (__c0Len % __side0LenMax == 1)
        {
          const AN5D_TYPE __side0Len = 1;
          const AN5D_TYPE __side1Len = 512;
          const AN5D_TYPE __side2Len = 254;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 2)
        {
          const AN5D_TYPE __side0Len = 2;
          const AN5D_TYPE __side1Len = 512;
          const AN5D_TYPE __side2Len = 252;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 3)
        {
          const AN5D_TYPE __side0Len = 3;
          const AN5D_TYPE __side1Len = 512;
          const AN5D_TYPE __side2Len = 250;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 4)
        {
          const AN5D_TYPE __side0Len = 4;
          const AN5D_TYPE __side1Len = 512;
          const AN5D_TYPE __side2Len = 248;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_4<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 5)
        {
          const AN5D_TYPE __side0Len = 5;
          const AN5D_TYPE __side1Len = 512;
          const AN5D_TYPE __side2Len = 246;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_5<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 6)
        {
          const AN5D_TYPE __side0Len = 6;
          const AN5D_TYPE __side1Len = 512;
          const AN5D_TYPE __side2Len = 244;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_6<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 7)
        {
          const AN5D_TYPE __side0Len = 7;
          const AN5D_TYPE __side1Len = 512;
          const AN5D_TYPE __side2Len = 242;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_7<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 8)
        {
          const AN5D_TYPE __side0Len = 8;
          const AN5D_TYPE __side1Len = 512;
          const AN5D_TYPE __side2Len = 240;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_8<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
        else if (__c0Len % __side0LenMax == 9)
        {
          const AN5D_TYPE __side0Len = 9;
          const AN5D_TYPE __side1Len = 512;
          const AN5D_TYPE __side2Len = 238;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_9<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
      }
    }
    cudaCheckKernel();
{
#ifdef STENCILBENCH
hipDeviceSynchronize();
SB_STOP_INSTRUMENTS;
#endif
      cudaCheckReturn(hipMemcpy(A, dev_A, (size_t)(2) * (size_t)(dimsize) * (size_t)(dimsize) * sizeof(float), hipMemcpyDeviceToHost));
}
      cudaCheckReturn(hipFree(dev_A));
    }
  }
  else {
    for (int t = 0; t < timestep; t++)
#pragma omp parallel for
      for (int i = BENCH_RAD; i < dimsize - BENCH_RAD; i++)
        for (int j = BENCH_RAD; j < dimsize - BENCH_RAD; j++)
          A[(t+1)%2][i][j] =
            0.09371f * A[t%2][i-1][j-1] + 0.09374f * A[t%2][i-1][j] + 0.09376f * A[t%2][i-1][j+1] +
            0.09372f * A[t%2][i][j-1]   + 0.25001f * A[t%2][i][j]   + 0.09377f * A[t%2][i][j+1] +
            0.09373f * A[t%2][i+1][j-1] + 0.09375f * A[t%2][i+1][j] + 0.09378f * A[t%2][i+1][j+1];
  }

  return (((end_time != 0.0) ? end_time : sb_time()) - start_time);
}
