#include "hip/hip_runtime.h"
#include "box3d2r-32x16-1-128_kernel.hu"
__device__ float __sbref_wrap(float *sb, size_t index) { return sb[index]; }

__global__ void kernel0_1(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 2 - 2);
    const AN5D_TYPE __c1Pad = (2);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 2 - 2);
    const AN5D_TYPE __c2Pad = (2);
    #define __c2 c2
    const AN5D_TYPE __c3Len = (dimsize - 2 - 2);
    const AN5D_TYPE __c3Pad = (2);
    #define __c3 c3
    const AN5D_TYPE __halo1 = 2;
    const AN5D_TYPE __halo2 = 2;
    const AN5D_TYPE __halo3 = 2;
    const AN5D_TYPE __side0Len = 1;
    const AN5D_TYPE __side1Len = 128;
    const AN5D_TYPE __side2Len = 12;
    const AN5D_TYPE __side3Len = 28;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __OlLen3 = (__halo3 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __side3LenOl = (__side3Len + 2 * __OlLen3);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl * __side3LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __side3Num = (__c3Len + __side3Len - 1) / __side3Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid / __side3LenOl;
    const AN5D_TYPE __local_c3 = __tid % __side3LenOl;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num / __side3Num;
    const AN5D_TYPE __c2 = (blockIdx.x / __side3Num % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    const AN5D_TYPE __c3 = (blockIdx.x % __side3Num) * __side3Len + __local_c3 + __c3Pad - __OlLen3;
    float __reg_0;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    __shared__ float __a_sb_double[__blockSize * 2];
    float *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2 && __c3 >= __c3Pad - __halo3 && __c3 < __c3Pad + __c3Len + __halo3;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len && __c3 >= __c3Pad && __c3 < __c3Pad + __c3Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1) && __local_c3 >= (__halo3 * 1) && __local_c3 < __side3LenOl - (__halo3 * 1);
    const AN5D_TYPE __storeValid = __writeValid1;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[(((__c0 % 2) * dimsize + __c1) * dimsize + __c2) * dimsize + __c3]; }} while (0)
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((-0.324f) * (__REGREF(__a, 0, 0))) + (0.0020f * (__SBREF(__a_sb, -2, -2)))) + (0.0030f * (__SBREF(__a_sb, -2, -1)))) + (0.0040f * (__SBREF(__a_sb, -2, 0)))) + (0.0050f * (__SBREF(__a_sb, -2, 1)))) + (0.0060f * (__SBREF(__a_sb, -2, 2)))) + (0.0070f * (__SBREF(__a_sb, -1, -2)))) + (0.0080f * (__SBREF(__a_sb, -1, -1)))) + (0.0090f * (__SBREF(__a_sb, -1, 0)))) + (0.0100f * (__SBREF(__a_sb, -1, 1)))) + (0.0110f * (__SBREF(__a_sb, -1, 2)))) + (0.0120f * (__SBREF(__a_sb, 0, -2)))) + (0.0130f * (__SBREF(__a_sb, 0, -1)))) + (0.0140f * (__SBREF(__a_sb, 0, 1)))) + (0.0150f * (__SBREF(__a_sb, 0, 2)))) + (0.0160f * (__SBREF(__a_sb, 1, -2)))) + (0.0170f * (__SBREF(__a_sb, 1, -1)))) + (0.0180f * (__SBREF(__a_sb, 1, 0)))) + (0.0190f * (__SBREF(__a_sb, 1, 1)))) + (0.0200f * (__SBREF(__a_sb, 1, 2)))) + (0.0210f * (__SBREF(__a_sb, 2, -2)))) + (0.0220f * (__SBREF(__a_sb, 2, -1)))) + (0.0230f * (__SBREF(__a_sb, 2, 0)))) + (0.0240f * (__SBREF(__a_sb, 2, 1)))) + (0.0250f * (__SBREF(__a_sb, 2, 2)))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((-(0.3264f * (__REGREF(__a, 0, 0)))) + (0.0021f * (__SBREF(__a_sb, -2, -2)))) + (0.0031f * (__SBREF(__a_sb, -2, -1)))) + (0.0041f * (__SBREF(__a_sb, -2, 0)))) + (0.0051f * (__SBREF(__a_sb, -2, 1)))) + (0.0061f * (__SBREF(__a_sb, -2, 2)))) + (0.0071f * (__SBREF(__a_sb, -1, -2)))) + (0.0081f * (__SBREF(__a_sb, -1, -1)))) + (0.0091f * (__SBREF(__a_sb, -1, 0)))) + (0.0101f * (__SBREF(__a_sb, -1, 1)))) + (0.0111f * (__SBREF(__a_sb, -1, 2)))) + (0.0121f * (__SBREF(__a_sb, 0, -2)))) + (0.0131f * (__SBREF(__a_sb, 0, -1)))) + (0.0141f * (__SBREF(__a_sb, 0, 1)))) + (0.0151f * (__SBREF(__a_sb, 0, 2)))) + (0.0161f * (__SBREF(__a_sb, 1, -2)))) + (0.0171f * (__SBREF(__a_sb, 1, -1)))) + (0.0181f * (__SBREF(__a_sb, 1, 0)))) + (0.0191f * (__SBREF(__a_sb, 1, 1)))) + (0.0201f * (__SBREF(__a_sb, 1, 2)))) + (0.0211f * (__SBREF(__a_sb, 2, -2)))) + (0.0221f * (__SBREF(__a_sb, 2, -1)))) + (0.0231f * (__SBREF(__a_sb, 2, 0)))) + (0.0241f * (__SBREF(__a_sb, 2, 1)))) + (0.0251f * (__SBREF(__a_sb, 2, 2))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { float etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.6712f * (__REGREF(__a, 0, 0)))) + (0.0022f * (__SBREF(__a_sb, -2, -2)))) + (0.0032f * (__SBREF(__a_sb, -2, -1)))) + (0.0042f * (__SBREF(__a_sb, -2, 0)))) + (0.0052f * (__SBREF(__a_sb, -2, 1)))) + (0.0062f * (__SBREF(__a_sb, -2, 2)))) + (0.0072f * (__SBREF(__a_sb, -1, -2)))) + (0.0082f * (__SBREF(__a_sb, -1, -1)))) + (0.0092f * (__SBREF(__a_sb, -1, 0)))) + (0.0102f * (__SBREF(__a_sb, -1, 1)))) + (0.0112f * (__SBREF(__a_sb, -1, 2)))) + (0.0122f * (__SBREF(__a_sb, 0, -2)))) + (0.0132f * (__SBREF(__a_sb, 0, -1)))) + (0.0142f * (__SBREF(__a_sb, 0, 1)))) + (0.0152f * (__SBREF(__a_sb, 0, 2)))) + (0.0162f * (__SBREF(__a_sb, 1, -2)))) + (0.0172f * (__SBREF(__a_sb, 1, -1)))) + (0.0182f * (__SBREF(__a_sb, 1, 0)))) + (0.0192f * (__SBREF(__a_sb, 1, 1)))) + (0.0202f * (__SBREF(__a_sb, 1, 2)))) + (0.0212f * (__SBREF(__a_sb, 2, -2)))) + (0.0222f * (__SBREF(__a_sb, 2, -1)))) + (0.0232f * (__SBREF(__a_sb, 2, 0)))) + (0.0242f * (__SBREF(__a_sb, 2, 1)))) + (0.0252f * (__SBREF(__a_sb, 2, 2)))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { float etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_3_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((((((-(0.3312f * (__REGREF(__a, 0, 0)))) + (0.0023f * (__SBREF(__a_sb, -2, -2)))) + (0.0033f * (__SBREF(__a_sb, -2, -1)))) + (0.0043f * (__SBREF(__a_sb, -2, 0)))) + (0.0053f * (__SBREF(__a_sb, -2, 1)))) + (0.0063f * (__SBREF(__a_sb, -2, 2)))) + (0.0073f * (__SBREF(__a_sb, -1, -2)))) + (0.0083f * (__SBREF(__a_sb, -1, -1)))) + (0.0093f * (__SBREF(__a_sb, -1, 0)))) + (0.0103f * (__SBREF(__a_sb, -1, 1)))) + (0.0113f * (__SBREF(__a_sb, -1, 2)))) + (0.0123f * (__SBREF(__a_sb, 0, -2)))) + (0.0133f * (__SBREF(__a_sb, 0, -1)))) + (0.0143f * (__SBREF(__a_sb, 0, 1)))) + (0.0153f * (__SBREF(__a_sb, 0, 2)))) + (0.0163f * (__SBREF(__a_sb, 1, -2)))) + (0.0173f * (__SBREF(__a_sb, 1, -1)))) + (0.0183f * (__SBREF(__a_sb, 1, 0)))) + (0.0193f * (__SBREF(__a_sb, 1, 1)))) + (0.0203f * (__SBREF(__a_sb, 1, 2)))) + (0.0213f * (__SBREF(__a_sb, 2, -2)))) + (0.0223f * (__SBREF(__a_sb, 2, -1)))) + (0.0233f * (__SBREF(__a_sb, 2, 0)))) + (0.0243f * (__SBREF(__a_sb, 2, 1)))) + (0.0253f * (__SBREF(__a_sb, 2, 2))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_3(out, a) do { float etmp; __CALCEXPR_3_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_4_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((-(0.3336f * (__REGREF(__a, 0, 0)))) + (0.0024f * (__SBREF(__a_sb, -2, -2)))) + (0.0034f * (__SBREF(__a_sb, -2, -1)))) + (0.0044f * (__SBREF(__a_sb, -2, 0)))) + (0.0054f * (__SBREF(__a_sb, -2, 1)))) + (0.0064f * (__SBREF(__a_sb, -2, 2)))) + (0.0074f * (__SBREF(__a_sb, -1, -2)))) + (0.0084f * (__SBREF(__a_sb, -1, -1)))) + (0.0094f * (__SBREF(__a_sb, -1, 0)))) + (0.0104f * (__SBREF(__a_sb, -1, 1)))) + (0.0114f * (__SBREF(__a_sb, -1, 2)))) + (0.0124f * (__SBREF(__a_sb, 0, -2)))) + (0.0134f * (__SBREF(__a_sb, 0, -1)))) + (0.0144f * (__SBREF(__a_sb, 0, 1)))) + (0.0154f * (__SBREF(__a_sb, 0, 2)))) + (0.0164f * (__SBREF(__a_sb, 1, -2)))) + (0.0174f * (__SBREF(__a_sb, 1, -1)))) + (0.0184f * (__SBREF(__a_sb, 1, 0)))) + (0.0194f * (__SBREF(__a_sb, 1, 1)))) + (0.0204f * (__SBREF(__a_sb, 1, 2)))) + (0.0214f * (__SBREF(__a_sb, 2, -2)))) + (0.0224f * (__SBREF(__a_sb, 2, -1)))) + (0.0234f * (__SBREF(__a_sb, 2, 0)))) + (0.0244f * (__SBREF(__a_sb, 2, 1)))) + (0.0254f * (__SBREF(__a_sb, 2, 2)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_4(out, a) do { float etmp; __CALCEXPR_4_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, out3, out4, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); __CALCEXPR_3(out3, reg); __CALCEXPR_4(out4, reg); } while (0);
    #define __CALC1(out0, out1, out2, out3, out4, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, out3, out4, reg); } else out2 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __STORE(2, __reg_1_2);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __STORE(2, __reg_1_2);
    }
    __a_sb = __a_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 5; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 7;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 2, __reg_1_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __STORE(__h - 2, __reg_1_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 2, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 2, __reg_1_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h - 2, __reg_1_2);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 2, __reg_1_3);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_4, __reg_0);
        __STORE(__h - 1, __reg_1_4);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 2, __reg_1_3);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_0, __reg_1_4, __reg_0);
        __STORE(__h - 1, __reg_1_4);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 2, __reg_1_3);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __STORE(__h - 1, __reg_1_4);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 2, __reg_1_3);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __STORE(__h - 1, __reg_1_4);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 2, __reg_1_3);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __STORE(__h - 1, __reg_1_4);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
      }
    }
    else
    {
      for (__h = 5; __h <= __side1LenOl - 5;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 2, __reg_1_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
        __STORE(__h - 2, __reg_1_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 2, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 2, __reg_1_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h - 2, __reg_1_2);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_0);
      __STORE(__h - 2, __reg_1_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_4, __reg_0);
      __STORE(__h - 2, __reg_1_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __STORE(__h - 2, __reg_1_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __STORE(__h - 2, __reg_1_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __STORE(__h - 2, __reg_1_2);
      __h++;
    }
}
