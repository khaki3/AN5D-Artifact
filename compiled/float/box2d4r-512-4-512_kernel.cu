#include "hip/hip_runtime.h"
#include "box2d4r-512-4-512_kernel.hu"
__device__ float __sbref_wrap(float *sb, size_t index) { return sb[index]; }

__global__ void kernel0_4(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 4 - 4);
    const AN5D_TYPE __c1Pad = (4);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 4 - 4);
    const AN5D_TYPE __c2Pad = (4);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 4;
    const AN5D_TYPE __halo2 = 4;
    const AN5D_TYPE __side0Len = 4;
    const AN5D_TYPE __side1Len = 512;
    const AN5D_TYPE __side2Len = 480;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_1_5;
    float __reg_1_6;
    float __reg_1_7;
    float __reg_1_8;
    float __reg_2_0;
    float __reg_2_1;
    float __reg_2_2;
    float __reg_2_3;
    float __reg_2_4;
    float __reg_2_5;
    float __reg_2_6;
    float __reg_2_7;
    float __reg_2_8;
    float __reg_3_0;
    float __reg_3_1;
    float __reg_3_2;
    float __reg_3_3;
    float __reg_3_4;
    float __reg_3_5;
    float __reg_3_6;
    float __reg_3_7;
    float __reg_3_8;
    float __reg_4_0;
    float __reg_4_1;
    float __reg_4_2;
    float __reg_4_3;
    float __reg_4_4;
    float __reg_4_5;
    float __reg_4_6;
    float __reg_4_7;
    float __reg_4_8;
    __shared__ float __a_sb_double[__blockSize * 2];
    float *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __writeValid3 = __updateValid && __local_c2 >= (__halo2 * 3) && __local_c2 < __side2LenOl - (__halo2 * 3);
    const AN5D_TYPE __writeValid4 = __updateValid && __local_c2 >= (__halo2 * 4) && __local_c2 < __side2LenOl - (__halo2 * 4);
    const AN5D_TYPE __storeValid = __writeValid4;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00930f * (__SBREF(__a_sb, -4))) + (0.00931f * (__SBREF(__a_sb, -3)))) + (0.00932f * (__SBREF(__a_sb, -2)))) + (0.00933f * (__SBREF(__a_sb, -1)))) + (0.00934f * (__REGREF(__a, 0)))) + (0.00935f * (__SBREF(__a_sb, 1)))) + (0.00936f * (__SBREF(__a_sb, 2)))) + (0.00937f * (__SBREF(__a_sb, 3)))) + (0.00938f * (__SBREF(__a_sb, 4)))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00939f * (__SBREF(__a_sb, -4)))) + (0.00940f * (__SBREF(__a_sb, -3)))) + (0.00941f * (__SBREF(__a_sb, -2)))) + (0.00942f * (__SBREF(__a_sb, -1)))) + (0.00943f * (__REGREF(__a, 0)))) + (0.00944f * (__SBREF(__a_sb, 1)))) + (0.00945f * (__SBREF(__a_sb, 2)))) + (0.00946f * (__SBREF(__a_sb, 3)))) + (0.00947f * (__SBREF(__a_sb, 4))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { float etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00948f * (__SBREF(__a_sb, -4)))) + (0.00949f * (__SBREF(__a_sb, -3)))) + (0.00950f * (__SBREF(__a_sb, -2)))) + (0.00951f * (__SBREF(__a_sb, -1)))) + (0.00952f * (__REGREF(__a, 0)))) + (0.00953f * (__SBREF(__a_sb, 1)))) + (0.00954f * (__SBREF(__a_sb, 2)))) + (0.00955f * (__SBREF(__a_sb, 3)))) + (0.00956f * (__SBREF(__a_sb, 4)))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { float etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_3_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00957f * (__SBREF(__a_sb, -4)))) + (0.00958f * (__SBREF(__a_sb, -3)))) + (0.00959f * (__SBREF(__a_sb, -2)))) + (0.00960f * (__SBREF(__a_sb, -1)))) + (0.00961f * (__REGREF(__a, 0)))) + (0.00962f * (__SBREF(__a_sb, 1)))) + (0.00963f * (__SBREF(__a_sb, 2)))) + (0.00964f * (__SBREF(__a_sb, 3)))) + (0.00965f * (__SBREF(__a_sb, 4))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_3(out, a) do { float etmp; __CALCEXPR_3_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_4_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((0.00966f * (__SBREF(__a_sb, -4)))) + (0.00967f * (__SBREF(__a_sb, -3)))) + (0.00968f * (__SBREF(__a_sb, -2)))) + (0.00969f * (__SBREF(__a_sb, -1)))) + (0.22400f * (__REGREF(__a, 0)))) + (0.00971f * (__SBREF(__a_sb, 1)))) + (0.00972f * (__SBREF(__a_sb, 2)))) + (0.00973f * (__SBREF(__a_sb, 3)))) + (0.00974f * (__SBREF(__a_sb, 4)))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_4(out, a) do { float etmp; __CALCEXPR_4_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_5_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((0.00975f * (__SBREF(__a_sb, -4)))) + (0.00976f * (__SBREF(__a_sb, -3)))) + (0.00977f * (__SBREF(__a_sb, -2)))) + (0.00978f * (__SBREF(__a_sb, -1)))) + (0.00979f * (__REGREF(__a, 0)))) + (0.00980f * (__SBREF(__a_sb, 1)))) + (0.00981f * (__SBREF(__a_sb, 2)))) + (0.00982f * (__SBREF(__a_sb, 3)))) + (0.00983f * (__SBREF(__a_sb, 4))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_5(out, a) do { float etmp; __CALCEXPR_5_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_6_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((0.00984f * (__SBREF(__a_sb, -4)))) + (0.00985f * (__SBREF(__a_sb, -3)))) + (0.00986f * (__SBREF(__a_sb, -2)))) + (0.00987f * (__SBREF(__a_sb, -1)))) + (0.00988f * (__REGREF(__a, 0)))) + (0.00989f * (__SBREF(__a_sb, 1)))) + (0.00990f * (__SBREF(__a_sb, 2)))) + (0.00991f * (__SBREF(__a_sb, 3)))) + (0.00992f * (__SBREF(__a_sb, 4)))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_6(out, a) do { float etmp; __CALCEXPR_6_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_7_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((0.00993f * (__SBREF(__a_sb, -4)))) + (0.00994f * (__SBREF(__a_sb, -3)))) + (0.00995f * (__SBREF(__a_sb, -2)))) + (0.00996f * (__SBREF(__a_sb, -1)))) + (0.00997f * (__REGREF(__a, 0)))) + (0.00998f * (__SBREF(__a_sb, 1)))) + (0.00999f * (__SBREF(__a_sb, 2)))) + (0.01000f * (__SBREF(__a_sb, 3)))) + (0.01001f * (__SBREF(__a_sb, 4))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_7(out, a) do { float etmp; __CALCEXPR_7_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_8_wrap(__rn0, __a) do { __rn0 = ((((((((((0.01002f * (__SBREF(__a_sb, -4)))) + (0.01003f * (__SBREF(__a_sb, -3)))) + (0.01004f * (__SBREF(__a_sb, -2)))) + (0.01005f * (__SBREF(__a_sb, -1)))) + (0.01006f * (__REGREF(__a, 0)))) + (0.01007f * (__SBREF(__a_sb, 1)))) + (0.01008f * (__SBREF(__a_sb, 2)))) + (0.01009f * (__SBREF(__a_sb, 3)))) + (0.01010f * (__SBREF(__a_sb, 4)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_8(out, a) do { float etmp; __CALCEXPR_8_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); __CALCEXPR_3(out3, reg); __CALCEXPR_4(out4, reg); __CALCEXPR_5(out5, reg); __CALCEXPR_6(out6, reg); __CALCEXPR_7(out7, reg); __CALCEXPR_8(out8, reg); } while (0);
    #define __CALC1(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg); } else out4 = reg; } while (0)
    #define __CALC2(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCSETUP(reg); if (__writeValid2) { __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg); } else out4 = reg; } while (0)
    #define __CALC3(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCSETUP(reg); if (__writeValid3) { __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg); } else out4 = reg; } while (0)
    #define __CALC4(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCSETUP(reg); if (__writeValid4) { __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg); } else out4 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_0);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_0);
      __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_0);
      __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_0);
      __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_0);
      __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_0);
      __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_0);
      __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_0);
      __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
      __LOAD(__reg_0, 9);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
      __LOAD(__reg_0, 10);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
      __LOAD(__reg_0, 11);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
      __LOAD(__reg_0, 12);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
      __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
      __LOAD(__reg_0, 13);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
      __LOAD(__reg_0, 14);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
      __LOAD(__reg_0, 15);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
      __LOAD(__reg_0, 16);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
      __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
      __LOAD(__reg_0, 17);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
      __LOAD(__reg_0, 18);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
      __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
      __LOAD(__reg_0, 19);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
      __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
      __LOAD(__reg_0, 20);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
      __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
      __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
      __STORE(4, __reg_4_4);
      __LOAD(__reg_0, 21);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
      __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
      __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
      __STORE(5, __reg_4_5);
      __LOAD(__reg_0, 22);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
      __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
      __STORE(6, __reg_4_6);
      __LOAD(__reg_0, 23);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
      __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
      __STORE(7, __reg_4_7);
      __LOAD(__reg_0, 24);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
      __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
      __STORE(8, __reg_4_8);
      __LOAD(__reg_0, 25);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
      __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
      __STORE(9, __reg_4_0);
      __LOAD(__reg_0, 26);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
      __STORE(10, __reg_4_1);
      __LOAD(__reg_0, 27);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
      __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
      __STORE(11, __reg_4_2);
      __LOAD(__reg_0, 28);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
      __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
      __STORE(12, __reg_4_3);
      __LOAD(__reg_0, 29);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
      __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
      __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
      __STORE(13, __reg_4_4);
      __LOAD(__reg_0, 30);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
      __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
      __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
      __STORE(14, __reg_4_5);
      __LOAD(__reg_0, 31);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
      __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
      __STORE(15, __reg_4_6);
      __LOAD(__reg_0, 32);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
      __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
      __STORE(16, __reg_4_7);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
      __LOAD(__reg_0, 9);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
      __LOAD(__reg_0, 10);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
      __LOAD(__reg_0, 11);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
      __LOAD(__reg_0, 12);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
      __LOAD(__reg_0, 13);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
      __LOAD(__reg_0, 14);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
      __LOAD(__reg_0, 15);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
      __LOAD(__reg_0, 16);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
      __LOAD(__reg_0, 17);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
      __LOAD(__reg_0, 18);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
      __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
      __LOAD(__reg_0, 19);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
      __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
      __LOAD(__reg_0, 20);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
      __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
      __LOAD(__reg_0, 21);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
      __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
      __LOAD(__reg_0, 22);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
      __LOAD(__reg_0, 23);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
      __LOAD(__reg_0, 24);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
      __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
      __LOAD(__reg_0, 25);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
      __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
      __LOAD(__reg_0, 26);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
      __LOAD(__reg_0, 27);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
      __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
      __LOAD(__reg_0, 28);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
      __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
      __LOAD(__reg_0, 29);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
      __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
      __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
      __LOAD(__reg_0, 30);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
      __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
      __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
      __LOAD(__reg_0, 31);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
      __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
      __LOAD(__reg_0, 32);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
      __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
      __STORE(16, __reg_4_7);
    }
    __a_sb = __a_sb_double + __blockSize * 0;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 33; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 13;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 16, __reg_4_8);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 16, __reg_4_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 16, __reg_4_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 16, __reg_4_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 16, __reg_4_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 16, __reg_4_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 16, __reg_4_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h - 16, __reg_4_6);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h - 16, __reg_4_7);
        __h++;
      }
      if (0) {}
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 16, __reg_4_8);
        __reg_1_6 = __reg_0;
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 15, __reg_4_0);
        __reg_1_7 = __reg_0;
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 14, __reg_4_1);
        __reg_1_8 = __reg_0;
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 13, __reg_4_2);
        __reg_1_0 = __reg_0;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 12, __reg_4_3);
        __reg_2_6 = __reg_1_6;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 11, __reg_4_4);
        __reg_2_7 = __reg_1_7;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 10, __reg_4_5);
        __reg_2_8 = __reg_1_8;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h - 9, __reg_4_6);
        __reg_2_0 = __reg_1_0;
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h - 8, __reg_4_7);
        __reg_3_6 = __reg_2_6;
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 7, __reg_4_8);
        __reg_3_7 = __reg_2_7;
        __CALC3(__reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 6, __reg_4_0);
        __reg_3_8 = __reg_2_8;
        __CALC3(__reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 5, __reg_4_1);
        __reg_3_0 = __reg_2_0;
        __CALC4(__reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 4, __reg_4_2);
        __CALC4(__reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 3, __reg_4_3);
        __CALC4(__reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 2, __reg_4_4);
        __CALC4(__reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_5, __reg_3_0);
        __STORE(__h - 1, __reg_4_5);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 16, __reg_4_8);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 15, __reg_4_0);
        __reg_1_7 = __reg_0;
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 14, __reg_4_1);
        __reg_1_8 = __reg_0;
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 13, __reg_4_2);
        __reg_1_0 = __reg_0;
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 12, __reg_4_3);
        __reg_1_1 = __reg_0;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 11, __reg_4_4);
        __reg_2_7 = __reg_1_7;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 10, __reg_4_5);
        __reg_2_8 = __reg_1_8;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h - 9, __reg_4_6);
        __reg_2_0 = __reg_1_0;
        __CALC2(__reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h - 8, __reg_4_7);
        __reg_2_1 = __reg_1_1;
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 7, __reg_4_8);
        __reg_3_7 = __reg_2_7;
        __CALC3(__reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 6, __reg_4_0);
        __reg_3_8 = __reg_2_8;
        __CALC3(__reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 5, __reg_4_1);
        __reg_3_0 = __reg_2_0;
        __CALC3(__reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 4, __reg_4_2);
        __reg_3_1 = __reg_2_1;
        __CALC4(__reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 3, __reg_4_3);
        __CALC4(__reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 2, __reg_4_4);
        __CALC4(__reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 1, __reg_4_5);
        __CALC4(__reg_4_5, __reg_4_5, __reg_4_5, __reg_4_5, __reg_4_5, __reg_4_5, __reg_4_5, __reg_4_5, __reg_4_6, __reg_3_1);
        __STORE(__h + 0, __reg_4_6);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 16, __reg_4_8);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 15, __reg_4_0);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 14, __reg_4_1);
        __reg_1_8 = __reg_0;
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 13, __reg_4_2);
        __reg_1_0 = __reg_0;
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 12, __reg_4_3);
        __reg_1_1 = __reg_0;
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 11, __reg_4_4);
        __reg_1_2 = __reg_0;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 10, __reg_4_5);
        __reg_2_8 = __reg_1_8;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h - 9, __reg_4_6);
        __reg_2_0 = __reg_1_0;
        __CALC2(__reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h - 8, __reg_4_7);
        __reg_2_1 = __reg_1_1;
        __CALC2(__reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 7, __reg_4_8);
        __reg_2_2 = __reg_1_2;
        __CALC3(__reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 6, __reg_4_0);
        __reg_3_8 = __reg_2_8;
        __CALC3(__reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 5, __reg_4_1);
        __reg_3_0 = __reg_2_0;
        __CALC3(__reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 4, __reg_4_2);
        __reg_3_1 = __reg_2_1;
        __CALC3(__reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 3, __reg_4_3);
        __reg_3_2 = __reg_2_2;
        __CALC4(__reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 2, __reg_4_4);
        __CALC4(__reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 1, __reg_4_5);
        __CALC4(__reg_4_5, __reg_4_5, __reg_4_5, __reg_4_5, __reg_4_5, __reg_4_5, __reg_4_5, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h + 0, __reg_4_6);
        __CALC4(__reg_4_6, __reg_4_6, __reg_4_6, __reg_4_6, __reg_4_6, __reg_4_6, __reg_4_6, __reg_4_6, __reg_4_7, __reg_3_2);
        __STORE(__h + 1, __reg_4_7);
      }
      else if (__h + 7 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 16, __reg_4_8);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 15, __reg_4_0);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 14, __reg_4_1);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 13, __reg_4_2);
        __reg_1_0 = __reg_0;
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 12, __reg_4_3);
        __reg_1_1 = __reg_0;
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 11, __reg_4_4);
        __reg_1_2 = __reg_0;
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 10, __reg_4_5);
        __reg_1_3 = __reg_0;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h - 9, __reg_4_6);
        __reg_2_0 = __reg_1_0;
        __CALC2(__reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h - 8, __reg_4_7);
        __reg_2_1 = __reg_1_1;
        __CALC2(__reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 7, __reg_4_8);
        __reg_2_2 = __reg_1_2;
        __CALC2(__reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 6, __reg_4_0);
        __reg_2_3 = __reg_1_3;
        __CALC3(__reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 5, __reg_4_1);
        __reg_3_0 = __reg_2_0;
        __CALC3(__reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 4, __reg_4_2);
        __reg_3_1 = __reg_2_1;
        __CALC3(__reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 3, __reg_4_3);
        __reg_3_2 = __reg_2_2;
        __CALC3(__reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 2, __reg_4_4);
        __reg_3_3 = __reg_2_3;
        __CALC4(__reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_4, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 1, __reg_4_5);
        __CALC4(__reg_4_5, __reg_4_5, __reg_4_5, __reg_4_5, __reg_4_5, __reg_4_5, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h + 0, __reg_4_6);
        __CALC4(__reg_4_6, __reg_4_6, __reg_4_6, __reg_4_6, __reg_4_6, __reg_4_6, __reg_4_6, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h + 1, __reg_4_7);
        __CALC4(__reg_4_7, __reg_4_7, __reg_4_7, __reg_4_7, __reg_4_7, __reg_4_7, __reg_4_7, __reg_4_7, __reg_4_8, __reg_3_3);
        __STORE(__h + 2, __reg_4_8);
      }
      else if (__h + 8 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 16, __reg_4_8);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 15, __reg_4_0);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 14, __reg_4_1);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 13, __reg_4_2);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 12, __reg_4_3);
        __reg_1_1 = __reg_0;
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 11, __reg_4_4);
        __reg_1_2 = __reg_0;
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 10, __reg_4_5);
        __reg_1_3 = __reg_0;
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h - 9, __reg_4_6);
        __reg_1_4 = __reg_0;
        __CALC2(__reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h - 8, __reg_4_7);
        __reg_2_1 = __reg_1_1;
        __CALC2(__reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 7, __reg_4_8);
        __reg_2_2 = __reg_1_2;
        __CALC2(__reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 6, __reg_4_0);
        __reg_2_3 = __reg_1_3;
        __CALC2(__reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 5, __reg_4_1);
        __reg_2_4 = __reg_1_4;
        __CALC3(__reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 4, __reg_4_2);
        __reg_3_1 = __reg_2_1;
        __CALC3(__reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 3, __reg_4_3);
        __reg_3_2 = __reg_2_2;
        __CALC3(__reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 2, __reg_4_4);
        __reg_3_3 = __reg_2_3;
        __CALC3(__reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 1, __reg_4_5);
        __reg_3_4 = __reg_2_4;
        __CALC4(__reg_4_5, __reg_4_5, __reg_4_5, __reg_4_5, __reg_4_5, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h + 0, __reg_4_6);
        __CALC4(__reg_4_6, __reg_4_6, __reg_4_6, __reg_4_6, __reg_4_6, __reg_4_6, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h + 1, __reg_4_7);
        __CALC4(__reg_4_7, __reg_4_7, __reg_4_7, __reg_4_7, __reg_4_7, __reg_4_7, __reg_4_7, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h + 2, __reg_4_8);
        __CALC4(__reg_4_8, __reg_4_8, __reg_4_8, __reg_4_8, __reg_4_8, __reg_4_8, __reg_4_8, __reg_4_8, __reg_4_0, __reg_3_4);
        __STORE(__h + 3, __reg_4_0);
      }
      else if (__h + 9 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 16, __reg_4_8);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 15, __reg_4_0);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 14, __reg_4_1);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 13, __reg_4_2);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 12, __reg_4_3);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 11, __reg_4_4);
        __reg_1_2 = __reg_0;
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 10, __reg_4_5);
        __reg_1_3 = __reg_0;
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h - 9, __reg_4_6);
        __reg_1_4 = __reg_0;
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h - 8, __reg_4_7);
        __reg_1_5 = __reg_0;
        __CALC2(__reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 7, __reg_4_8);
        __reg_2_2 = __reg_1_2;
        __CALC2(__reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 6, __reg_4_0);
        __reg_2_3 = __reg_1_3;
        __CALC2(__reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 5, __reg_4_1);
        __reg_2_4 = __reg_1_4;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 4, __reg_4_2);
        __reg_2_5 = __reg_1_5;
        __CALC3(__reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 3, __reg_4_3);
        __reg_3_2 = __reg_2_2;
        __CALC3(__reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 2, __reg_4_4);
        __reg_3_3 = __reg_2_3;
        __CALC3(__reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_1, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 1, __reg_4_5);
        __reg_3_4 = __reg_2_4;
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_1, __reg_2_5);
        __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h + 0, __reg_4_6);
        __reg_3_5 = __reg_2_5;
        __CALC4(__reg_4_6, __reg_4_6, __reg_4_6, __reg_4_6, __reg_4_6, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h + 1, __reg_4_7);
        __CALC4(__reg_4_7, __reg_4_7, __reg_4_7, __reg_4_7, __reg_4_7, __reg_4_7, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h + 2, __reg_4_8);
        __CALC4(__reg_4_8, __reg_4_8, __reg_4_8, __reg_4_8, __reg_4_8, __reg_4_8, __reg_4_8, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h + 3, __reg_4_0);
        __CALC4(__reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_1, __reg_3_5);
        __STORE(__h + 4, __reg_4_1);
      }
      else if (__h + 10 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 16, __reg_4_8);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 15, __reg_4_0);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 14, __reg_4_1);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 13, __reg_4_2);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 12, __reg_4_3);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 11, __reg_4_4);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 10, __reg_4_5);
        __reg_1_3 = __reg_0;
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h - 9, __reg_4_6);
        __reg_1_4 = __reg_0;
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h - 8, __reg_4_7);
        __reg_1_5 = __reg_0;
        __LOAD(__reg_0, __h + 9);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 7, __reg_4_8);
        __reg_1_6 = __reg_0;
        __CALC2(__reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 6, __reg_4_0);
        __reg_2_3 = __reg_1_3;
        __CALC2(__reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 5, __reg_4_1);
        __reg_2_4 = __reg_1_4;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 4, __reg_4_2);
        __reg_2_5 = __reg_1_5;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 3, __reg_4_3);
        __reg_2_6 = __reg_1_6;
        __CALC3(__reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 2, __reg_4_4);
        __reg_3_3 = __reg_2_3;
        __CALC3(__reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 1, __reg_4_5);
        __reg_3_4 = __reg_2_4;
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_2, __reg_3_1, __reg_2_5);
        __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h + 0, __reg_4_6);
        __reg_3_5 = __reg_2_5;
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_2, __reg_2_6);
        __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h + 1, __reg_4_7);
        __reg_3_6 = __reg_2_6;
        __CALC4(__reg_4_7, __reg_4_7, __reg_4_7, __reg_4_7, __reg_4_7, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h + 2, __reg_4_8);
        __CALC4(__reg_4_8, __reg_4_8, __reg_4_8, __reg_4_8, __reg_4_8, __reg_4_8, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h + 3, __reg_4_0);
        __CALC4(__reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h + 4, __reg_4_1);
        __CALC4(__reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_2, __reg_3_6);
        __STORE(__h + 5, __reg_4_2);
      }
      else if (__h + 11 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 16, __reg_4_8);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 15, __reg_4_0);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 14, __reg_4_1);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 13, __reg_4_2);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 12, __reg_4_3);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 11, __reg_4_4);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 10, __reg_4_5);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h - 9, __reg_4_6);
        __reg_1_4 = __reg_0;
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h - 8, __reg_4_7);
        __reg_1_5 = __reg_0;
        __LOAD(__reg_0, __h + 9);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 7, __reg_4_8);
        __reg_1_6 = __reg_0;
        __LOAD(__reg_0, __h + 10);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 6, __reg_4_0);
        __reg_1_7 = __reg_0;
        __CALC2(__reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 5, __reg_4_1);
        __reg_2_4 = __reg_1_4;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 4, __reg_4_2);
        __reg_2_5 = __reg_1_5;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 3, __reg_4_3);
        __reg_2_6 = __reg_1_6;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 2, __reg_4_4);
        __reg_2_7 = __reg_1_7;
        __CALC3(__reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 1, __reg_4_5);
        __reg_3_4 = __reg_2_4;
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h + 0, __reg_4_6);
        __reg_3_5 = __reg_2_5;
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_3, __reg_3_2, __reg_2_6);
        __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h + 1, __reg_4_7);
        __reg_3_6 = __reg_2_6;
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h + 2, __reg_4_8);
        __reg_3_7 = __reg_2_7;
        __CALC4(__reg_4_8, __reg_4_8, __reg_4_8, __reg_4_8, __reg_4_8, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h + 3, __reg_4_0);
        __CALC4(__reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h + 4, __reg_4_1);
        __CALC4(__reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h + 5, __reg_4_2);
        __CALC4(__reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_3, __reg_3_7);
        __STORE(__h + 6, __reg_4_3);
      }
      else if (__h + 12 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 16, __reg_4_8);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 15, __reg_4_0);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 14, __reg_4_1);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 13, __reg_4_2);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 12, __reg_4_3);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 11, __reg_4_4);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 10, __reg_4_5);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h - 9, __reg_4_6);
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h - 8, __reg_4_7);
        __reg_1_5 = __reg_0;
        __LOAD(__reg_0, __h + 9);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 7, __reg_4_8);
        __reg_1_6 = __reg_0;
        __LOAD(__reg_0, __h + 10);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 6, __reg_4_0);
        __reg_1_7 = __reg_0;
        __LOAD(__reg_0, __h + 11);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 5, __reg_4_1);
        __reg_1_8 = __reg_0;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 4, __reg_4_2);
        __reg_2_5 = __reg_1_5;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 3, __reg_4_3);
        __reg_2_6 = __reg_1_6;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 2, __reg_4_4);
        __reg_2_7 = __reg_1_7;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 1, __reg_4_5);
        __reg_2_8 = __reg_1_8;
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h + 0, __reg_4_6);
        __reg_3_5 = __reg_2_5;
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h + 1, __reg_4_7);
        __reg_3_6 = __reg_2_6;
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h + 2, __reg_4_8);
        __reg_3_7 = __reg_2_7;
        __CALC3(__reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h + 3, __reg_4_0);
        __reg_3_8 = __reg_2_8;
        __CALC4(__reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_0, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h + 4, __reg_4_1);
        __CALC4(__reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_1, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h + 5, __reg_4_2);
        __CALC4(__reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_2, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h + 6, __reg_4_3);
        __CALC4(__reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_3, __reg_4_4, __reg_3_8);
        __STORE(__h + 7, __reg_4_4);
      }
    }
    else
    {
      for (__h = 33; __h <= __side1LenOl - 9;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
        __STORE(__h - 16, __reg_4_8);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
        __STORE(__h - 16, __reg_4_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
        __STORE(__h - 16, __reg_4_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
        __STORE(__h - 16, __reg_4_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
        __STORE(__h - 16, __reg_4_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
        __STORE(__h - 16, __reg_4_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
        __STORE(__h - 16, __reg_4_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
        __STORE(__h - 16, __reg_4_6);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
        __STORE(__h - 16, __reg_4_7);
        __h++;
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
      __CALC4(__reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_3_3);
      __STORE(__h - 16, __reg_4_8);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
      __CALC4(__reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_3_4);
      __STORE(__h - 16, __reg_4_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_3_5);
      __STORE(__h - 16, __reg_4_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
      __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_3_6);
      __STORE(__h - 16, __reg_4_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
      __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_3_7);
      __STORE(__h - 16, __reg_4_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
      __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
      __CALC4(__reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_4_4, __reg_3_8);
      __STORE(__h - 16, __reg_4_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
      __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
      __CALC4(__reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_4_5, __reg_3_0);
      __STORE(__h - 16, __reg_4_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
      __CALC4(__reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_4_6, __reg_3_1);
      __STORE(__h - 16, __reg_4_6);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
      __CALC4(__reg_4_6, __reg_4_5, __reg_4_4, __reg_4_3, __reg_4_2, __reg_4_1, __reg_4_0, __reg_4_8, __reg_4_7, __reg_3_2);
      __STORE(__h - 16, __reg_4_7);
      __h++;
    }
}
__global__ void kernel0_3(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 4 - 4);
    const AN5D_TYPE __c1Pad = (4);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 4 - 4);
    const AN5D_TYPE __c2Pad = (4);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 4;
    const AN5D_TYPE __halo2 = 4;
    const AN5D_TYPE __side0Len = 3;
    const AN5D_TYPE __side1Len = 512;
    const AN5D_TYPE __side2Len = 488;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_1_5;
    float __reg_1_6;
    float __reg_1_7;
    float __reg_1_8;
    float __reg_2_0;
    float __reg_2_1;
    float __reg_2_2;
    float __reg_2_3;
    float __reg_2_4;
    float __reg_2_5;
    float __reg_2_6;
    float __reg_2_7;
    float __reg_2_8;
    float __reg_3_0;
    float __reg_3_1;
    float __reg_3_2;
    float __reg_3_3;
    float __reg_3_4;
    float __reg_3_5;
    float __reg_3_6;
    float __reg_3_7;
    float __reg_3_8;
    __shared__ float __a_sb_double[__blockSize * 2];
    float *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __writeValid3 = __updateValid && __local_c2 >= (__halo2 * 3) && __local_c2 < __side2LenOl - (__halo2 * 3);
    const AN5D_TYPE __storeValid = __writeValid3;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00930f * (__SBREF(__a_sb, -4))) + (0.00931f * (__SBREF(__a_sb, -3)))) + (0.00932f * (__SBREF(__a_sb, -2)))) + (0.00933f * (__SBREF(__a_sb, -1)))) + (0.00934f * (__REGREF(__a, 0)))) + (0.00935f * (__SBREF(__a_sb, 1)))) + (0.00936f * (__SBREF(__a_sb, 2)))) + (0.00937f * (__SBREF(__a_sb, 3)))) + (0.00938f * (__SBREF(__a_sb, 4)))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00939f * (__SBREF(__a_sb, -4)))) + (0.00940f * (__SBREF(__a_sb, -3)))) + (0.00941f * (__SBREF(__a_sb, -2)))) + (0.00942f * (__SBREF(__a_sb, -1)))) + (0.00943f * (__REGREF(__a, 0)))) + (0.00944f * (__SBREF(__a_sb, 1)))) + (0.00945f * (__SBREF(__a_sb, 2)))) + (0.00946f * (__SBREF(__a_sb, 3)))) + (0.00947f * (__SBREF(__a_sb, 4))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { float etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00948f * (__SBREF(__a_sb, -4)))) + (0.00949f * (__SBREF(__a_sb, -3)))) + (0.00950f * (__SBREF(__a_sb, -2)))) + (0.00951f * (__SBREF(__a_sb, -1)))) + (0.00952f * (__REGREF(__a, 0)))) + (0.00953f * (__SBREF(__a_sb, 1)))) + (0.00954f * (__SBREF(__a_sb, 2)))) + (0.00955f * (__SBREF(__a_sb, 3)))) + (0.00956f * (__SBREF(__a_sb, 4)))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { float etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_3_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00957f * (__SBREF(__a_sb, -4)))) + (0.00958f * (__SBREF(__a_sb, -3)))) + (0.00959f * (__SBREF(__a_sb, -2)))) + (0.00960f * (__SBREF(__a_sb, -1)))) + (0.00961f * (__REGREF(__a, 0)))) + (0.00962f * (__SBREF(__a_sb, 1)))) + (0.00963f * (__SBREF(__a_sb, 2)))) + (0.00964f * (__SBREF(__a_sb, 3)))) + (0.00965f * (__SBREF(__a_sb, 4))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_3(out, a) do { float etmp; __CALCEXPR_3_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_4_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((0.00966f * (__SBREF(__a_sb, -4)))) + (0.00967f * (__SBREF(__a_sb, -3)))) + (0.00968f * (__SBREF(__a_sb, -2)))) + (0.00969f * (__SBREF(__a_sb, -1)))) + (0.22400f * (__REGREF(__a, 0)))) + (0.00971f * (__SBREF(__a_sb, 1)))) + (0.00972f * (__SBREF(__a_sb, 2)))) + (0.00973f * (__SBREF(__a_sb, 3)))) + (0.00974f * (__SBREF(__a_sb, 4)))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_4(out, a) do { float etmp; __CALCEXPR_4_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_5_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((0.00975f * (__SBREF(__a_sb, -4)))) + (0.00976f * (__SBREF(__a_sb, -3)))) + (0.00977f * (__SBREF(__a_sb, -2)))) + (0.00978f * (__SBREF(__a_sb, -1)))) + (0.00979f * (__REGREF(__a, 0)))) + (0.00980f * (__SBREF(__a_sb, 1)))) + (0.00981f * (__SBREF(__a_sb, 2)))) + (0.00982f * (__SBREF(__a_sb, 3)))) + (0.00983f * (__SBREF(__a_sb, 4))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_5(out, a) do { float etmp; __CALCEXPR_5_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_6_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((0.00984f * (__SBREF(__a_sb, -4)))) + (0.00985f * (__SBREF(__a_sb, -3)))) + (0.00986f * (__SBREF(__a_sb, -2)))) + (0.00987f * (__SBREF(__a_sb, -1)))) + (0.00988f * (__REGREF(__a, 0)))) + (0.00989f * (__SBREF(__a_sb, 1)))) + (0.00990f * (__SBREF(__a_sb, 2)))) + (0.00991f * (__SBREF(__a_sb, 3)))) + (0.00992f * (__SBREF(__a_sb, 4)))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_6(out, a) do { float etmp; __CALCEXPR_6_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_7_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((0.00993f * (__SBREF(__a_sb, -4)))) + (0.00994f * (__SBREF(__a_sb, -3)))) + (0.00995f * (__SBREF(__a_sb, -2)))) + (0.00996f * (__SBREF(__a_sb, -1)))) + (0.00997f * (__REGREF(__a, 0)))) + (0.00998f * (__SBREF(__a_sb, 1)))) + (0.00999f * (__SBREF(__a_sb, 2)))) + (0.01000f * (__SBREF(__a_sb, 3)))) + (0.01001f * (__SBREF(__a_sb, 4))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_7(out, a) do { float etmp; __CALCEXPR_7_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_8_wrap(__rn0, __a) do { __rn0 = ((((((((((0.01002f * (__SBREF(__a_sb, -4)))) + (0.01003f * (__SBREF(__a_sb, -3)))) + (0.01004f * (__SBREF(__a_sb, -2)))) + (0.01005f * (__SBREF(__a_sb, -1)))) + (0.01006f * (__REGREF(__a, 0)))) + (0.01007f * (__SBREF(__a_sb, 1)))) + (0.01008f * (__SBREF(__a_sb, 2)))) + (0.01009f * (__SBREF(__a_sb, 3)))) + (0.01010f * (__SBREF(__a_sb, 4)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_8(out, a) do { float etmp; __CALCEXPR_8_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); __CALCEXPR_3(out3, reg); __CALCEXPR_4(out4, reg); __CALCEXPR_5(out5, reg); __CALCEXPR_6(out6, reg); __CALCEXPR_7(out7, reg); __CALCEXPR_8(out8, reg); } while (0);
    #define __CALC1(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg); } else out4 = reg; } while (0)
    #define __CALC2(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCSETUP(reg); if (__writeValid2) { __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg); } else out4 = reg; } while (0)
    #define __CALC3(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCSETUP(reg); if (__writeValid3) { __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg); } else out4 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_0);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_0);
      __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_0);
      __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_0);
      __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
      __LOAD(__reg_0, 9);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
      __LOAD(__reg_0, 10);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
      __LOAD(__reg_0, 11);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
      __LOAD(__reg_0, 12);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
      __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
      __LOAD(__reg_0, 13);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
      __LOAD(__reg_0, 14);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
      __LOAD(__reg_0, 15);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
      __LOAD(__reg_0, 16);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
      __STORE(4, __reg_3_4);
      __LOAD(__reg_0, 17);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
      __STORE(5, __reg_3_5);
      __LOAD(__reg_0, 18);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
      __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
      __STORE(6, __reg_3_6);
      __LOAD(__reg_0, 19);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
      __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
      __STORE(7, __reg_3_7);
      __LOAD(__reg_0, 20);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
      __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
      __STORE(8, __reg_3_8);
      __LOAD(__reg_0, 21);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
      __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
      __STORE(9, __reg_3_0);
      __LOAD(__reg_0, 22);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
      __STORE(10, __reg_3_1);
      __LOAD(__reg_0, 23);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
      __STORE(11, __reg_3_2);
      __LOAD(__reg_0, 24);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
      __STORE(12, __reg_3_3);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
      __LOAD(__reg_0, 9);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
      __LOAD(__reg_0, 10);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
      __LOAD(__reg_0, 11);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
      __LOAD(__reg_0, 12);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
      __LOAD(__reg_0, 13);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
      __LOAD(__reg_0, 14);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
      __LOAD(__reg_0, 15);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
      __LOAD(__reg_0, 16);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
      __LOAD(__reg_0, 17);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
      __LOAD(__reg_0, 18);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
      __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
      __LOAD(__reg_0, 19);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
      __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
      __LOAD(__reg_0, 20);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
      __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
      __LOAD(__reg_0, 21);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
      __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
      __LOAD(__reg_0, 22);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
      __LOAD(__reg_0, 23);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
      __LOAD(__reg_0, 24);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
      __STORE(12, __reg_3_3);
    }
    __a_sb = __a_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 25; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 13;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 12, __reg_3_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 12, __reg_3_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 12, __reg_3_6);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h - 12, __reg_3_7);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h - 12, __reg_3_8);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h - 12, __reg_3_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __STORE(__h - 12, __reg_3_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __STORE(__h - 12, __reg_3_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __STORE(__h - 12, __reg_3_3);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 12, __reg_3_4);
        __reg_1_7 = __reg_0;
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 11, __reg_3_5);
        __reg_1_8 = __reg_0;
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 10, __reg_3_6);
        __reg_1_0 = __reg_0;
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h - 9, __reg_3_7);
        __reg_1_1 = __reg_0;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h - 8, __reg_3_8);
        __reg_2_7 = __reg_1_7;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h - 7, __reg_3_0);
        __reg_2_8 = __reg_1_8;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __STORE(__h - 6, __reg_3_1);
        __reg_2_0 = __reg_1_0;
        __CALC2(__reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __STORE(__h - 5, __reg_3_2);
        __reg_2_1 = __reg_1_1;
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __STORE(__h - 4, __reg_3_3);
        __CALC3(__reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 3, __reg_3_4);
        __CALC3(__reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 2, __reg_3_5);
        __CALC3(__reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_6, __reg_2_1);
        __STORE(__h - 1, __reg_3_6);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 12, __reg_3_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 11, __reg_3_5);
        __reg_1_8 = __reg_0;
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 10, __reg_3_6);
        __reg_1_0 = __reg_0;
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h - 9, __reg_3_7);
        __reg_1_1 = __reg_0;
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h - 8, __reg_3_8);
        __reg_1_2 = __reg_0;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h - 7, __reg_3_0);
        __reg_2_8 = __reg_1_8;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __STORE(__h - 6, __reg_3_1);
        __reg_2_0 = __reg_1_0;
        __CALC2(__reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __STORE(__h - 5, __reg_3_2);
        __reg_2_1 = __reg_1_1;
        __CALC2(__reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __STORE(__h - 4, __reg_3_3);
        __reg_2_2 = __reg_1_2;
        __CALC3(__reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 3, __reg_3_4);
        __CALC3(__reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 2, __reg_3_5);
        __CALC3(__reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 1, __reg_3_6);
        __CALC3(__reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_7, __reg_2_2);
        __STORE(__h + 0, __reg_3_7);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 12, __reg_3_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 11, __reg_3_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 10, __reg_3_6);
        __reg_1_0 = __reg_0;
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h - 9, __reg_3_7);
        __reg_1_1 = __reg_0;
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h - 8, __reg_3_8);
        __reg_1_2 = __reg_0;
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h - 7, __reg_3_0);
        __reg_1_3 = __reg_0;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __STORE(__h - 6, __reg_3_1);
        __reg_2_0 = __reg_1_0;
        __CALC2(__reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __STORE(__h - 5, __reg_3_2);
        __reg_2_1 = __reg_1_1;
        __CALC2(__reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __STORE(__h - 4, __reg_3_3);
        __reg_2_2 = __reg_1_2;
        __CALC2(__reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 3, __reg_3_4);
        __reg_2_3 = __reg_1_3;
        __CALC3(__reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 2, __reg_3_5);
        __CALC3(__reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 1, __reg_3_6);
        __CALC3(__reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h + 0, __reg_3_7);
        __CALC3(__reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_8, __reg_2_3);
        __STORE(__h + 1, __reg_3_8);
      }
      else if (__h + 7 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 12, __reg_3_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 11, __reg_3_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 10, __reg_3_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h - 9, __reg_3_7);
        __reg_1_1 = __reg_0;
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h - 8, __reg_3_8);
        __reg_1_2 = __reg_0;
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h - 7, __reg_3_0);
        __reg_1_3 = __reg_0;
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __STORE(__h - 6, __reg_3_1);
        __reg_1_4 = __reg_0;
        __CALC2(__reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __STORE(__h - 5, __reg_3_2);
        __reg_2_1 = __reg_1_1;
        __CALC2(__reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __STORE(__h - 4, __reg_3_3);
        __reg_2_2 = __reg_1_2;
        __CALC2(__reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 3, __reg_3_4);
        __reg_2_3 = __reg_1_3;
        __CALC2(__reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 2, __reg_3_5);
        __reg_2_4 = __reg_1_4;
        __CALC3(__reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_5, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 1, __reg_3_6);
        __CALC3(__reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h + 0, __reg_3_7);
        __CALC3(__reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h + 1, __reg_3_8);
        __CALC3(__reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_0, __reg_2_4);
        __STORE(__h + 2, __reg_3_0);
      }
      else if (__h + 8 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 12, __reg_3_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 11, __reg_3_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 10, __reg_3_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h - 9, __reg_3_7);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h - 8, __reg_3_8);
        __reg_1_2 = __reg_0;
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h - 7, __reg_3_0);
        __reg_1_3 = __reg_0;
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __STORE(__h - 6, __reg_3_1);
        __reg_1_4 = __reg_0;
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __STORE(__h - 5, __reg_3_2);
        __reg_1_5 = __reg_0;
        __CALC2(__reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __STORE(__h - 4, __reg_3_3);
        __reg_2_2 = __reg_1_2;
        __CALC2(__reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 3, __reg_3_4);
        __reg_2_3 = __reg_1_3;
        __CALC2(__reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 2, __reg_3_5);
        __reg_2_4 = __reg_1_4;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 1, __reg_3_6);
        __reg_2_5 = __reg_1_5;
        __CALC3(__reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_6, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h + 0, __reg_3_7);
        __CALC3(__reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h + 1, __reg_3_8);
        __CALC3(__reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h + 2, __reg_3_0);
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_1, __reg_2_5);
        __STORE(__h + 3, __reg_3_1);
      }
      else if (__h + 9 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 12, __reg_3_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 11, __reg_3_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 10, __reg_3_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h - 9, __reg_3_7);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h - 8, __reg_3_8);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h - 7, __reg_3_0);
        __reg_1_3 = __reg_0;
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __STORE(__h - 6, __reg_3_1);
        __reg_1_4 = __reg_0;
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __STORE(__h - 5, __reg_3_2);
        __reg_1_5 = __reg_0;
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __STORE(__h - 4, __reg_3_3);
        __reg_1_6 = __reg_0;
        __CALC2(__reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 3, __reg_3_4);
        __reg_2_3 = __reg_1_3;
        __CALC2(__reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 2, __reg_3_5);
        __reg_2_4 = __reg_1_4;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 1, __reg_3_6);
        __reg_2_5 = __reg_1_5;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h + 0, __reg_3_7);
        __reg_2_6 = __reg_1_6;
        __CALC3(__reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_7, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h + 1, __reg_3_8);
        __CALC3(__reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h + 2, __reg_3_0);
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_2, __reg_3_1, __reg_2_5);
        __STORE(__h + 3, __reg_3_1);
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_2, __reg_2_6);
        __STORE(__h + 4, __reg_3_2);
      }
      else if (__h + 10 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 12, __reg_3_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 11, __reg_3_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 10, __reg_3_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h - 9, __reg_3_7);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h - 8, __reg_3_8);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h - 7, __reg_3_0);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __STORE(__h - 6, __reg_3_1);
        __reg_1_4 = __reg_0;
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __STORE(__h - 5, __reg_3_2);
        __reg_1_5 = __reg_0;
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __STORE(__h - 4, __reg_3_3);
        __reg_1_6 = __reg_0;
        __LOAD(__reg_0, __h + 9);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 3, __reg_3_4);
        __reg_1_7 = __reg_0;
        __CALC2(__reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 2, __reg_3_5);
        __reg_2_4 = __reg_1_4;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 1, __reg_3_6);
        __reg_2_5 = __reg_1_5;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h + 0, __reg_3_7);
        __reg_2_6 = __reg_1_6;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h + 1, __reg_3_8);
        __reg_2_7 = __reg_1_7;
        __CALC3(__reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_8, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h + 2, __reg_3_0);
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __STORE(__h + 3, __reg_3_1);
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_3, __reg_3_2, __reg_2_6);
        __STORE(__h + 4, __reg_3_2);
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_3, __reg_2_7);
        __STORE(__h + 5, __reg_3_3);
      }
      else if (__h + 11 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 12, __reg_3_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 11, __reg_3_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 10, __reg_3_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h - 9, __reg_3_7);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h - 8, __reg_3_8);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h - 7, __reg_3_0);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __STORE(__h - 6, __reg_3_1);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __STORE(__h - 5, __reg_3_2);
        __reg_1_5 = __reg_0;
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __STORE(__h - 4, __reg_3_3);
        __reg_1_6 = __reg_0;
        __LOAD(__reg_0, __h + 9);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 3, __reg_3_4);
        __reg_1_7 = __reg_0;
        __LOAD(__reg_0, __h + 10);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 2, __reg_3_5);
        __reg_1_8 = __reg_0;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 1, __reg_3_6);
        __reg_2_5 = __reg_1_5;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h + 0, __reg_3_7);
        __reg_2_6 = __reg_1_6;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h + 1, __reg_3_8);
        __reg_2_7 = __reg_1_7;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h + 2, __reg_3_0);
        __reg_2_8 = __reg_1_8;
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_0, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __STORE(__h + 3, __reg_3_1);
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __STORE(__h + 4, __reg_3_2);
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_4, __reg_3_3, __reg_2_7);
        __STORE(__h + 5, __reg_3_3);
        __CALC3(__reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_4, __reg_2_8);
        __STORE(__h + 6, __reg_3_4);
      }
      else if (__h + 12 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 12, __reg_3_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 11, __reg_3_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 10, __reg_3_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h - 9, __reg_3_7);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h - 8, __reg_3_8);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h - 7, __reg_3_0);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __STORE(__h - 6, __reg_3_1);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __STORE(__h - 5, __reg_3_2);
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __STORE(__h - 4, __reg_3_3);
        __reg_1_6 = __reg_0;
        __LOAD(__reg_0, __h + 9);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 3, __reg_3_4);
        __reg_1_7 = __reg_0;
        __LOAD(__reg_0, __h + 10);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 2, __reg_3_5);
        __reg_1_8 = __reg_0;
        __LOAD(__reg_0, __h + 11);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 1, __reg_3_6);
        __reg_1_0 = __reg_0;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h + 0, __reg_3_7);
        __reg_2_6 = __reg_1_6;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h + 1, __reg_3_8);
        __reg_2_7 = __reg_1_7;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h + 2, __reg_3_0);
        __reg_2_8 = __reg_1_8;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __STORE(__h + 3, __reg_3_1);
        __reg_2_0 = __reg_1_0;
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_1, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __STORE(__h + 4, __reg_3_2);
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_2, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __STORE(__h + 5, __reg_3_3);
        __CALC3(__reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_3, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h + 6, __reg_3_4);
        __CALC3(__reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_4, __reg_3_5, __reg_2_0);
        __STORE(__h + 7, __reg_3_5);
      }
    }
    else
    {
      for (__h = 25; __h <= __side1LenOl - 9;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
        __STORE(__h - 12, __reg_3_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
        __STORE(__h - 12, __reg_3_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
        __STORE(__h - 12, __reg_3_6);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
        __STORE(__h - 12, __reg_3_7);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
        __STORE(__h - 12, __reg_3_8);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
        __STORE(__h - 12, __reg_3_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
        __STORE(__h - 12, __reg_3_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
        __STORE(__h - 12, __reg_3_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
        __STORE(__h - 12, __reg_3_3);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
      __CALC3(__reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_2_8);
      __STORE(__h - 12, __reg_3_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
      __CALC3(__reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_2_0);
      __STORE(__h - 12, __reg_3_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
      __CALC3(__reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_2_1);
      __STORE(__h - 12, __reg_3_6);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
      __CALC3(__reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_2_2);
      __STORE(__h - 12, __reg_3_7);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
      __CALC3(__reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_2_3);
      __STORE(__h - 12, __reg_3_8);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
      __CALC3(__reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_3_0, __reg_2_4);
      __STORE(__h - 12, __reg_3_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_3_1, __reg_2_5);
      __STORE(__h - 12, __reg_3_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_3_2, __reg_2_6);
      __STORE(__h - 12, __reg_3_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_3_8, __reg_3_7, __reg_3_6, __reg_3_5, __reg_3_4, __reg_3_3, __reg_2_7);
      __STORE(__h - 12, __reg_3_3);
      __h++;
    }
}
__global__ void kernel0_2(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 4 - 4);
    const AN5D_TYPE __c1Pad = (4);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 4 - 4);
    const AN5D_TYPE __c2Pad = (4);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 4;
    const AN5D_TYPE __halo2 = 4;
    const AN5D_TYPE __side0Len = 2;
    const AN5D_TYPE __side1Len = 512;
    const AN5D_TYPE __side2Len = 496;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_1_5;
    float __reg_1_6;
    float __reg_1_7;
    float __reg_1_8;
    float __reg_2_0;
    float __reg_2_1;
    float __reg_2_2;
    float __reg_2_3;
    float __reg_2_4;
    float __reg_2_5;
    float __reg_2_6;
    float __reg_2_7;
    float __reg_2_8;
    __shared__ float __a_sb_double[__blockSize * 2];
    float *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __storeValid = __writeValid2;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00930f * (__SBREF(__a_sb, -4))) + (0.00931f * (__SBREF(__a_sb, -3)))) + (0.00932f * (__SBREF(__a_sb, -2)))) + (0.00933f * (__SBREF(__a_sb, -1)))) + (0.00934f * (__REGREF(__a, 0)))) + (0.00935f * (__SBREF(__a_sb, 1)))) + (0.00936f * (__SBREF(__a_sb, 2)))) + (0.00937f * (__SBREF(__a_sb, 3)))) + (0.00938f * (__SBREF(__a_sb, 4)))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00939f * (__SBREF(__a_sb, -4)))) + (0.00940f * (__SBREF(__a_sb, -3)))) + (0.00941f * (__SBREF(__a_sb, -2)))) + (0.00942f * (__SBREF(__a_sb, -1)))) + (0.00943f * (__REGREF(__a, 0)))) + (0.00944f * (__SBREF(__a_sb, 1)))) + (0.00945f * (__SBREF(__a_sb, 2)))) + (0.00946f * (__SBREF(__a_sb, 3)))) + (0.00947f * (__SBREF(__a_sb, 4))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { float etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00948f * (__SBREF(__a_sb, -4)))) + (0.00949f * (__SBREF(__a_sb, -3)))) + (0.00950f * (__SBREF(__a_sb, -2)))) + (0.00951f * (__SBREF(__a_sb, -1)))) + (0.00952f * (__REGREF(__a, 0)))) + (0.00953f * (__SBREF(__a_sb, 1)))) + (0.00954f * (__SBREF(__a_sb, 2)))) + (0.00955f * (__SBREF(__a_sb, 3)))) + (0.00956f * (__SBREF(__a_sb, 4)))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { float etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_3_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00957f * (__SBREF(__a_sb, -4)))) + (0.00958f * (__SBREF(__a_sb, -3)))) + (0.00959f * (__SBREF(__a_sb, -2)))) + (0.00960f * (__SBREF(__a_sb, -1)))) + (0.00961f * (__REGREF(__a, 0)))) + (0.00962f * (__SBREF(__a_sb, 1)))) + (0.00963f * (__SBREF(__a_sb, 2)))) + (0.00964f * (__SBREF(__a_sb, 3)))) + (0.00965f * (__SBREF(__a_sb, 4))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_3(out, a) do { float etmp; __CALCEXPR_3_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_4_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((0.00966f * (__SBREF(__a_sb, -4)))) + (0.00967f * (__SBREF(__a_sb, -3)))) + (0.00968f * (__SBREF(__a_sb, -2)))) + (0.00969f * (__SBREF(__a_sb, -1)))) + (0.22400f * (__REGREF(__a, 0)))) + (0.00971f * (__SBREF(__a_sb, 1)))) + (0.00972f * (__SBREF(__a_sb, 2)))) + (0.00973f * (__SBREF(__a_sb, 3)))) + (0.00974f * (__SBREF(__a_sb, 4)))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_4(out, a) do { float etmp; __CALCEXPR_4_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_5_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((0.00975f * (__SBREF(__a_sb, -4)))) + (0.00976f * (__SBREF(__a_sb, -3)))) + (0.00977f * (__SBREF(__a_sb, -2)))) + (0.00978f * (__SBREF(__a_sb, -1)))) + (0.00979f * (__REGREF(__a, 0)))) + (0.00980f * (__SBREF(__a_sb, 1)))) + (0.00981f * (__SBREF(__a_sb, 2)))) + (0.00982f * (__SBREF(__a_sb, 3)))) + (0.00983f * (__SBREF(__a_sb, 4))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_5(out, a) do { float etmp; __CALCEXPR_5_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_6_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((0.00984f * (__SBREF(__a_sb, -4)))) + (0.00985f * (__SBREF(__a_sb, -3)))) + (0.00986f * (__SBREF(__a_sb, -2)))) + (0.00987f * (__SBREF(__a_sb, -1)))) + (0.00988f * (__REGREF(__a, 0)))) + (0.00989f * (__SBREF(__a_sb, 1)))) + (0.00990f * (__SBREF(__a_sb, 2)))) + (0.00991f * (__SBREF(__a_sb, 3)))) + (0.00992f * (__SBREF(__a_sb, 4)))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_6(out, a) do { float etmp; __CALCEXPR_6_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_7_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((0.00993f * (__SBREF(__a_sb, -4)))) + (0.00994f * (__SBREF(__a_sb, -3)))) + (0.00995f * (__SBREF(__a_sb, -2)))) + (0.00996f * (__SBREF(__a_sb, -1)))) + (0.00997f * (__REGREF(__a, 0)))) + (0.00998f * (__SBREF(__a_sb, 1)))) + (0.00999f * (__SBREF(__a_sb, 2)))) + (0.01000f * (__SBREF(__a_sb, 3)))) + (0.01001f * (__SBREF(__a_sb, 4))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_7(out, a) do { float etmp; __CALCEXPR_7_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_8_wrap(__rn0, __a) do { __rn0 = ((((((((((0.01002f * (__SBREF(__a_sb, -4)))) + (0.01003f * (__SBREF(__a_sb, -3)))) + (0.01004f * (__SBREF(__a_sb, -2)))) + (0.01005f * (__SBREF(__a_sb, -1)))) + (0.01006f * (__REGREF(__a, 0)))) + (0.01007f * (__SBREF(__a_sb, 1)))) + (0.01008f * (__SBREF(__a_sb, 2)))) + (0.01009f * (__SBREF(__a_sb, 3)))) + (0.01010f * (__SBREF(__a_sb, 4)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_8(out, a) do { float etmp; __CALCEXPR_8_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); __CALCEXPR_3(out3, reg); __CALCEXPR_4(out4, reg); __CALCEXPR_5(out5, reg); __CALCEXPR_6(out6, reg); __CALCEXPR_7(out7, reg); __CALCEXPR_8(out8, reg); } while (0);
    #define __CALC1(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg); } else out4 = reg; } while (0)
    #define __CALC2(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCSETUP(reg); if (__writeValid2) { __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg); } else out4 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
      __LOAD(__reg_0, 9);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
      __LOAD(__reg_0, 10);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
      __LOAD(__reg_0, 11);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
      __LOAD(__reg_0, 12);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
      __STORE(4, __reg_2_4);
      __LOAD(__reg_0, 13);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
      __STORE(5, __reg_2_5);
      __LOAD(__reg_0, 14);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
      __STORE(6, __reg_2_6);
      __LOAD(__reg_0, 15);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
      __STORE(7, __reg_2_7);
      __LOAD(__reg_0, 16);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
      __STORE(8, __reg_2_8);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
      __LOAD(__reg_0, 9);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
      __LOAD(__reg_0, 10);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
      __LOAD(__reg_0, 11);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
      __LOAD(__reg_0, 12);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
      __LOAD(__reg_0, 13);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
      __LOAD(__reg_0, 14);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
      __LOAD(__reg_0, 15);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
      __LOAD(__reg_0, 16);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
      __STORE(8, __reg_2_8);
    }
    __a_sb = __a_sb_double + __blockSize * 0;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 17; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 13;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h - 8, __reg_2_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h - 8, __reg_2_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __STORE(__h - 8, __reg_2_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __STORE(__h - 8, __reg_2_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __STORE(__h - 8, __reg_2_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __STORE(__h - 8, __reg_2_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __STORE(__h - 8, __reg_2_6);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __STORE(__h - 8, __reg_2_7);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __STORE(__h - 8, __reg_2_8);
        __h++;
      }
      if (0) {}
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h - 8, __reg_2_0);
        __reg_1_8 = __reg_0;
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h - 7, __reg_2_1);
        __reg_1_0 = __reg_0;
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __STORE(__h - 6, __reg_2_2);
        __reg_1_1 = __reg_0;
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __STORE(__h - 5, __reg_2_3);
        __reg_1_2 = __reg_0;
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __STORE(__h - 4, __reg_2_4);
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __STORE(__h - 3, __reg_2_5);
        __CALC2(__reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_7, __reg_2_6, __reg_1_1);
        __STORE(__h - 2, __reg_2_6);
        __CALC2(__reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_7, __reg_1_2);
        __STORE(__h - 1, __reg_2_7);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h - 8, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h - 7, __reg_2_1);
        __reg_1_0 = __reg_0;
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __STORE(__h - 6, __reg_2_2);
        __reg_1_1 = __reg_0;
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __STORE(__h - 5, __reg_2_3);
        __reg_1_2 = __reg_0;
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __STORE(__h - 4, __reg_2_4);
        __reg_1_3 = __reg_0;
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __STORE(__h - 3, __reg_2_5);
        __CALC2(__reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __STORE(__h - 2, __reg_2_6);
        __CALC2(__reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_8, __reg_2_7, __reg_1_2);
        __STORE(__h - 1, __reg_2_7);
        __CALC2(__reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_8, __reg_1_3);
        __STORE(__h + 0, __reg_2_8);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h - 8, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h - 7, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __STORE(__h - 6, __reg_2_2);
        __reg_1_1 = __reg_0;
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __STORE(__h - 5, __reg_2_3);
        __reg_1_2 = __reg_0;
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __STORE(__h - 4, __reg_2_4);
        __reg_1_3 = __reg_0;
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __STORE(__h - 3, __reg_2_5);
        __reg_1_4 = __reg_0;
        __CALC2(__reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __STORE(__h - 2, __reg_2_6);
        __CALC2(__reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __STORE(__h - 1, __reg_2_7);
        __CALC2(__reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_0, __reg_2_8, __reg_1_3);
        __STORE(__h + 0, __reg_2_8);
        __CALC2(__reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_0, __reg_1_4);
        __STORE(__h + 1, __reg_2_0);
      }
      else if (__h + 7 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h - 8, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h - 7, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __STORE(__h - 6, __reg_2_2);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __STORE(__h - 5, __reg_2_3);
        __reg_1_2 = __reg_0;
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __STORE(__h - 4, __reg_2_4);
        __reg_1_3 = __reg_0;
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __STORE(__h - 3, __reg_2_5);
        __reg_1_4 = __reg_0;
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __STORE(__h - 2, __reg_2_6);
        __reg_1_5 = __reg_0;
        __CALC2(__reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_6, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __STORE(__h - 1, __reg_2_7);
        __CALC2(__reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __STORE(__h + 0, __reg_2_8);
        __CALC2(__reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h + 1, __reg_2_0);
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_1, __reg_1_5);
        __STORE(__h + 2, __reg_2_1);
      }
      else if (__h + 8 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h - 8, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h - 7, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __STORE(__h - 6, __reg_2_2);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __STORE(__h - 5, __reg_2_3);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __STORE(__h - 4, __reg_2_4);
        __reg_1_3 = __reg_0;
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __STORE(__h - 3, __reg_2_5);
        __reg_1_4 = __reg_0;
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __STORE(__h - 2, __reg_2_6);
        __reg_1_5 = __reg_0;
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __STORE(__h - 1, __reg_2_7);
        __reg_1_6 = __reg_0;
        __CALC2(__reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_7, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __STORE(__h + 0, __reg_2_8);
        __CALC2(__reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h + 1, __reg_2_0);
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h + 2, __reg_2_1);
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_2, __reg_1_6);
        __STORE(__h + 3, __reg_2_2);
      }
      else if (__h + 9 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h - 8, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h - 7, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __STORE(__h - 6, __reg_2_2);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __STORE(__h - 5, __reg_2_3);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __STORE(__h - 4, __reg_2_4);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __STORE(__h - 3, __reg_2_5);
        __reg_1_4 = __reg_0;
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __STORE(__h - 2, __reg_2_6);
        __reg_1_5 = __reg_0;
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __STORE(__h - 1, __reg_2_7);
        __reg_1_6 = __reg_0;
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __STORE(__h + 0, __reg_2_8);
        __reg_1_7 = __reg_0;
        __CALC2(__reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_8, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h + 1, __reg_2_0);
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h + 2, __reg_2_1);
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_3, __reg_2_2, __reg_1_6);
        __STORE(__h + 3, __reg_2_2);
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_3, __reg_1_7);
        __STORE(__h + 4, __reg_2_3);
      }
      else if (__h + 10 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h - 8, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h - 7, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __STORE(__h - 6, __reg_2_2);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __STORE(__h - 5, __reg_2_3);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __STORE(__h - 4, __reg_2_4);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __STORE(__h - 3, __reg_2_5);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __STORE(__h - 2, __reg_2_6);
        __reg_1_5 = __reg_0;
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __STORE(__h - 1, __reg_2_7);
        __reg_1_6 = __reg_0;
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __STORE(__h + 0, __reg_2_8);
        __reg_1_7 = __reg_0;
        __LOAD(__reg_0, __h + 9);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h + 1, __reg_2_0);
        __reg_1_8 = __reg_0;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_0, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h + 2, __reg_2_1);
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __STORE(__h + 3, __reg_2_2);
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_4, __reg_2_3, __reg_1_7);
        __STORE(__h + 4, __reg_2_3);
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_4, __reg_1_8);
        __STORE(__h + 5, __reg_2_4);
      }
      else if (__h + 11 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h - 8, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h - 7, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __STORE(__h - 6, __reg_2_2);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __STORE(__h - 5, __reg_2_3);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __STORE(__h - 4, __reg_2_4);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __STORE(__h - 3, __reg_2_5);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __STORE(__h - 2, __reg_2_6);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __STORE(__h - 1, __reg_2_7);
        __reg_1_6 = __reg_0;
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __STORE(__h + 0, __reg_2_8);
        __reg_1_7 = __reg_0;
        __LOAD(__reg_0, __h + 9);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h + 1, __reg_2_0);
        __reg_1_8 = __reg_0;
        __LOAD(__reg_0, __h + 10);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h + 2, __reg_2_1);
        __reg_1_0 = __reg_0;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_1, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __STORE(__h + 3, __reg_2_2);
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __STORE(__h + 4, __reg_2_3);
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_5, __reg_2_4, __reg_1_8);
        __STORE(__h + 5, __reg_2_4);
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_5, __reg_1_0);
        __STORE(__h + 6, __reg_2_5);
      }
      else if (__h + 12 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h - 8, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h - 7, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __STORE(__h - 6, __reg_2_2);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __STORE(__h - 5, __reg_2_3);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __STORE(__h - 4, __reg_2_4);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __STORE(__h - 3, __reg_2_5);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __STORE(__h - 2, __reg_2_6);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __STORE(__h - 1, __reg_2_7);
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __STORE(__h + 0, __reg_2_8);
        __reg_1_7 = __reg_0;
        __LOAD(__reg_0, __h + 9);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h + 1, __reg_2_0);
        __reg_1_8 = __reg_0;
        __LOAD(__reg_0, __h + 10);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h + 2, __reg_2_1);
        __reg_1_0 = __reg_0;
        __LOAD(__reg_0, __h + 11);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __STORE(__h + 3, __reg_2_2);
        __reg_1_1 = __reg_0;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_2, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __STORE(__h + 4, __reg_2_3);
        __CALC2(__reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_3, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __STORE(__h + 5, __reg_2_4);
        __CALC2(__reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_4, __reg_2_6, __reg_2_5, __reg_1_0);
        __STORE(__h + 6, __reg_2_5);
        __CALC2(__reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_5, __reg_2_6, __reg_1_1);
        __STORE(__h + 7, __reg_2_6);
      }
    }
    else
    {
      for (__h = 17; __h <= __side1LenOl - 9;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
        __STORE(__h - 8, __reg_2_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
        __STORE(__h - 8, __reg_2_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
        __STORE(__h - 8, __reg_2_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
        __STORE(__h - 8, __reg_2_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
        __STORE(__h - 8, __reg_2_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
        __STORE(__h - 8, __reg_2_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
        __STORE(__h - 8, __reg_2_6);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
        __STORE(__h - 8, __reg_2_7);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
        __STORE(__h - 8, __reg_2_8);
        __h++;
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __CALC2(__reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_1_4);
      __STORE(__h - 8, __reg_2_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __CALC2(__reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_1_5);
      __STORE(__h - 8, __reg_2_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_1_6);
      __STORE(__h - 8, __reg_2_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_1_7);
      __STORE(__h - 8, __reg_2_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __CALC2(__reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_1_8);
      __STORE(__h - 8, __reg_2_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_2_5, __reg_1_0);
      __STORE(__h - 8, __reg_2_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_2_6, __reg_1_1);
      __STORE(__h - 8, __reg_2_6);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __CALC2(__reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_2_7, __reg_1_2);
      __STORE(__h - 8, __reg_2_7);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __CALC2(__reg_2_7, __reg_2_6, __reg_2_5, __reg_2_4, __reg_2_3, __reg_2_2, __reg_2_1, __reg_2_0, __reg_2_8, __reg_1_3);
      __STORE(__h - 8, __reg_2_8);
      __h++;
    }
}
__global__ void kernel0_1(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 4 - 4);
    const AN5D_TYPE __c1Pad = (4);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 4 - 4);
    const AN5D_TYPE __c2Pad = (4);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 4;
    const AN5D_TYPE __halo2 = 4;
    const AN5D_TYPE __side0Len = 1;
    const AN5D_TYPE __side1Len = 512;
    const AN5D_TYPE __side2Len = 504;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    float __reg_0;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_1_5;
    float __reg_1_6;
    float __reg_1_7;
    float __reg_1_8;
    __shared__ float __a_sb_double[__blockSize * 2];
    float *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __storeValid = __writeValid1;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00930f * (__SBREF(__a_sb, -4))) + (0.00931f * (__SBREF(__a_sb, -3)))) + (0.00932f * (__SBREF(__a_sb, -2)))) + (0.00933f * (__SBREF(__a_sb, -1)))) + (0.00934f * (__REGREF(__a, 0)))) + (0.00935f * (__SBREF(__a_sb, 1)))) + (0.00936f * (__SBREF(__a_sb, 2)))) + (0.00937f * (__SBREF(__a_sb, 3)))) + (0.00938f * (__SBREF(__a_sb, 4)))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00939f * (__SBREF(__a_sb, -4)))) + (0.00940f * (__SBREF(__a_sb, -3)))) + (0.00941f * (__SBREF(__a_sb, -2)))) + (0.00942f * (__SBREF(__a_sb, -1)))) + (0.00943f * (__REGREF(__a, 0)))) + (0.00944f * (__SBREF(__a_sb, 1)))) + (0.00945f * (__SBREF(__a_sb, 2)))) + (0.00946f * (__SBREF(__a_sb, 3)))) + (0.00947f * (__SBREF(__a_sb, 4))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { float etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00948f * (__SBREF(__a_sb, -4)))) + (0.00949f * (__SBREF(__a_sb, -3)))) + (0.00950f * (__SBREF(__a_sb, -2)))) + (0.00951f * (__SBREF(__a_sb, -1)))) + (0.00952f * (__REGREF(__a, 0)))) + (0.00953f * (__SBREF(__a_sb, 1)))) + (0.00954f * (__SBREF(__a_sb, 2)))) + (0.00955f * (__SBREF(__a_sb, 3)))) + (0.00956f * (__SBREF(__a_sb, 4)))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { float etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_3_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00957f * (__SBREF(__a_sb, -4)))) + (0.00958f * (__SBREF(__a_sb, -3)))) + (0.00959f * (__SBREF(__a_sb, -2)))) + (0.00960f * (__SBREF(__a_sb, -1)))) + (0.00961f * (__REGREF(__a, 0)))) + (0.00962f * (__SBREF(__a_sb, 1)))) + (0.00963f * (__SBREF(__a_sb, 2)))) + (0.00964f * (__SBREF(__a_sb, 3)))) + (0.00965f * (__SBREF(__a_sb, 4))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_3(out, a) do { float etmp; __CALCEXPR_3_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_4_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((0.00966f * (__SBREF(__a_sb, -4)))) + (0.00967f * (__SBREF(__a_sb, -3)))) + (0.00968f * (__SBREF(__a_sb, -2)))) + (0.00969f * (__SBREF(__a_sb, -1)))) + (0.22400f * (__REGREF(__a, 0)))) + (0.00971f * (__SBREF(__a_sb, 1)))) + (0.00972f * (__SBREF(__a_sb, 2)))) + (0.00973f * (__SBREF(__a_sb, 3)))) + (0.00974f * (__SBREF(__a_sb, 4)))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_4(out, a) do { float etmp; __CALCEXPR_4_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_5_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((0.00975f * (__SBREF(__a_sb, -4)))) + (0.00976f * (__SBREF(__a_sb, -3)))) + (0.00977f * (__SBREF(__a_sb, -2)))) + (0.00978f * (__SBREF(__a_sb, -1)))) + (0.00979f * (__REGREF(__a, 0)))) + (0.00980f * (__SBREF(__a_sb, 1)))) + (0.00981f * (__SBREF(__a_sb, 2)))) + (0.00982f * (__SBREF(__a_sb, 3)))) + (0.00983f * (__SBREF(__a_sb, 4))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_5(out, a) do { float etmp; __CALCEXPR_5_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_6_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((0.00984f * (__SBREF(__a_sb, -4)))) + (0.00985f * (__SBREF(__a_sb, -3)))) + (0.00986f * (__SBREF(__a_sb, -2)))) + (0.00987f * (__SBREF(__a_sb, -1)))) + (0.00988f * (__REGREF(__a, 0)))) + (0.00989f * (__SBREF(__a_sb, 1)))) + (0.00990f * (__SBREF(__a_sb, 2)))) + (0.00991f * (__SBREF(__a_sb, 3)))) + (0.00992f * (__SBREF(__a_sb, 4)))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_6(out, a) do { float etmp; __CALCEXPR_6_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_7_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((0.00993f * (__SBREF(__a_sb, -4)))) + (0.00994f * (__SBREF(__a_sb, -3)))) + (0.00995f * (__SBREF(__a_sb, -2)))) + (0.00996f * (__SBREF(__a_sb, -1)))) + (0.00997f * (__REGREF(__a, 0)))) + (0.00998f * (__SBREF(__a_sb, 1)))) + (0.00999f * (__SBREF(__a_sb, 2)))) + (0.01000f * (__SBREF(__a_sb, 3)))) + (0.01001f * (__SBREF(__a_sb, 4))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_7(out, a) do { float etmp; __CALCEXPR_7_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_8_wrap(__rn0, __a) do { __rn0 = ((((((((((0.01002f * (__SBREF(__a_sb, -4)))) + (0.01003f * (__SBREF(__a_sb, -3)))) + (0.01004f * (__SBREF(__a_sb, -2)))) + (0.01005f * (__SBREF(__a_sb, -1)))) + (0.01006f * (__REGREF(__a, 0)))) + (0.01007f * (__SBREF(__a_sb, 1)))) + (0.01008f * (__SBREF(__a_sb, 2)))) + (0.01009f * (__SBREF(__a_sb, 3)))) + (0.01010f * (__SBREF(__a_sb, 4)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_8(out, a) do { float etmp; __CALCEXPR_8_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); __CALCEXPR_3(out3, reg); __CALCEXPR_4(out4, reg); __CALCEXPR_5(out5, reg); __CALCEXPR_6(out6, reg); __CALCEXPR_7(out7, reg); __CALCEXPR_8(out8, reg); } while (0);
    #define __CALC1(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg); } else out4 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __STORE(4, __reg_1_4);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __STORE(4, __reg_1_4);
    }
    __a_sb = __a_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 9; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 13;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 4, __reg_1_6);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 4, __reg_1_7);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 4, __reg_1_8);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 4, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 4, __reg_1_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h - 4, __reg_1_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 4, __reg_1_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 4, __reg_1_4);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
      }
      else if (__h + 7 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
      }
      else if (__h + 8 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
      }
      else if (__h + 9 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_4, __reg_0);
        __STORE(__h + 4, __reg_1_4);
      }
      else if (__h + 10 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h + 4, __reg_1_4);
        __LOAD(__reg_0, __h + 9);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_5, __reg_0);
        __STORE(__h + 5, __reg_1_5);
      }
      else if (__h + 11 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h + 4, __reg_1_4);
        __LOAD(__reg_0, __h + 9);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h + 5, __reg_1_5);
        __LOAD(__reg_0, __h + 10);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_6, __reg_0);
        __STORE(__h + 6, __reg_1_6);
      }
      else if (__h + 12 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h + 4, __reg_1_4);
        __LOAD(__reg_0, __h + 9);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h + 5, __reg_1_5);
        __LOAD(__reg_0, __h + 10);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h + 6, __reg_1_6);
        __LOAD(__reg_0, __h + 11);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_7, __reg_0);
        __STORE(__h + 7, __reg_1_7);
      }
    }
    else
    {
      for (__h = 9; __h <= __side1LenOl - 9;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 4, __reg_1_6);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 4, __reg_1_7);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 4, __reg_1_8);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 4, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 4, __reg_1_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h - 4, __reg_1_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 4, __reg_1_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 4, __reg_1_4);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __STORE(__h - 4, __reg_1_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __STORE(__h - 4, __reg_1_6);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __STORE(__h - 4, __reg_1_7);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __STORE(__h - 4, __reg_1_8);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __STORE(__h - 4, __reg_1_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __STORE(__h - 4, __reg_1_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __STORE(__h - 4, __reg_1_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __STORE(__h - 4, __reg_1_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __STORE(__h - 4, __reg_1_4);
      __h++;
    }
}
