#include "hip/hip_runtime.h"
#include "star3d3r-32x32-2-128_kernel.hu"
__device__ float __sbref_wrap(float *sb, size_t index) { return sb[index]; }

__global__ void kernel0_2(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c1Pad = (3);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c2Pad = (3);
    #define __c2 c2
    const AN5D_TYPE __c3Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c3Pad = (3);
    #define __c3 c3
    const AN5D_TYPE __halo1 = 3;
    const AN5D_TYPE __halo2 = 3;
    const AN5D_TYPE __halo3 = 3;
    const AN5D_TYPE __side0Len = 2;
    const AN5D_TYPE __side1Len = 128;
    const AN5D_TYPE __side2Len = 20;
    const AN5D_TYPE __side3Len = 20;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __OlLen3 = (__halo3 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __side3LenOl = (__side3Len + 2 * __OlLen3);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl * __side3LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __side3Num = (__c3Len + __side3Len - 1) / __side3Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid / __side3LenOl;
    const AN5D_TYPE __local_c3 = __tid % __side3LenOl;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num / __side3Num;
    const AN5D_TYPE __c2 = (blockIdx.x / __side3Num % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    const AN5D_TYPE __c3 = (blockIdx.x % __side3Num) * __side3Len + __local_c3 + __c3Pad - __OlLen3;
    float __reg_0_0;
    float __reg_0_1;
    float __reg_0_2;
    float __reg_0_3;
    float __reg_0_4;
    float __reg_0_5;
    float __reg_0_6;
    float __reg_1_0;
    float __reg_1_1;
    float __reg_1_2;
    float __reg_1_3;
    float __reg_1_4;
    float __reg_1_5;
    float __reg_1_6;
    __shared__ float __d_sb_double[__blockSize * 2];
    float *__d_sb = __d_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2 && __c3 >= __c3Pad - __halo3 && __c3 < __c3Pad + __c3Len + __halo3;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len && __c3 >= __c3Pad && __c3 < __c3Pad + __c3Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1) && __local_c3 >= (__halo3 * 1) && __local_c3 < __side3LenOl - (__halo3 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2) && __local_c3 >= (__halo3 * 2) && __local_c3 < __side3LenOl - (__halo3 * 2);
    const AN5D_TYPE __storeValid = __writeValid2;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[(((__c0 % 2) * dimsize + __c1) * dimsize + __c2) * dimsize + __c3]; }} while (0)
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR(__rn0, __a, __b, __c, __d, __e, __f, __g) do { __rn0 = (((((((((((((((((((0.25000f * (__REGREF(__d, 0, 0))) + (0.04276f * (__SBREF(__d_sb, 0, -3)))) + (0.04176f * (__SBREF(__d_sb, 0, -2)))) + (0.04076f * (__SBREF(__d_sb, 0, -1)))) + (0.04046f * (__SBREF(__d_sb, 0, 1)))) + (0.04146f * (__SBREF(__d_sb, 0, 2)))) + (0.04246f * (__SBREF(__d_sb, 0, 3)))) + (0.04096f * (__REGREF(__c, 0, 0)))) + (0.04066f * (__REGREF(__e, 0, 0)))) + (0.04086f * (__SBREF(__d_sb, -1, 0)))) + (0.04056f * (__SBREF(__d_sb, 1, 0)))) + (0.04196f * (__REGREF(__b, 0, 0)))) + (0.04166f * (__REGREF(__f, 0, 0)))) + (0.04186f * (__SBREF(__d_sb, -2, 0)))) + (0.04156f * (__SBREF(__d_sb, 2, 0)))) + (0.04296f * (__REGREF(__a, 0, 0)))) + (0.04266f * (__REGREF(__g, 0, 0)))) + (0.04286f * (__SBREF(__d_sb, -3, 0)))) + (0.04256f * (__SBREF(__d_sb, 3, 0)))); } while (0)
    #define __DB_SWITCH() do { __d_sb = &__d_sb_double[(__d_sb == __d_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a, b, c, d, e, f, g) do { __DB_SWITCH(); __d_sb[__tid] = d; __syncthreads(); } while (0)
    #define __CALC1(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__writeValid1) __CALCEXPR(out, reg0, reg1, reg2, reg3, reg4, reg5, reg6); else out = reg3; } while (0)
    #define __STORE(h, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __CALCEXPR(__DEST, reg0, reg1, reg2, reg3, reg4, reg5, reg6); } } while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_1_0, 0);
      __LOAD(__reg_1_1, 1);
      __LOAD(__reg_1_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __CALC1(__reg_1_3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __LOAD(__reg_0_0, 7);
      __CALC1(__reg_1_4, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __LOAD(__reg_0_1, 8);
      __CALC1(__reg_1_5, __reg_1_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __LOAD(__reg_0_2, 9);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __STORE(3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __LOAD(__reg_0_3, 10);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __STORE(4, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __LOAD(__reg_0_4, 11);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __STORE(5, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __LOAD(__reg_0_5, 12);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __STORE(6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
    }
    else
    {
      __LOAD(__reg_0_0, 0);
      __LOAD(__reg_0_1, 1);
      __LOAD(__reg_0_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __LOAD(__reg_0_0, 7);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __LOAD(__reg_0_1, 8);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __LOAD(__reg_0_2, 9);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __LOAD(__reg_0_3, 10);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __LOAD(__reg_0_4, 11);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __LOAD(__reg_0_5, 12);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __STORE(6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __DB_SWITCH(); __syncthreads();
    }
    __d_sb = __d_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 13; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 7;)
      {
        __LOAD(__reg_0_6, __h);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __h++;
        __LOAD(__reg_0_0, __h);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 6, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __h++;
        __LOAD(__reg_0_1, __h);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 6, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __h++;
        __LOAD(__reg_0_2, __h);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __h++;
        __LOAD(__reg_0_3, __h);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 6, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __h++;
        __LOAD(__reg_0_4, __h);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 6, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __h++;
        __LOAD(__reg_0_5, __h);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __h++;
      }
      if (0) {}
      else if (__h + 0 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3);
        __STORE(__h - 5, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 4, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_0_3, __reg_0_4, __reg_0_5);
      }
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_6, __h + 0);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __STORE(__h - 5, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4);
        __STORE(__h - 4, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_0_4, __reg_0_5, __reg_0_6);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_6, __h + 0);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __LOAD(__reg_0_0, __h + 1);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 5, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __STORE(__h - 4, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5);
        __STORE(__h - 3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 2, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_0_5, __reg_0_6, __reg_0_0);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_6, __h + 0);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __LOAD(__reg_0_0, __h + 1);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 5, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __LOAD(__reg_0_1, __h + 2);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 4, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __STORE(__h - 3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6);
        __STORE(__h - 2, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_0_6, __reg_0_0, __reg_0_1);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_6, __h + 0);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __LOAD(__reg_0_0, __h + 1);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 5, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __LOAD(__reg_0_1, __h + 2);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 4, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __LOAD(__reg_0_2, __h + 3);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __STORE(__h - 2, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0);
        __STORE(__h - 1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0, __reg_0_1);
        __STORE(__h + 0, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_0_0, __reg_0_1, __reg_0_2);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_6, __h + 0);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __LOAD(__reg_0_0, __h + 1);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 5, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __LOAD(__reg_0_1, __h + 2);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 4, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __LOAD(__reg_0_2, __h + 3);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __LOAD(__reg_0_3, __h + 4);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 2, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __STORE(__h - 1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1);
        __STORE(__h + 0, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1, __reg_0_2);
        __STORE(__h + 1, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_0_1, __reg_0_2, __reg_0_3);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_6, __h + 0);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __LOAD(__reg_0_0, __h + 1);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 5, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __LOAD(__reg_0_1, __h + 2);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 4, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __LOAD(__reg_0_2, __h + 3);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 3, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __LOAD(__reg_0_3, __h + 4);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 2, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __LOAD(__reg_0_4, __h + 5);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __STORE(__h + 0, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2);
        __STORE(__h + 1, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2, __reg_0_3);
        __STORE(__h + 2, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_0_2, __reg_0_3, __reg_0_4);
      }
    }
    else
    {
      for (__h = 13; __h <= __side1LenOl - 7;)
      {
        __LOAD(__reg_0_6, __h);
        __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
        __h++;
        __LOAD(__reg_0_0, __h);
        __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __STORE(__h - 6, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
        __h++;
        __LOAD(__reg_0_1, __h);
        __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __STORE(__h - 6, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
        __h++;
        __LOAD(__reg_0_2, __h);
        __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __STORE(__h - 6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
        __h++;
        __LOAD(__reg_0_3, __h);
        __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __STORE(__h - 6, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
        __h++;
        __LOAD(__reg_0_4, __h);
        __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __STORE(__h - 6, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
        __h++;
        __LOAD(__reg_0_5, __h);
        __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __STORE(__h - 6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
        __h++;
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_6, __h);
      __CALC1(__reg_1_3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __STORE(__h - 6, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_0, __h);
      __CALC1(__reg_1_4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __STORE(__h - 6, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_1, __h);
      __CALC1(__reg_1_5, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __STORE(__h - 6, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_2, __h);
      __CALC1(__reg_1_6, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __STORE(__h - 6, __reg_1_0, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_3, __h);
      __CALC1(__reg_1_0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __STORE(__h - 6, __reg_1_1, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_4, __h);
      __CALC1(__reg_1_1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __STORE(__h - 6, __reg_1_2, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_5, __h);
      __CALC1(__reg_1_2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __STORE(__h - 6, __reg_1_3, __reg_1_4, __reg_1_5, __reg_1_6, __reg_1_0, __reg_1_1, __reg_1_2);
      __h++;
    }
}
__global__ void kernel0_1(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c1Pad = (3);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c2Pad = (3);
    #define __c2 c2
    const AN5D_TYPE __c3Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c3Pad = (3);
    #define __c3 c3
    const AN5D_TYPE __halo1 = 3;
    const AN5D_TYPE __halo2 = 3;
    const AN5D_TYPE __halo3 = 3;
    const AN5D_TYPE __side0Len = 1;
    const AN5D_TYPE __side1Len = 128;
    const AN5D_TYPE __side2Len = 26;
    const AN5D_TYPE __side3Len = 26;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __OlLen3 = (__halo3 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __side3LenOl = (__side3Len + 2 * __OlLen3);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl * __side3LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __side3Num = (__c3Len + __side3Len - 1) / __side3Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid / __side3LenOl;
    const AN5D_TYPE __local_c3 = __tid % __side3LenOl;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num / __side3Num;
    const AN5D_TYPE __c2 = (blockIdx.x / __side3Num % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    const AN5D_TYPE __c3 = (blockIdx.x % __side3Num) * __side3Len + __local_c3 + __c3Pad - __OlLen3;
    float __reg_0_0;
    float __reg_0_1;
    float __reg_0_2;
    float __reg_0_3;
    float __reg_0_4;
    float __reg_0_5;
    float __reg_0_6;
    __shared__ float __d_sb_double[__blockSize * 2];
    float *__d_sb = __d_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2 && __c3 >= __c3Pad - __halo3 && __c3 < __c3Pad + __c3Len + __halo3;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len && __c3 >= __c3Pad && __c3 < __c3Pad + __c3Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1) && __local_c3 >= (__halo3 * 1) && __local_c3 < __side3LenOl - (__halo3 * 1);
    const AN5D_TYPE __storeValid = __writeValid1;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[(((__c0 % 2) * dimsize + __c1) * dimsize + __c2) * dimsize + __c3]; }} while (0)
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR(__rn0, __a, __b, __c, __d, __e, __f, __g) do { __rn0 = (((((((((((((((((((0.25000f * (__REGREF(__d, 0, 0))) + (0.04276f * (__SBREF(__d_sb, 0, -3)))) + (0.04176f * (__SBREF(__d_sb, 0, -2)))) + (0.04076f * (__SBREF(__d_sb, 0, -1)))) + (0.04046f * (__SBREF(__d_sb, 0, 1)))) + (0.04146f * (__SBREF(__d_sb, 0, 2)))) + (0.04246f * (__SBREF(__d_sb, 0, 3)))) + (0.04096f * (__REGREF(__c, 0, 0)))) + (0.04066f * (__REGREF(__e, 0, 0)))) + (0.04086f * (__SBREF(__d_sb, -1, 0)))) + (0.04056f * (__SBREF(__d_sb, 1, 0)))) + (0.04196f * (__REGREF(__b, 0, 0)))) + (0.04166f * (__REGREF(__f, 0, 0)))) + (0.04186f * (__SBREF(__d_sb, -2, 0)))) + (0.04156f * (__SBREF(__d_sb, 2, 0)))) + (0.04296f * (__REGREF(__a, 0, 0)))) + (0.04266f * (__REGREF(__g, 0, 0)))) + (0.04286f * (__SBREF(__d_sb, -3, 0)))) + (0.04256f * (__SBREF(__d_sb, 3, 0)))); } while (0)
    #define __DB_SWITCH() do { __d_sb = &__d_sb_double[(__d_sb == __d_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a, b, c, d, e, f, g) do { __DB_SWITCH(); __d_sb[__tid] = d; __syncthreads(); } while (0)
    #define __STORE(h, reg0, reg1, reg2, reg3, reg4, reg5, reg6) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6); if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __CALCEXPR(__DEST, reg0, reg1, reg2, reg3, reg4, reg5, reg6); } } while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0_0, 0);
      __LOAD(__reg_0_1, 1);
      __LOAD(__reg_0_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __STORE(3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
    }
    else
    {
      __LOAD(__reg_0_0, 0);
      __LOAD(__reg_0_1, 1);
      __LOAD(__reg_0_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __STORE(3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
    }
    __d_sb = __d_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 7; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 7;)
      {
        __LOAD(__reg_0_0, __h);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __h++;
        __LOAD(__reg_0_1, __h);
        __STORE(__h - 3, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __h++;
        __LOAD(__reg_0_2, __h);
        __STORE(__h - 3, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __h++;
        __LOAD(__reg_0_3, __h);
        __STORE(__h - 3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __h++;
        __LOAD(__reg_0_4, __h);
        __STORE(__h - 3, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __h++;
        __LOAD(__reg_0_5, __h);
        __STORE(__h - 3, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __h++;
        __LOAD(__reg_0_6, __h);
        __STORE(__h - 3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 0 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
      }
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 2, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 2, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __LOAD(__reg_0_2, __h + 2);
        __STORE(__h - 1, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 2, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __LOAD(__reg_0_2, __h + 2);
        __STORE(__h - 1, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __LOAD(__reg_0_3, __h + 3);
        __STORE(__h + 0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 2, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __LOAD(__reg_0_2, __h + 2);
        __STORE(__h - 1, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __LOAD(__reg_0_3, __h + 3);
        __STORE(__h + 0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __LOAD(__reg_0_4, __h + 4);
        __STORE(__h + 1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 2, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __LOAD(__reg_0_2, __h + 2);
        __STORE(__h - 1, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __LOAD(__reg_0_3, __h + 3);
        __STORE(__h + 0, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __LOAD(__reg_0_4, __h + 4);
        __STORE(__h + 1, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __LOAD(__reg_0_5, __h + 5);
        __STORE(__h + 2, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      }
    }
    else
    {
      for (__h = 7; __h <= __side1LenOl - 7;)
      {
        __LOAD(__reg_0_0, __h);
        __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
        __h++;
        __LOAD(__reg_0_1, __h);
        __STORE(__h - 3, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
        __h++;
        __LOAD(__reg_0_2, __h);
        __STORE(__h - 3, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
        __h++;
        __LOAD(__reg_0_3, __h);
        __STORE(__h - 3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __h++;
        __LOAD(__reg_0_4, __h);
        __STORE(__h - 3, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __h++;
        __LOAD(__reg_0_5, __h);
        __STORE(__h - 3, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __h++;
        __LOAD(__reg_0_6, __h);
        __STORE(__h - 3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __h++;
        __DB_SWITCH();  __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_0, __h);
      __STORE(__h - 3, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_1, __h);
      __STORE(__h - 3, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_2, __h);
      __STORE(__h - 3, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_3, __h);
      __STORE(__h - 3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_4, __h);
      __STORE(__h - 3, __reg_0_5, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_5, __h);
      __STORE(__h - 3, __reg_0_6, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_6, __h);
      __STORE(__h - 3, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __h++;
    }
}
