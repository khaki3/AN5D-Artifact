#include "hip/hip_runtime.h"
#include "star3d4r-64x16-1-256_kernel.hu"
__device__ float __sbref_wrap(float *sb, size_t index) { return sb[index]; }

__global__ void kernel0_1(float *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 4 - 4);
    const AN5D_TYPE __c1Pad = (4);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 4 - 4);
    const AN5D_TYPE __c2Pad = (4);
    #define __c2 c2
    const AN5D_TYPE __c3Len = (dimsize - 4 - 4);
    const AN5D_TYPE __c3Pad = (4);
    #define __c3 c3
    const AN5D_TYPE __halo1 = 4;
    const AN5D_TYPE __halo2 = 4;
    const AN5D_TYPE __halo3 = 4;
    const AN5D_TYPE __side0Len = 1;
    const AN5D_TYPE __side1Len = 256;
    const AN5D_TYPE __side2Len = 8;
    const AN5D_TYPE __side3Len = 56;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __OlLen3 = (__halo3 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __side3LenOl = (__side3Len + 2 * __OlLen3);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl * __side3LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __side3Num = (__c3Len + __side3Len - 1) / __side3Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid / __side3LenOl;
    const AN5D_TYPE __local_c3 = __tid % __side3LenOl;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num / __side3Num;
    const AN5D_TYPE __c2 = (blockIdx.x / __side3Num % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    const AN5D_TYPE __c3 = (blockIdx.x % __side3Num) * __side3Len + __local_c3 + __c3Pad - __OlLen3;
    float __reg_0_0;
    float __reg_0_1;
    float __reg_0_2;
    float __reg_0_3;
    float __reg_0_4;
    float __reg_0_5;
    float __reg_0_6;
    float __reg_0_7;
    float __reg_0_8;
    __shared__ float __e_sb_double[__blockSize * 2];
    float *__e_sb = __e_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2 && __c3 >= __c3Pad - __halo3 && __c3 < __c3Pad + __c3Len + __halo3;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len && __c3 >= __c3Pad && __c3 < __c3Pad + __c3Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1) && __local_c3 >= (__halo3 * 1) && __local_c3 < __side3LenOl - (__halo3 * 1);
    const AN5D_TYPE __storeValid = __writeValid1;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[(((__c0 % 2) * dimsize + __c1) * dimsize + __c2) * dimsize + __c3]; }} while (0)
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR(__rn0, __a, __b, __c, __d, __e, __f, __g, __h, __i) do { __rn0 = (((((((((((((((((((((((((0.25000f * (__REGREF(__e, 0, 0))) + (0.03228f * (__SBREF(__e_sb, 0, -4)))) + (0.03138f * (__SBREF(__e_sb, 0, -3)))) + (0.03118f * (__SBREF(__e_sb, 0, -2)))) + (0.03027f * (__SBREF(__e_sb, 0, -1)))) + (0.03022f * (__SBREF(__e_sb, 0, 1)))) + (0.03112f * (__SBREF(__e_sb, 0, 2)))) + (0.03132f * (__SBREF(__e_sb, 0, 3)))) + (0.03222f * (__SBREF(__e_sb, 0, 4)))) + (0.03026f * (__REGREF(__d, 0, 0)))) + (0.03024f * (__REGREF(__f, 0, 0)))) + (0.03027f * (__SBREF(__e_sb, -1, 0)))) + (0.03023f * (__SBREF(__e_sb, 1, 0)))) + (0.03116f * (__REGREF(__c, 0, 0)))) + (0.03114f * (__REGREF(__g, 0, 0)))) + (0.03117f * (__SBREF(__e_sb, -2, 0)))) + (0.03113f * (__SBREF(__e_sb, 2, 0)))) + (0.03136f * (__REGREF(__b, 0, 0)))) + (0.03134f * (__REGREF(__h, 0, 0)))) + (0.03137f * (__SBREF(__e_sb, -3, 0)))) + (0.03133f * (__SBREF(__e_sb, 3, 0)))) + (0.03226f * (__REGREF(__a, 0, 0)))) + (0.03224f * (__REGREF(__i, 0, 0)))) + (0.03227f * (__SBREF(__e_sb, -4, 0)))) + (0.03223f * (__SBREF(__e_sb, 4, 0)))); } while (0)
    #define __DB_SWITCH() do { __e_sb = &__e_sb_double[(__e_sb == __e_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a, b, c, d, e, f, g, h, i) do { __DB_SWITCH(); __e_sb[__tid] = e; __syncthreads(); } while (0)
    #define __STORE(h, reg0, reg1, reg2, reg3, reg4, reg5, reg6, reg7, reg8) do { __CALCSETUP(reg0, reg1, reg2, reg3, reg4, reg5, reg6, reg7, reg8); if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __CALCEXPR(__DEST, reg0, reg1, reg2, reg3, reg4, reg5, reg6, reg7, reg8); } } while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0_0, 0);
      __LOAD(__reg_0_1, 1);
      __LOAD(__reg_0_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __LOAD(__reg_0_7, 7);
      __LOAD(__reg_0_8, 8);
      __STORE(4, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8);
    }
    else
    {
      __LOAD(__reg_0_0, 0);
      __LOAD(__reg_0_1, 1);
      __LOAD(__reg_0_2, 2);
      __LOAD(__reg_0_3, 3);
      __LOAD(__reg_0_4, 4);
      __LOAD(__reg_0_5, 5);
      __LOAD(__reg_0_6, 6);
      __LOAD(__reg_0_7, 7);
      __LOAD(__reg_0_8, 8);
      __STORE(4, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8);
    }
    __e_sb = __e_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 9; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 9;)
      {
        __LOAD(__reg_0_0, __h);
        __STORE(__h - 4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0);
        __h++;
        __LOAD(__reg_0_1, __h);
        __STORE(__h - 4, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1);
        __h++;
        __LOAD(__reg_0_2, __h);
        __STORE(__h - 4, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2);
        __h++;
        __LOAD(__reg_0_3, __h);
        __STORE(__h - 4, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __h++;
        __LOAD(__reg_0_4, __h);
        __STORE(__h - 4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __h++;
        __LOAD(__reg_0_5, __h);
        __STORE(__h - 4, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __h++;
        __LOAD(__reg_0_6, __h);
        __STORE(__h - 4, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __h++;
        __LOAD(__reg_0_7, __h);
        __STORE(__h - 4, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7);
        __h++;
        __LOAD(__reg_0_8, __h);
        __STORE(__h - 4, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 0 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
      }
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 3, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 3, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1);
        __LOAD(__reg_0_2, __h + 2);
        __STORE(__h - 2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 3, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1);
        __LOAD(__reg_0_2, __h + 2);
        __STORE(__h - 2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2);
        __LOAD(__reg_0_3, __h + 3);
        __STORE(__h - 1, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 3, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1);
        __LOAD(__reg_0_2, __h + 2);
        __STORE(__h - 2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2);
        __LOAD(__reg_0_3, __h + 3);
        __STORE(__h - 1, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __LOAD(__reg_0_4, __h + 4);
        __STORE(__h + 0, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 3, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1);
        __LOAD(__reg_0_2, __h + 2);
        __STORE(__h - 2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2);
        __LOAD(__reg_0_3, __h + 3);
        __STORE(__h - 1, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __LOAD(__reg_0_4, __h + 4);
        __STORE(__h + 0, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __LOAD(__reg_0_5, __h + 5);
        __STORE(__h + 1, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      }
      else if (__h + 7 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 3, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1);
        __LOAD(__reg_0_2, __h + 2);
        __STORE(__h - 2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2);
        __LOAD(__reg_0_3, __h + 3);
        __STORE(__h - 1, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __LOAD(__reg_0_4, __h + 4);
        __STORE(__h + 0, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __LOAD(__reg_0_5, __h + 5);
        __STORE(__h + 1, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __LOAD(__reg_0_6, __h + 6);
        __STORE(__h + 2, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      }
      else if (__h + 8 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0_0, __h + 0);
        __STORE(__h - 4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0);
        __LOAD(__reg_0_1, __h + 1);
        __STORE(__h - 3, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1);
        __LOAD(__reg_0_2, __h + 2);
        __STORE(__h - 2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2);
        __LOAD(__reg_0_3, __h + 3);
        __STORE(__h - 1, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __LOAD(__reg_0_4, __h + 4);
        __STORE(__h + 0, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __LOAD(__reg_0_5, __h + 5);
        __STORE(__h + 1, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __LOAD(__reg_0_6, __h + 6);
        __STORE(__h + 2, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __LOAD(__reg_0_7, __h + 7);
        __STORE(__h + 3, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7);
      }
    }
    else
    {
      for (__h = 9; __h <= __side1LenOl - 9;)
      {
        __LOAD(__reg_0_0, __h);
        __STORE(__h - 4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0);
        __h++;
        __LOAD(__reg_0_1, __h);
        __STORE(__h - 4, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1);
        __h++;
        __LOAD(__reg_0_2, __h);
        __STORE(__h - 4, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2);
        __h++;
        __LOAD(__reg_0_3, __h);
        __STORE(__h - 4, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
        __h++;
        __LOAD(__reg_0_4, __h);
        __STORE(__h - 4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
        __h++;
        __LOAD(__reg_0_5, __h);
        __STORE(__h - 4, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
        __h++;
        __LOAD(__reg_0_6, __h);
        __STORE(__h - 4, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
        __h++;
        __LOAD(__reg_0_7, __h);
        __STORE(__h - 4, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7);
        __h++;
        __LOAD(__reg_0_8, __h);
        __STORE(__h - 4, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8);
        __h++;
        __DB_SWITCH();  __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_0, __h);
      __STORE(__h - 4, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_1, __h);
      __STORE(__h - 4, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_2, __h);
      __STORE(__h - 4, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_3, __h);
      __STORE(__h - 4, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_4, __h);
      __STORE(__h - 4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_5, __h);
      __STORE(__h - 4, __reg_0_6, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_6, __h);
      __STORE(__h - 4, __reg_0_7, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_7, __h);
      __STORE(__h - 4, __reg_0_8, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0_8, __h);
      __STORE(__h - 4, __reg_0_0, __reg_0_1, __reg_0_2, __reg_0_3, __reg_0_4, __reg_0_5, __reg_0_6, __reg_0_7, __reg_0_8);
      __h++;
    }
}
