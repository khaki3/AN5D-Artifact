#include "hip/hip_runtime.h"
#include "box3d3r-32x16-1-128_kernel.hu"
__device__ double __sbref_wrap(double *sb, size_t index) { return sb[index]; }

__global__ void kernel0_1(double *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c1Pad = (3);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c2Pad = (3);
    #define __c2 c2
    const AN5D_TYPE __c3Len = (dimsize - 3 - 3);
    const AN5D_TYPE __c3Pad = (3);
    #define __c3 c3
    const AN5D_TYPE __halo1 = 3;
    const AN5D_TYPE __halo2 = 3;
    const AN5D_TYPE __halo3 = 3;
    const AN5D_TYPE __side0Len = 1;
    const AN5D_TYPE __side1Len = 128;
    const AN5D_TYPE __side2Len = 10;
    const AN5D_TYPE __side3Len = 26;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __OlLen3 = (__halo3 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __side3LenOl = (__side3Len + 2 * __OlLen3);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl * __side3LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __side3Num = (__c3Len + __side3Len - 1) / __side3Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid / __side3LenOl;
    const AN5D_TYPE __local_c3 = __tid % __side3LenOl;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num / __side3Num;
    const AN5D_TYPE __c2 = (blockIdx.x / __side3Num % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    const AN5D_TYPE __c3 = (blockIdx.x % __side3Num) * __side3Len + __local_c3 + __c3Pad - __OlLen3;
    double __reg_0;
    double __reg_1_0;
    double __reg_1_1;
    double __reg_1_2;
    double __reg_1_3;
    double __reg_1_4;
    double __reg_1_5;
    double __reg_1_6;
    __shared__ double __a_sb_double[__blockSize * 2];
    double *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2 && __c3 >= __c3Pad - __halo3 && __c3 < __c3Pad + __c3Len + __halo3;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len && __c3 >= __c3Pad && __c3 < __c3Pad + __c3Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1) && __local_c3 >= (__halo3 * 1) && __local_c3 < __side3LenOl - (__halo3 * 1);
    const AN5D_TYPE __storeValid = __writeValid1;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[(((__c0 % 2) * dimsize + __c1) * dimsize + __c2) * dimsize + __c3]; }} while (0)
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((-0.176f) * (__REGREF(__a, 0, 0))) + (0.0010f * (__SBREF(__a_sb, -3, -3)))) + (0.0020f * (__SBREF(__a_sb, -3, -2)))) + (0.0030f * (__SBREF(__a_sb, -3, -1)))) + (0.0040f * (__SBREF(__a_sb, -3, 0)))) + (0.0050f * (__SBREF(__a_sb, -3, 1)))) + (0.0060f * (__SBREF(__a_sb, -3, 2)))) + (0.0070f * (__SBREF(__a_sb, -3, 3)))) + (0.0080f * (__SBREF(__a_sb, -2, -3)))) + (0.0090f * (__SBREF(__a_sb, -2, -2)))) + (0.0100f * (__SBREF(__a_sb, -2, -1)))) + (0.0110f * (__SBREF(__a_sb, -2, 0)))) + (0.0120f * (__SBREF(__a_sb, -2, 1)))) + (0.0130f * (__SBREF(__a_sb, -2, 2)))) + (0.0140f * (__SBREF(__a_sb, -2, 3)))) + (0.0150f * (__SBREF(__a_sb, -1, -3)))) + (0.0160f * (__SBREF(__a_sb, -1, -2)))) + (0.0170f * (__SBREF(__a_sb, -1, -1)))) + (0.0180f * (__SBREF(__a_sb, -1, 0)))) + (0.0190f * (__SBREF(__a_sb, -1, 1)))) + (0.0200f * (__SBREF(__a_sb, -1, 2)))) + (0.0210f * (__SBREF(__a_sb, -1, 3)))) + (0.0220f * (__SBREF(__a_sb, 0, -3)))) + (0.0230f * (__SBREF(__a_sb, 0, -2)))) + (0.0240f * (__SBREF(__a_sb, 0, -1)))) + (0.0250f * (__SBREF(__a_sb, 0, 1)))) + (0.0260f * (__SBREF(__a_sb, 0, 2)))) + (0.0270f * (__SBREF(__a_sb, 0, 3)))) + (0.0280f * (__SBREF(__a_sb, 1, -3)))) + (0.0290f * (__SBREF(__a_sb, 1, -2)))) + (0.0300f * (__SBREF(__a_sb, 1, -1)))) + (0.0310f * (__SBREF(__a_sb, 1, 0)))) + (0.0320f * (__SBREF(__a_sb, 1, 1)))) + (0.0330f * (__SBREF(__a_sb, 1, 2)))) + (0.0340f * (__SBREF(__a_sb, 1, 3)))) + (0.0350f * (__SBREF(__a_sb, 2, -3)))) + (0.0360f * (__SBREF(__a_sb, 2, -2)))) + (0.0370f * (__SBREF(__a_sb, 2, -1)))) + (0.0380f * (__SBREF(__a_sb, 2, 0)))) + (0.0390f * (__SBREF(__a_sb, 2, 1)))) + (0.0400f * (__SBREF(__a_sb, 2, 2)))) + (0.0410f * (__SBREF(__a_sb, 2, 3)))) + (0.0420f * (__SBREF(__a_sb, 3, -3)))) + (0.0430f * (__SBREF(__a_sb, 3, -2)))) + (0.0440f * (__SBREF(__a_sb, 3, -1)))) + (0.0450f * (__SBREF(__a_sb, 3, 0)))) + (0.0460f * (__SBREF(__a_sb, 3, 1)))) + (0.0470f * (__SBREF(__a_sb, 3, 2)))) + (0.0480f * (__SBREF(__a_sb, 3, 3)))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.1808f * (__REGREF(__a, 0, 0)))) - (0.0011f * (__SBREF(__a_sb, -3, -3)))) - (0.0021f * (__SBREF(__a_sb, -3, -2)))) - (0.0031f * (__SBREF(__a_sb, -3, -1)))) - (0.0041f * (__SBREF(__a_sb, -3, 0)))) - (0.0051f * (__SBREF(__a_sb, -3, 1)))) - (0.0061f * (__SBREF(__a_sb, -3, 2)))) - (0.0071f * (__SBREF(__a_sb, -3, 3)))) - (0.0081f * (__SBREF(__a_sb, -2, -3)))) - (0.0091f * (__SBREF(__a_sb, -2, -2)))) - (0.0101f * (__SBREF(__a_sb, -2, -1)))) - (0.0111f * (__SBREF(__a_sb, -2, 0)))) - (0.0121f * (__SBREF(__a_sb, -2, 1)))) - (0.0131f * (__SBREF(__a_sb, -2, 2)))) - (0.0141f * (__SBREF(__a_sb, -2, 3)))) - (0.0151f * (__SBREF(__a_sb, -1, -3)))) - (0.0161f * (__SBREF(__a_sb, -1, -2)))) - (0.0171f * (__SBREF(__a_sb, -1, -1)))) - (0.0181f * (__SBREF(__a_sb, -1, 0)))) - (0.0191f * (__SBREF(__a_sb, -1, 1)))) - (0.0201f * (__SBREF(__a_sb, -1, 2)))) - (0.0211f * (__SBREF(__a_sb, -1, 3)))) - (0.0221f * (__SBREF(__a_sb, 0, -3)))) - (0.0231f * (__SBREF(__a_sb, 0, -2)))) - (0.0241f * (__SBREF(__a_sb, 0, -1)))) - (0.0251f * (__SBREF(__a_sb, 0, 1)))) - (0.0261f * (__SBREF(__a_sb, 0, 2)))) - (0.0271f * (__SBREF(__a_sb, 0, 3)))) - (0.0281f * (__SBREF(__a_sb, 1, -3)))) - (0.0291f * (__SBREF(__a_sb, 1, -2)))) - (0.0301f * (__SBREF(__a_sb, 1, -1)))) - (0.0311f * (__SBREF(__a_sb, 1, 0)))) - (0.0321f * (__SBREF(__a_sb, 1, 1)))) - (0.0331f * (__SBREF(__a_sb, 1, 2)))) - (0.0341f * (__SBREF(__a_sb, 1, 3)))) - (0.0351f * (__SBREF(__a_sb, 2, -3)))) - (0.0361f * (__SBREF(__a_sb, 2, -2)))) - (0.0371f * (__SBREF(__a_sb, 2, -1)))) - (0.0381f * (__SBREF(__a_sb, 2, 0)))) - (0.0391f * (__SBREF(__a_sb, 2, 1)))) - (0.0401f * (__SBREF(__a_sb, 2, 2)))) - (0.0411f * (__SBREF(__a_sb, 2, 3)))) - (0.0421f * (__SBREF(__a_sb, 3, -3)))) - (0.0431f * (__SBREF(__a_sb, 3, -2)))) - (0.0441f * (__SBREF(__a_sb, 3, -1)))) - (0.0451f * (__SBREF(__a_sb, 3, 0)))) - (0.0461f * (__SBREF(__a_sb, 3, 1)))) - (0.0471f * (__SBREF(__a_sb, 3, 2)))) - (0.0481f * (__SBREF(__a_sb, 3, 3))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { double etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((-0.1856f) * (__REGREF(__a, 0, 0)))) + (0.0012f * (__SBREF(__a_sb, -3, -3)))) + (0.0022f * (__SBREF(__a_sb, -3, -2)))) + (0.0032f * (__SBREF(__a_sb, -3, -1)))) + (0.0042f * (__SBREF(__a_sb, -3, 0)))) + (0.0052f * (__SBREF(__a_sb, -3, 1)))) + (0.0062f * (__SBREF(__a_sb, -3, 2)))) + (0.0072f * (__SBREF(__a_sb, -3, 3)))) + (0.0082f * (__SBREF(__a_sb, -2, -3)))) + (0.0092f * (__SBREF(__a_sb, -2, -2)))) + (0.0102f * (__SBREF(__a_sb, -2, -1)))) + (0.0112f * (__SBREF(__a_sb, -2, 0)))) + (0.0122f * (__SBREF(__a_sb, -2, 1)))) + (0.0132f * (__SBREF(__a_sb, -2, 2)))) + (0.0142f * (__SBREF(__a_sb, -2, 3)))) + (0.0152f * (__SBREF(__a_sb, -1, -3)))) + (0.0162f * (__SBREF(__a_sb, -1, -2)))) + (0.0172f * (__SBREF(__a_sb, -1, -1)))) + (0.0182f * (__SBREF(__a_sb, -1, 0)))) + (0.0192f * (__SBREF(__a_sb, -1, 1)))) + (0.0202f * (__SBREF(__a_sb, -1, 2)))) + (0.0212f * (__SBREF(__a_sb, -1, 3)))) + (0.0222f * (__SBREF(__a_sb, 0, -3)))) + (0.0232f * (__SBREF(__a_sb, 0, -2)))) + (0.0242f * (__SBREF(__a_sb, 0, -1)))) + (0.0252f * (__SBREF(__a_sb, 0, 1)))) + (0.0262f * (__SBREF(__a_sb, 0, 2)))) + (0.0272f * (__SBREF(__a_sb, 0, 3)))) + (0.0282f * (__SBREF(__a_sb, 1, -3)))) + (0.0292f * (__SBREF(__a_sb, 1, -2)))) + (0.0302f * (__SBREF(__a_sb, 1, -1)))) + (0.0312f * (__SBREF(__a_sb, 1, 0)))) + (0.0322f * (__SBREF(__a_sb, 1, 1)))) + (0.0332f * (__SBREF(__a_sb, 1, 2)))) + (0.0342f * (__SBREF(__a_sb, 1, 3)))) + (0.0352f * (__SBREF(__a_sb, 2, -3)))) + (0.0362f * (__SBREF(__a_sb, 2, -2)))) + (0.0372f * (__SBREF(__a_sb, 2, -1)))) + (0.0382f * (__SBREF(__a_sb, 2, 0)))) + (0.0392f * (__SBREF(__a_sb, 2, 1)))) + (0.0402f * (__SBREF(__a_sb, 2, 2)))) + (0.0412f * (__SBREF(__a_sb, 2, 3)))) + (0.0422f * (__SBREF(__a_sb, 3, -3)))) + (0.0432f * (__SBREF(__a_sb, 3, -2)))) + (0.0442f * (__SBREF(__a_sb, 3, -1)))) + (0.0452f * (__SBREF(__a_sb, 3, 0)))) + (0.0462f * (__SBREF(__a_sb, 3, 1)))) + (0.0472f * (__SBREF(__a_sb, 3, 2)))) + (0.0482f * (__SBREF(__a_sb, 3, 3)))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { double etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_3_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((-0.1904f) * (__REGREF(__a, 0, 0)))) + (0.0013f * (__SBREF(__a_sb, -3, -3)))) + (0.0023f * (__SBREF(__a_sb, -3, -2)))) + (0.0033f * (__SBREF(__a_sb, -3, -1)))) + (0.0043f * (__SBREF(__a_sb, -3, 0)))) + (0.0053f * (__SBREF(__a_sb, -3, 1)))) + (0.0063f * (__SBREF(__a_sb, -3, 2)))) + (0.0073f * (__SBREF(__a_sb, -3, 3)))) + (0.0083f * (__SBREF(__a_sb, -2, -3)))) + (0.0093f * (__SBREF(__a_sb, -2, -2)))) + (0.0103f * (__SBREF(__a_sb, -2, -1)))) + (0.0113f * (__SBREF(__a_sb, -2, 0)))) + (0.0123f * (__SBREF(__a_sb, -2, 1)))) + (0.0133f * (__SBREF(__a_sb, -2, 2)))) + (0.0143f * (__SBREF(__a_sb, -2, 3)))) + (0.0153f * (__SBREF(__a_sb, -1, -3)))) + (0.0163f * (__SBREF(__a_sb, -1, -2)))) + (0.0173f * (__SBREF(__a_sb, -1, -1)))) + (0.0183f * (__SBREF(__a_sb, -1, 0)))) + (0.0193f * (__SBREF(__a_sb, -1, 1)))) + (0.0203f * (__SBREF(__a_sb, -1, 2)))) + (0.0213f * (__SBREF(__a_sb, -1, 3)))) + (0.0223f * (__SBREF(__a_sb, 0, -3)))) + (0.0233f * (__SBREF(__a_sb, 0, -2)))) + (0.0243f * (__SBREF(__a_sb, 0, -1)))) + (0.0253f * (__SBREF(__a_sb, 0, 1)))) + (0.0263f * (__SBREF(__a_sb, 0, 2)))) + (0.0273f * (__SBREF(__a_sb, 0, 3)))) + (0.0283f * (__SBREF(__a_sb, 1, -3)))) + (0.0293f * (__SBREF(__a_sb, 1, -2)))) + (0.0303f * (__SBREF(__a_sb, 1, -1)))) + (0.0313f * (__SBREF(__a_sb, 1, 0)))) + (0.0323f * (__SBREF(__a_sb, 1, 1)))) + (0.0333f * (__SBREF(__a_sb, 1, 2)))) + (0.0343f * (__SBREF(__a_sb, 1, 3)))) + (0.0353f * (__SBREF(__a_sb, 2, -3)))) + (0.0363f * (__SBREF(__a_sb, 2, -2)))) + (0.0373f * (__SBREF(__a_sb, 2, -1)))) + (0.0383f * (__SBREF(__a_sb, 2, 0)))) + (0.0393f * (__SBREF(__a_sb, 2, 1)))) + (0.0403f * (__SBREF(__a_sb, 2, 2)))) + (0.0413f * (__SBREF(__a_sb, 2, 3)))) + (0.0423f * (__SBREF(__a_sb, 3, -3)))) + (0.0433f * (__SBREF(__a_sb, 3, -2)))) + (0.0443f * (__SBREF(__a_sb, 3, -1)))) + (0.0453f * (__SBREF(__a_sb, 3, 0)))) + (0.0463f * (__SBREF(__a_sb, 3, 1)))) + (0.0473f * (__SBREF(__a_sb, 3, 2)))) + (0.0483f * (__SBREF(__a_sb, 3, 3))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_3(out, a) do { double etmp; __CALCEXPR_3_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_4_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.1952f * (__REGREF(__a, 0, 0)))) - (0.0014f * (__SBREF(__a_sb, -3, -3)))) - (0.0024f * (__SBREF(__a_sb, -3, -2)))) - (0.0034f * (__SBREF(__a_sb, -3, -1)))) - (0.0044f * (__SBREF(__a_sb, -3, 0)))) - (0.0054f * (__SBREF(__a_sb, -3, 1)))) - (0.0064f * (__SBREF(__a_sb, -3, 2)))) - (0.0074f * (__SBREF(__a_sb, -3, 3)))) - (0.0084f * (__SBREF(__a_sb, -2, -3)))) - (0.0094f * (__SBREF(__a_sb, -2, -2)))) - (0.0104f * (__SBREF(__a_sb, -2, -1)))) - (0.0114f * (__SBREF(__a_sb, -2, 0)))) - (0.0124f * (__SBREF(__a_sb, -2, 1)))) - (0.0134f * (__SBREF(__a_sb, -2, 2)))) - (0.0144f * (__SBREF(__a_sb, -2, 3)))) - (0.0154f * (__SBREF(__a_sb, -1, -3)))) - (0.0164f * (__SBREF(__a_sb, -1, -2)))) - (0.0174f * (__SBREF(__a_sb, -1, -1)))) - (0.0184f * (__SBREF(__a_sb, -1, 0)))) - (0.0194f * (__SBREF(__a_sb, -1, 1)))) - (0.0204f * (__SBREF(__a_sb, -1, 2)))) - (0.0214f * (__SBREF(__a_sb, -1, 3)))) - (0.0224f * (__SBREF(__a_sb, 0, -3)))) - (0.0234f * (__SBREF(__a_sb, 0, -2)))) - (0.0244f * (__SBREF(__a_sb, 0, -1)))) - (0.0254f * (__SBREF(__a_sb, 0, 1)))) - (0.0264f * (__SBREF(__a_sb, 0, 2)))) - (0.0274f * (__SBREF(__a_sb, 0, 3)))) - (0.0284f * (__SBREF(__a_sb, 1, -3)))) - (0.0294f * (__SBREF(__a_sb, 1, -2)))) - (0.0304f * (__SBREF(__a_sb, 1, -1)))) - (0.0314f * (__SBREF(__a_sb, 1, 0)))) - (0.0324f * (__SBREF(__a_sb, 1, 1)))) - (0.0334f * (__SBREF(__a_sb, 1, 2)))) - (0.0344f * (__SBREF(__a_sb, 1, 3)))) - (0.0354f * (__SBREF(__a_sb, 2, -3)))) - (0.0364f * (__SBREF(__a_sb, 2, -2)))) - (0.0374f * (__SBREF(__a_sb, 2, -1)))) - (0.0384f * (__SBREF(__a_sb, 2, 0)))) - (0.0394f * (__SBREF(__a_sb, 2, 1)))) - (0.0404f * (__SBREF(__a_sb, 2, 2)))) - (0.0414f * (__SBREF(__a_sb, 2, 3)))) - (0.0424f * (__SBREF(__a_sb, 3, -3)))) - (0.0434f * (__SBREF(__a_sb, 3, -2)))) - (0.0444f * (__SBREF(__a_sb, 3, -1)))) - (0.0454f * (__SBREF(__a_sb, 3, 0)))) - (0.0464f * (__SBREF(__a_sb, 3, 1)))) - (0.0474f * (__SBREF(__a_sb, 3, 2)))) - (0.0484f * (__SBREF(__a_sb, 3, 3)))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_4(out, a) do { double etmp; __CALCEXPR_4_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_5_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((-((-0.300f) * (__REGREF(__a, 0, 0)))) + (0.0015f * (__SBREF(__a_sb, -3, -3)))) + (0.0025f * (__SBREF(__a_sb, -3, -2)))) + (0.0035f * (__SBREF(__a_sb, -3, -1)))) + (0.0045f * (__SBREF(__a_sb, -3, 0)))) + (0.0055f * (__SBREF(__a_sb, -3, 1)))) + (0.0065f * (__SBREF(__a_sb, -3, 2)))) + (0.0075f * (__SBREF(__a_sb, -3, 3)))) + (0.0085f * (__SBREF(__a_sb, -2, -3)))) + (0.0095f * (__SBREF(__a_sb, -2, -2)))) + (0.0105f * (__SBREF(__a_sb, -2, -1)))) + (0.0115f * (__SBREF(__a_sb, -2, 0)))) + (0.0125f * (__SBREF(__a_sb, -2, 1)))) + (0.0135f * (__SBREF(__a_sb, -2, 2)))) + (0.0145f * (__SBREF(__a_sb, -2, 3)))) + (0.0155f * (__SBREF(__a_sb, -1, -3)))) + (0.0165f * (__SBREF(__a_sb, -1, -2)))) + (0.0175f * (__SBREF(__a_sb, -1, -1)))) + (0.0185f * (__SBREF(__a_sb, -1, 0)))) + (0.0195f * (__SBREF(__a_sb, -1, 1)))) + (0.0205f * (__SBREF(__a_sb, -1, 2)))) + (0.0215f * (__SBREF(__a_sb, -1, 3)))) + (0.0225f * (__SBREF(__a_sb, 0, -3)))) + (0.0235f * (__SBREF(__a_sb, 0, -2)))) + (0.0245f * (__SBREF(__a_sb, 0, -1)))) + (0.0255f * (__SBREF(__a_sb, 0, 1)))) + (0.0265f * (__SBREF(__a_sb, 0, 2)))) + (0.0275f * (__SBREF(__a_sb, 0, 3)))) + (0.0285f * (__SBREF(__a_sb, 1, -3)))) + (0.0295f * (__SBREF(__a_sb, 1, -2)))) + (0.0305f * (__SBREF(__a_sb, 1, -1)))) + (0.0315f * (__SBREF(__a_sb, 1, 0)))) + (0.0325f * (__SBREF(__a_sb, 1, 1)))) + (0.0335f * (__SBREF(__a_sb, 1, 2)))) + (0.0345f * (__SBREF(__a_sb, 1, 3)))) + (0.0355f * (__SBREF(__a_sb, 2, -3)))) + (0.0365f * (__SBREF(__a_sb, 2, -2)))) + (0.0375f * (__SBREF(__a_sb, 2, -1)))) + (0.0385f * (__SBREF(__a_sb, 2, 0)))) + (0.0395f * (__SBREF(__a_sb, 2, 1)))) + (0.0405f * (__SBREF(__a_sb, 2, 2)))) + (0.0415f * (__SBREF(__a_sb, 2, 3)))) + (0.0425f * (__SBREF(__a_sb, 3, -3)))) + (0.0435f * (__SBREF(__a_sb, 3, -2)))) + (0.0445f * (__SBREF(__a_sb, 3, -1)))) + (0.0455f * (__SBREF(__a_sb, 3, 0)))) + (0.0465f * (__SBREF(__a_sb, 3, 1)))) + (0.0475f * (__SBREF(__a_sb, 3, 2)))) + (0.1485f * (__SBREF(__a_sb, 3, 3))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_5(out, a) do { double etmp; __CALCEXPR_5_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_6_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((((((0.2048f * (__REGREF(__a, 0, 0)))) - (0.0016f * (__SBREF(__a_sb, -3, -3)))) - (0.0026f * (__SBREF(__a_sb, -3, -2)))) - (0.0036f * (__SBREF(__a_sb, -3, -1)))) - (0.0046f * (__SBREF(__a_sb, -3, 0)))) - (0.0056f * (__SBREF(__a_sb, -3, 1)))) - (0.0066f * (__SBREF(__a_sb, -3, 2)))) - (0.0076f * (__SBREF(__a_sb, -3, 3)))) - (0.0086f * (__SBREF(__a_sb, -2, -3)))) - (0.0096f * (__SBREF(__a_sb, -2, -2)))) - (0.0106f * (__SBREF(__a_sb, -2, -1)))) - (0.0116f * (__SBREF(__a_sb, -2, 0)))) - (0.0126f * (__SBREF(__a_sb, -2, 1)))) - (0.0136f * (__SBREF(__a_sb, -2, 2)))) - (0.0146f * (__SBREF(__a_sb, -2, 3)))) - (0.0156f * (__SBREF(__a_sb, -1, -3)))) - (0.0166f * (__SBREF(__a_sb, -1, -2)))) - (0.0176f * (__SBREF(__a_sb, -1, -1)))) - (0.0186f * (__SBREF(__a_sb, -1, 0)))) - (0.0196f * (__SBREF(__a_sb, -1, 1)))) - (0.0206f * (__SBREF(__a_sb, -1, 2)))) - (0.0216f * (__SBREF(__a_sb, -1, 3)))) - (0.0226f * (__SBREF(__a_sb, 0, -3)))) - (0.0236f * (__SBREF(__a_sb, 0, -2)))) - (0.0246f * (__SBREF(__a_sb, 0, -1)))) - (0.0256f * (__SBREF(__a_sb, 0, 1)))) - (0.0266f * (__SBREF(__a_sb, 0, 2)))) - (0.0276f * (__SBREF(__a_sb, 0, 3)))) - (0.0286f * (__SBREF(__a_sb, 1, -3)))) - (0.0296f * (__SBREF(__a_sb, 1, -2)))) - (0.0306f * (__SBREF(__a_sb, 1, -1)))) - (0.0316f * (__SBREF(__a_sb, 1, 0)))) - (0.0326f * (__SBREF(__a_sb, 1, 1)))) - (0.0336f * (__SBREF(__a_sb, 1, 2)))) - (0.0346f * (__SBREF(__a_sb, 1, 3)))) - (0.0356f * (__SBREF(__a_sb, 2, -3)))) - (0.0366f * (__SBREF(__a_sb, 2, -2)))) - (0.0376f * (__SBREF(__a_sb, 2, -1)))) - (0.0386f * (__SBREF(__a_sb, 2, 0)))) - (0.0396f * (__SBREF(__a_sb, 2, 1)))) - (0.0406f * (__SBREF(__a_sb, 2, 2)))) - (0.0416f * (__SBREF(__a_sb, 2, 3)))) - (0.0426f * (__SBREF(__a_sb, 3, -3)))) - (0.0436f * (__SBREF(__a_sb, 3, -2)))) - (0.0446f * (__SBREF(__a_sb, 3, -1)))) - (0.0456f * (__SBREF(__a_sb, 3, 0)))) - (0.0466f * (__SBREF(__a_sb, 3, 1)))) - (0.0476f * (__SBREF(__a_sb, 3, 2)))) - (0.0486f * (__SBREF(__a_sb, 3, 3)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_6(out, a) do { double etmp; __CALCEXPR_6_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); __CALCEXPR_3(out3, reg); __CALCEXPR_4(out4, reg); __CALCEXPR_5(out5, reg); __CALCEXPR_6(out6, reg); } while (0);
    #define __CALC1(out0, out1, out2, out3, out4, out5, out6, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, reg); } else out3 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __STORE(3, __reg_1_3);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __STORE(3, __reg_1_3);
    }
    __a_sb = __a_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 7; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 10;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 3, __reg_1_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 3, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 3, __reg_1_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h - 3, __reg_1_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 3, __reg_1_3);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 2, __reg_1_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_6, __reg_0);
        __STORE(__h - 1, __reg_1_6);
      }
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 2, __reg_1_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 1, __reg_1_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 2, __reg_1_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 1, __reg_1_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 2, __reg_1_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 1, __reg_1_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
      }
      else if (__h + 7 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 2, __reg_1_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 1, __reg_1_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
      }
      else if (__h + 8 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 2, __reg_1_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 1, __reg_1_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_4, __reg_0);
        __STORE(__h + 4, __reg_1_4);
      }
      else if (__h + 9 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 2, __reg_1_5);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 1, __reg_1_6);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h + 4, __reg_1_4);
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_5, __reg_0);
        __STORE(__h + 5, __reg_1_5);
      }
    }
    else
    {
      for (__h = 7; __h <= __side1LenOl - 7;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 3, __reg_1_4);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 3, __reg_1_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 3, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 3, __reg_1_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h - 3, __reg_1_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 3, __reg_1_3);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __STORE(__h - 3, __reg_1_4);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_0);
      __STORE(__h - 3, __reg_1_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_0);
      __STORE(__h - 3, __reg_1_6);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __STORE(__h - 3, __reg_1_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __STORE(__h - 3, __reg_1_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __STORE(__h - 3, __reg_1_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __STORE(__h - 3, __reg_1_3);
      __h++;
    }
}
