#include "hip/hip_runtime.h"
#include "box3d1r-32x32-3-128_kernel.hu"
__device__ double __sbref_wrap(double *sb, size_t index) { return sb[index]; }

__global__ void kernel0_3(double *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 1 - 1);
    const AN5D_TYPE __c1Pad = (1);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 1 - 1);
    const AN5D_TYPE __c2Pad = (1);
    #define __c2 c2
    const AN5D_TYPE __c3Len = (dimsize - 1 - 1);
    const AN5D_TYPE __c3Pad = (1);
    #define __c3 c3
    const AN5D_TYPE __halo1 = 1;
    const AN5D_TYPE __halo2 = 1;
    const AN5D_TYPE __halo3 = 1;
    const AN5D_TYPE __side0Len = 3;
    const AN5D_TYPE __side1Len = 128;
    const AN5D_TYPE __side2Len = 26;
    const AN5D_TYPE __side3Len = 26;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __OlLen3 = (__halo3 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __side3LenOl = (__side3Len + 2 * __OlLen3);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl * __side3LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __side3Num = (__c3Len + __side3Len - 1) / __side3Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid / __side3LenOl;
    const AN5D_TYPE __local_c3 = __tid % __side3LenOl;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num / __side3Num;
    const AN5D_TYPE __c2 = (blockIdx.x / __side3Num % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    const AN5D_TYPE __c3 = (blockIdx.x % __side3Num) * __side3Len + __local_c3 + __c3Pad - __OlLen3;
    double __reg_0;
    double __reg_1_0;
    double __reg_1_1;
    double __reg_1_2;
    double __reg_2_0;
    double __reg_2_1;
    double __reg_2_2;
    double __reg_3_0;
    double __reg_3_1;
    double __reg_3_2;
    __shared__ double __a_sb_double[__blockSize * 2];
    double *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2 && __c3 >= __c3Pad - __halo3 && __c3 < __c3Pad + __c3Len + __halo3;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len && __c3 >= __c3Pad && __c3 < __c3Pad + __c3Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1) && __local_c3 >= (__halo3 * 1) && __local_c3 < __side3LenOl - (__halo3 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2) && __local_c3 >= (__halo3 * 2) && __local_c3 < __side3LenOl - (__halo3 * 2);
    const AN5D_TYPE __writeValid3 = __updateValid && __local_c2 >= (__halo2 * 3) && __local_c2 < __side2LenOl - (__halo2 * 3) && __local_c3 >= (__halo3 * 3) && __local_c3 < __side3LenOl - (__halo3 * 3);
    const AN5D_TYPE __storeValid = __writeValid3;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[(((__c0 % 2) * dimsize + __c1) * dimsize + __c2) * dimsize + __c3]; }} while (0)
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((0.0375f * (__REGREF(__a, 0, 0))) + (0.0371f * (__SBREF(__a_sb, -1, -1)))) + (0.0372f * (__SBREF(__a_sb, -1, 0)))) + (0.0373f * (__SBREF(__a_sb, -1, 1)))) + (0.0374f * (__SBREF(__a_sb, 0, -1)))) + (0.0376f * (__SBREF(__a_sb, 0, 1)))) + (0.0377f * (__SBREF(__a_sb, 1, -1)))) + (0.0378f * (__SBREF(__a_sb, 1, 0)))) + (0.0379f * (__SBREF(__a_sb, 1, 1)))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((0.0355f * (__REGREF(__a, 0, 0)))) + (0.0351f * (__SBREF(__a_sb, -1, -1)))) + (0.0352f * (__SBREF(__a_sb, -1, 0)))) + (0.0353f * (__SBREF(__a_sb, -1, 1)))) + (0.0354f * (__SBREF(__a_sb, 0, -1)))) + (0.0356f * (__SBREF(__a_sb, 0, 1)))) + (0.0357f * (__SBREF(__a_sb, 1, -1)))) + (0.0358f * (__SBREF(__a_sb, 1, 0)))) + (0.0359f * (__SBREF(__a_sb, 1, 1))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { double etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((0.0365f * (__REGREF(__a, 0, 0)))) + (0.0361f * (__SBREF(__a_sb, -1, -1)))) + (0.0362f * (__SBREF(__a_sb, -1, 0)))) + (0.0363f * (__SBREF(__a_sb, -1, 1)))) + (0.0364f * (__SBREF(__a_sb, 0, -1)))) + (0.0366f * (__SBREF(__a_sb, 0, 1)))) + (0.0367f * (__SBREF(__a_sb, 1, -1)))) + (0.0368f * (__SBREF(__a_sb, 1, 0)))) + (0.0369f * (__SBREF(__a_sb, 1, 1)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { double etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); } while (0);
    #define __CALC1(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __CALC2(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid2) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __CALC3(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid3) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_0);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
      __STORE(1, __reg_3_1);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
      __STORE(2, __reg_3_2);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
      __STORE(3, __reg_3_0);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
      __STORE(3, __reg_3_0);
      __DB_SWITCH(); __syncthreads();
    }
    __a_sb = __a_sb_double + __blockSize * 0;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 7; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 4;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __STORE(__h - 3, __reg_3_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
        __STORE(__h - 3, __reg_3_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
        __STORE(__h - 3, __reg_3_0);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __STORE(__h - 3, __reg_3_1);
        __reg_1_1 = __reg_0;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_0, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
        __STORE(__h - 2, __reg_3_2);
        __reg_2_1 = __reg_1_1;
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_0, __reg_2_1);
        __STORE(__h - 1, __reg_3_0);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __STORE(__h - 3, __reg_3_1);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
        __STORE(__h - 2, __reg_3_2);
        __reg_1_2 = __reg_0;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_1, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
        __STORE(__h - 1, __reg_3_0);
        __reg_2_2 = __reg_1_2;
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_1, __reg_2_2);
        __STORE(__h + 0, __reg_3_1);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __STORE(__h - 3, __reg_3_1);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
        __STORE(__h - 2, __reg_3_2);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
        __STORE(__h - 1, __reg_3_0);
        __reg_1_0 = __reg_0;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __STORE(__h + 0, __reg_3_1);
        __reg_2_0 = __reg_1_0;
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_2, __reg_2_0);
        __STORE(__h + 1, __reg_3_2);
      }
    }
    else
    {
      for (__h = 7; __h <= __side1LenOl - 3;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __STORE(__h - 3, __reg_3_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
        __STORE(__h - 3, __reg_3_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
        __STORE(__h - 3, __reg_3_0);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
      __STORE(__h - 3, __reg_3_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
      __STORE(__h - 3, __reg_3_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
      __STORE(__h - 3, __reg_3_0);
      __h++;
    }
}
__global__ void kernel0_2(double *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 1 - 1);
    const AN5D_TYPE __c1Pad = (1);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 1 - 1);
    const AN5D_TYPE __c2Pad = (1);
    #define __c2 c2
    const AN5D_TYPE __c3Len = (dimsize - 1 - 1);
    const AN5D_TYPE __c3Pad = (1);
    #define __c3 c3
    const AN5D_TYPE __halo1 = 1;
    const AN5D_TYPE __halo2 = 1;
    const AN5D_TYPE __halo3 = 1;
    const AN5D_TYPE __side0Len = 2;
    const AN5D_TYPE __side1Len = 128;
    const AN5D_TYPE __side2Len = 28;
    const AN5D_TYPE __side3Len = 28;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __OlLen3 = (__halo3 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __side3LenOl = (__side3Len + 2 * __OlLen3);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl * __side3LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __side3Num = (__c3Len + __side3Len - 1) / __side3Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid / __side3LenOl;
    const AN5D_TYPE __local_c3 = __tid % __side3LenOl;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num / __side3Num;
    const AN5D_TYPE __c2 = (blockIdx.x / __side3Num % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    const AN5D_TYPE __c3 = (blockIdx.x % __side3Num) * __side3Len + __local_c3 + __c3Pad - __OlLen3;
    double __reg_0;
    double __reg_1_0;
    double __reg_1_1;
    double __reg_1_2;
    double __reg_2_0;
    double __reg_2_1;
    double __reg_2_2;
    __shared__ double __a_sb_double[__blockSize * 2];
    double *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2 && __c3 >= __c3Pad - __halo3 && __c3 < __c3Pad + __c3Len + __halo3;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len && __c3 >= __c3Pad && __c3 < __c3Pad + __c3Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1) && __local_c3 >= (__halo3 * 1) && __local_c3 < __side3LenOl - (__halo3 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2) && __local_c3 >= (__halo3 * 2) && __local_c3 < __side3LenOl - (__halo3 * 2);
    const AN5D_TYPE __storeValid = __writeValid2;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[(((__c0 % 2) * dimsize + __c1) * dimsize + __c2) * dimsize + __c3]; }} while (0)
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((0.0375f * (__REGREF(__a, 0, 0))) + (0.0371f * (__SBREF(__a_sb, -1, -1)))) + (0.0372f * (__SBREF(__a_sb, -1, 0)))) + (0.0373f * (__SBREF(__a_sb, -1, 1)))) + (0.0374f * (__SBREF(__a_sb, 0, -1)))) + (0.0376f * (__SBREF(__a_sb, 0, 1)))) + (0.0377f * (__SBREF(__a_sb, 1, -1)))) + (0.0378f * (__SBREF(__a_sb, 1, 0)))) + (0.0379f * (__SBREF(__a_sb, 1, 1)))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((0.0355f * (__REGREF(__a, 0, 0)))) + (0.0351f * (__SBREF(__a_sb, -1, -1)))) + (0.0352f * (__SBREF(__a_sb, -1, 0)))) + (0.0353f * (__SBREF(__a_sb, -1, 1)))) + (0.0354f * (__SBREF(__a_sb, 0, -1)))) + (0.0356f * (__SBREF(__a_sb, 0, 1)))) + (0.0357f * (__SBREF(__a_sb, 1, -1)))) + (0.0358f * (__SBREF(__a_sb, 1, 0)))) + (0.0359f * (__SBREF(__a_sb, 1, 1))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { double etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((0.0365f * (__REGREF(__a, 0, 0)))) + (0.0361f * (__SBREF(__a_sb, -1, -1)))) + (0.0362f * (__SBREF(__a_sb, -1, 0)))) + (0.0363f * (__SBREF(__a_sb, -1, 1)))) + (0.0364f * (__SBREF(__a_sb, 0, -1)))) + (0.0366f * (__SBREF(__a_sb, 0, 1)))) + (0.0367f * (__SBREF(__a_sb, 1, -1)))) + (0.0368f * (__SBREF(__a_sb, 1, 0)))) + (0.0369f * (__SBREF(__a_sb, 1, 1)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { double etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); } while (0);
    #define __CALC1(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __CALC2(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid2) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __STORE(1, __reg_2_1);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __STORE(2, __reg_2_2);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __STORE(2, __reg_2_2);
      __DB_SWITCH(); __syncthreads();
    }
    __a_sb = __a_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 5; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 4;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __STORE(__h - 2, __reg_2_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __STORE(__h - 2, __reg_2_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __STORE(__h - 2, __reg_2_2);
        __h++;
      }
      if (0) {}
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __STORE(__h - 2, __reg_2_0);
        __reg_1_2 = __reg_0;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_1, __reg_1_2);
        __STORE(__h - 1, __reg_2_1);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __STORE(__h - 2, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __STORE(__h - 1, __reg_2_1);
        __reg_1_0 = __reg_0;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_2, __reg_1_0);
        __STORE(__h + 0, __reg_2_2);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __STORE(__h - 2, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __STORE(__h - 1, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __STORE(__h + 0, __reg_2_2);
        __reg_1_1 = __reg_0;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_0, __reg_1_1);
        __STORE(__h + 1, __reg_2_0);
      }
    }
    else
    {
      for (__h = 5; __h <= __side1LenOl - 3;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __STORE(__h - 2, __reg_2_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __STORE(__h - 2, __reg_2_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __STORE(__h - 2, __reg_2_2);
        __h++;
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __STORE(__h - 2, __reg_2_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __STORE(__h - 2, __reg_2_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __STORE(__h - 2, __reg_2_2);
      __h++;
    }
}
__global__ void kernel0_1(double *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 1 - 1);
    const AN5D_TYPE __c1Pad = (1);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 1 - 1);
    const AN5D_TYPE __c2Pad = (1);
    #define __c2 c2
    const AN5D_TYPE __c3Len = (dimsize - 1 - 1);
    const AN5D_TYPE __c3Pad = (1);
    #define __c3 c3
    const AN5D_TYPE __halo1 = 1;
    const AN5D_TYPE __halo2 = 1;
    const AN5D_TYPE __halo3 = 1;
    const AN5D_TYPE __side0Len = 1;
    const AN5D_TYPE __side1Len = 128;
    const AN5D_TYPE __side2Len = 30;
    const AN5D_TYPE __side3Len = 30;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __OlLen3 = (__halo3 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __side3LenOl = (__side3Len + 2 * __OlLen3);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl * __side3LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __side3Num = (__c3Len + __side3Len - 1) / __side3Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid / __side3LenOl;
    const AN5D_TYPE __local_c3 = __tid % __side3LenOl;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num / __side3Num;
    const AN5D_TYPE __c2 = (blockIdx.x / __side3Num % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    const AN5D_TYPE __c3 = (blockIdx.x % __side3Num) * __side3Len + __local_c3 + __c3Pad - __OlLen3;
    double __reg_0;
    double __reg_1_0;
    double __reg_1_1;
    double __reg_1_2;
    __shared__ double __a_sb_double[__blockSize * 2];
    double *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2 && __c3 >= __c3Pad - __halo3 && __c3 < __c3Pad + __c3Len + __halo3;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len && __c3 >= __c3Pad && __c3 < __c3Pad + __c3Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1) && __local_c3 >= (__halo3 * 1) && __local_c3 < __side3LenOl - (__halo3 * 1);
    const AN5D_TYPE __storeValid = __writeValid1;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[(((__c0 % 2) * dimsize + __c1) * dimsize + __c2) * dimsize + __c3]; }} while (0)
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((0.0375f * (__REGREF(__a, 0, 0))) + (0.0371f * (__SBREF(__a_sb, -1, -1)))) + (0.0372f * (__SBREF(__a_sb, -1, 0)))) + (0.0373f * (__SBREF(__a_sb, -1, 1)))) + (0.0374f * (__SBREF(__a_sb, 0, -1)))) + (0.0376f * (__SBREF(__a_sb, 0, 1)))) + (0.0377f * (__SBREF(__a_sb, 1, -1)))) + (0.0378f * (__SBREF(__a_sb, 1, 0)))) + (0.0379f * (__SBREF(__a_sb, 1, 1)))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((0.0355f * (__REGREF(__a, 0, 0)))) + (0.0351f * (__SBREF(__a_sb, -1, -1)))) + (0.0352f * (__SBREF(__a_sb, -1, 0)))) + (0.0353f * (__SBREF(__a_sb, -1, 1)))) + (0.0354f * (__SBREF(__a_sb, 0, -1)))) + (0.0356f * (__SBREF(__a_sb, 0, 1)))) + (0.0357f * (__SBREF(__a_sb, 1, -1)))) + (0.0358f * (__SBREF(__a_sb, 1, 0)))) + (0.0359f * (__SBREF(__a_sb, 1, 1))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { double etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[((((c0 + 1) % 2) * dimsize + c1) * dimsize + c2) * dimsize + c3])
    #define __REGREF(reg, i2, i3) reg
    #define __SBREF(sb, i2, i3) __sbref_wrap(sb, (int)__tid + i2 * (int)__side3LenOl + i3)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((0.0365f * (__REGREF(__a, 0, 0)))) + (0.0361f * (__SBREF(__a_sb, -1, -1)))) + (0.0362f * (__SBREF(__a_sb, -1, 0)))) + (0.0363f * (__SBREF(__a_sb, -1, 1)))) + (0.0364f * (__SBREF(__a_sb, 0, -1)))) + (0.0366f * (__SBREF(__a_sb, 0, 1)))) + (0.0367f * (__SBREF(__a_sb, 1, -1)))) + (0.0368f * (__SBREF(__a_sb, 1, 0)))) + (0.0369f * (__SBREF(__a_sb, 1, 1)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { double etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); } while (0);
    #define __CALC1(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __STORE(1, __reg_1_1);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __STORE(1, __reg_1_1);
    }
    __a_sb = __a_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 3; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 4;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __STORE(__h - 1, __reg_1_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 1, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 1, __reg_1_1);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __STORE(__h - 1, __reg_1_2);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __STORE(__h - 1, __reg_1_2);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __STORE(__h - 1, __reg_1_2);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
      }
    }
    else
    {
      for (__h = 3; __h <= __side1LenOl - 3;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __STORE(__h - 1, __reg_1_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 1, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 1, __reg_1_1);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __STORE(__h - 1, __reg_1_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __STORE(__h - 1, __reg_1_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __STORE(__h - 1, __reg_1_1);
      __h++;
    }
}
