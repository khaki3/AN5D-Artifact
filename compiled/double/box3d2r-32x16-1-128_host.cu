#include <assert.h>
#include <stdio.h>
#include "box3d2r-32x16-1-128_kernel.hu"
#define BENCH_DIM 3
#define BENCH_FPP 249
#define BENCH_RAD 2

#include "common.h"

double kernel_stencil(SB_TYPE *A1, int compsize, int timestep, bool scop)
{
  double start_time = sb_time(), end_time = 0.0;
  int dimsize = compsize + BENCH_RAD * 2;
  SB_TYPE (*A)[dimsize][dimsize][dimsize]
    = (SB_TYPE (*)[dimsize][dimsize][dimsize])A1;

  if (scop) {
    if (dimsize >= 5 && timestep >= 1) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

      double *dev_A;
      
      cudaCheckReturn(hipMalloc((void **) &dev_A, (size_t)(2) * (size_t)(dimsize) * (size_t)(dimsize) * (size_t)(dimsize) * sizeof(double)));
      
{
      cudaCheckReturn(hipMemcpy(dev_A, A, (size_t)(2) * (size_t)(dimsize) * (size_t)(dimsize) * (size_t)(dimsize) * sizeof(double), hipMemcpyHostToDevice));
#ifdef STENCILBENCH
hipDeviceSynchronize();
SB_START_INSTRUMENTS;
#endif
}
    {
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
      const AN5D_TYPE __c0Len = (timestep - 0);
      const AN5D_TYPE __c0Pad = (0);
      #define __c0 c0
      const AN5D_TYPE __c1Len = (dimsize - 2 - 2);
      const AN5D_TYPE __c1Pad = (2);
      #define __c1 c1
      const AN5D_TYPE __c2Len = (dimsize - 2 - 2);
      const AN5D_TYPE __c2Pad = (2);
      #define __c2 c2
      const AN5D_TYPE __c3Len = (dimsize - 2 - 2);
      const AN5D_TYPE __c3Pad = (2);
      #define __c3 c3
      const AN5D_TYPE __halo1 = 2;
      const AN5D_TYPE __halo2 = 2;
      const AN5D_TYPE __halo3 = 2;
      AN5D_TYPE c0;
      AN5D_TYPE __side0LenMax;
      {
        const AN5D_TYPE __side0Len = 1;
        const AN5D_TYPE __side1Len = 128;
        const AN5D_TYPE __side2Len = 12;
        const AN5D_TYPE __side3Len = 28;
        const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
        const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
        const AN5D_TYPE __OlLen3 = (__halo3 * __side0Len);
        const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
        const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
        const AN5D_TYPE __side3LenOl = (__side3Len + 2 * __OlLen3);
        const AN5D_TYPE __blockSize = 1 * __side2LenOl * __side3LenOl;
        assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
        dim3 k0_dimBlock(__blockSize, 1, 1);
        dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len) * ((__c3Len + __side3Len - 1) / __side3Len), 1, 1);
        AN5D_TYPE __c0Padr = (__c0Len % 2) != (((__c0Len + __side0Len - 1) / __side0Len) % 2) && __c0Len % __side0Len < 2 ? 1 : 0;
        __side0LenMax = __side0Len;
        for (c0 = __c0Pad; c0 < __c0Pad + __c0Len / __side0Len - __c0Padr; c0 += 1)
        {
          kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
      }
      if ((__c0Len % 2) != (((__c0Len + __side0LenMax - 1) / __side0LenMax) % 2))
      {
        if (__c0Len % __side0LenMax == 0)
        {
        }
      }
      else if (__c0Len % __side0LenMax)
      {
      }
    }
    cudaCheckKernel();
{
#ifdef STENCILBENCH
hipDeviceSynchronize();
SB_STOP_INSTRUMENTS;
#endif
      cudaCheckReturn(hipMemcpy(A, dev_A, (size_t)(2) * (size_t)(dimsize) * (size_t)(dimsize) * (size_t)(dimsize) * sizeof(double), hipMemcpyDeviceToHost));
}
      cudaCheckReturn(hipFree(dev_A));
    }
  }
  else {
    for (int t = 0; t < timestep; t++)
#pragma omp parallel for
      for (int i = BENCH_RAD; i < dimsize - BENCH_RAD; i++)
        for (int j = BENCH_RAD; j < dimsize - BENCH_RAD; j++)
          for (int k = BENCH_RAD; k < dimsize - BENCH_RAD; k++)
            A[(t+1)%2][i][j][k] =
              -0.324f*A[t%2][i-2][j][k] +
              0.0020f*A[t%2][i-2][j-2][k-2] +
              0.0030f*A[t%2][i-2][j-2][k-1] +
              0.0040f*A[t%2][i-2][j-2][k] +
              0.0050f*A[t%2][i-2][j-2][k+1] +
              0.0060f*A[t%2][i-2][j-2][k+2] +
              0.0070f*A[t%2][i-2][j-1][k-2] +
              0.0080f*A[t%2][i-2][j-1][k-1] +
              0.0090f*A[t%2][i-2][j-1][k] +
              0.0100f*A[t%2][i-2][j-1][k+1] +
              0.0110f*A[t%2][i-2][j-1][k+2] +
              0.0120f*A[t%2][i-2][j][k-2] +
              0.0130f*A[t%2][i-2][j][k-1] +
              0.0140f*A[t%2][i-2][j][k+1] +
              0.0150f*A[t%2][i-2][j][k+2] +
              0.0160f*A[t%2][i-2][j+1][k-2] +
              0.0170f*A[t%2][i-2][j+1][k-1] +
              0.0180f*A[t%2][i-2][j+1][k] +
              0.0190f*A[t%2][i-2][j+1][k+1] +
              0.0200f*A[t%2][i-2][j+1][k+2] +
              0.0210f*A[t%2][i-2][j+2][k-2] +
              0.0220f*A[t%2][i-2][j+2][k-1] +
              0.0230f*A[t%2][i-2][j+2][k] +
              0.0240f*A[t%2][i-2][j+2][k+1] +
              0.0250f*A[t%2][i-2][j+2][k+2] -

              0.3264f*A[t%2][i-1][j][k] +
              0.0021f*A[t%2][i-1][j-2][k-2] +
              0.0031f*A[t%2][i-1][j-2][k-1] +
              0.0041f*A[t%2][i-1][j-2][k] +
              0.0051f*A[t%2][i-1][j-2][k+1] +
              0.0061f*A[t%2][i-1][j-2][k+2] +
              0.0071f*A[t%2][i-1][j-1][k-2] +
              0.0081f*A[t%2][i-1][j-1][k-1] +
              0.0091f*A[t%2][i-1][j-1][k] +
              0.0101f*A[t%2][i-1][j-1][k+1] +
              0.0111f*A[t%2][i-1][j-1][k+2] +
              0.0121f*A[t%2][i-1][j][k-2] +
              0.0131f*A[t%2][i-1][j][k-1] +
              0.0141f*A[t%2][i-1][j][k+1] +
              0.0151f*A[t%2][i-1][j][k+2] +
              0.0161f*A[t%2][i-1][j+1][k-2] +
              0.0171f*A[t%2][i-1][j+1][k-1] +
              0.0181f*A[t%2][i-1][j+1][k] +
              0.0191f*A[t%2][i-1][j+1][k+1] +
              0.0201f*A[t%2][i-1][j+1][k+2] +
              0.0211f*A[t%2][i-1][j+2][k-2] +
              0.0221f*A[t%2][i-1][j+2][k-1] +
              0.0231f*A[t%2][i-1][j+2][k] +
              0.0241f*A[t%2][i-1][j+2][k+1] +
              0.0251f*A[t%2][i-1][j+2][k+2] +

              0.6712f*A[t%2][i][j][k] +
              0.0022f*A[t%2][i][j-2][k-2] +
              0.0032f*A[t%2][i][j-2][k-1] +
              0.0042f*A[t%2][i][j-2][k] +
              0.0052f*A[t%2][i][j-2][k+1] +
              0.0062f*A[t%2][i][j-2][k+2] +
              0.0072f*A[t%2][i][j-1][k-2] +
              0.0082f*A[t%2][i][j-1][k-1] +
              0.0092f*A[t%2][i][j-1][k] +
              0.0102f*A[t%2][i][j-1][k+1] +
              0.0112f*A[t%2][i][j-1][k+2] +
              0.0122f*A[t%2][i][j][k-2] +
              0.0132f*A[t%2][i][j][k-1] +
              0.0142f*A[t%2][i][j][k+1] +
              0.0152f*A[t%2][i][j][k+2] +
              0.0162f*A[t%2][i][j+1][k-2] +
              0.0172f*A[t%2][i][j+1][k-1] +
              0.0182f*A[t%2][i][j+1][k] +
              0.0192f*A[t%2][i][j+1][k+1] +
              0.0202f*A[t%2][i][j+1][k+2] +
              0.0212f*A[t%2][i][j+2][k-2] +
              0.0222f*A[t%2][i][j+2][k-1] +
              0.0232f*A[t%2][i][j+2][k] +
              0.0242f*A[t%2][i][j+2][k+1] +
              0.0252f*A[t%2][i][j+2][k+2] -

              0.3312f*A[t%2][i+1][j][k] +
              0.0023f*A[t%2][i+1][j-2][k-2] +
              0.0033f*A[t%2][i+1][j-2][k-1] +
              0.0043f*A[t%2][i+1][j-2][k] +
              0.0053f*A[t%2][i+1][j-2][k+1] +
              0.0063f*A[t%2][i+1][j-2][k+2] +
              0.0073f*A[t%2][i+1][j-1][k-2] +
              0.0083f*A[t%2][i+1][j-1][k-1] +
              0.0093f*A[t%2][i+1][j-1][k] +
              0.0103f*A[t%2][i+1][j-1][k+1] +
              0.0113f*A[t%2][i+1][j-1][k+2] +
              0.0123f*A[t%2][i+1][j][k-2] +
              0.0133f*A[t%2][i+1][j][k-1] +
              0.0143f*A[t%2][i+1][j][k+1] +
              0.0153f*A[t%2][i+1][j][k+2] +
              0.0163f*A[t%2][i+1][j+1][k-2] +
              0.0173f*A[t%2][i+1][j+1][k-1] +
              0.0183f*A[t%2][i+1][j+1][k] +
              0.0193f*A[t%2][i+1][j+1][k+1] +
              0.0203f*A[t%2][i+1][j+1][k+2] +
              0.0213f*A[t%2][i+1][j+2][k-2] +
              0.0223f*A[t%2][i+1][j+2][k-1] +
              0.0233f*A[t%2][i+1][j+2][k] +
              0.0243f*A[t%2][i+1][j+2][k+1] +
              0.0253f*A[t%2][i+1][j+2][k+2] -

              0.3336f*A[t%2][i+2][j][k] +
              0.0024f*A[t%2][i+2][j-2][k-2] +
              0.0034f*A[t%2][i+2][j-2][k-1] +
              0.0044f*A[t%2][i+2][j-2][k] +
              0.0054f*A[t%2][i+2][j-2][k+1] +
              0.0064f*A[t%2][i+2][j-2][k+2] +
              0.0074f*A[t%2][i+2][j-1][k-2] +
              0.0084f*A[t%2][i+2][j-1][k-1] +
              0.0094f*A[t%2][i+2][j-1][k] +
              0.0104f*A[t%2][i+2][j-1][k+1] +
              0.0114f*A[t%2][i+2][j-1][k+2] +
              0.0124f*A[t%2][i+2][j][k-2] +
              0.0134f*A[t%2][i+2][j][k-1] +
              0.0144f*A[t%2][i+2][j][k+1] +
              0.0154f*A[t%2][i+2][j][k+2] +
              0.0164f*A[t%2][i+2][j+1][k-2] +
              0.0174f*A[t%2][i+2][j+1][k-1] +
              0.0184f*A[t%2][i+2][j+1][k] +
              0.0194f*A[t%2][i+2][j+1][k+1] +
              0.0204f*A[t%2][i+2][j+1][k+2] +
              0.0214f*A[t%2][i+2][j+2][k-2] +
              0.0224f*A[t%2][i+2][j+2][k-1] +
              0.0234f*A[t%2][i+2][j+2][k] +
              0.0244f*A[t%2][i+2][j+2][k+1] +
              0.0254f*A[t%2][i+2][j+2][k+2];
  }

  return (((end_time != 0.0) ? end_time : sb_time()) - start_time);
}
