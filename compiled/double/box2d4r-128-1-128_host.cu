#include <assert.h>
#include <stdio.h>
#include "box2d4r-128-1-128_kernel.hu"
#define BENCH_DIM 2
#define BENCH_FPP 161
#define BENCH_RAD 4

#include "common.h"

double kernel_stencil(SB_TYPE *A1, int compsize, int timestep, bool scop)
{
  double start_time = sb_time(), end_time = 0.0;
  int dimsize = compsize + BENCH_RAD * 2;
  SB_TYPE (*A)[dimsize][dimsize] = (SB_TYPE (*)[dimsize][dimsize])A1;

  if (scop) {
    if (dimsize >= 9 && timestep >= 1) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

      double *dev_A;
      
      cudaCheckReturn(hipMalloc((void **) &dev_A, (size_t)(2) * (size_t)(dimsize) * (size_t)(dimsize) * sizeof(double)));
      
{
      cudaCheckReturn(hipMemcpy(dev_A, A, (size_t)(2) * (size_t)(dimsize) * (size_t)(dimsize) * sizeof(double), hipMemcpyHostToDevice));
#ifdef STENCILBENCH
hipDeviceSynchronize();
SB_START_INSTRUMENTS;
#endif
}
    {
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
      const AN5D_TYPE __c0Len = (timestep - 0);
      const AN5D_TYPE __c0Pad = (0);
      #define __c0 c0
      const AN5D_TYPE __c1Len = (dimsize - 4 - 4);
      const AN5D_TYPE __c1Pad = (4);
      #define __c1 c1
      const AN5D_TYPE __c2Len = (dimsize - 4 - 4);
      const AN5D_TYPE __c2Pad = (4);
      #define __c2 c2
      const AN5D_TYPE __halo1 = 4;
      const AN5D_TYPE __halo2 = 4;
      AN5D_TYPE c0;
      AN5D_TYPE __side0LenMax;
      {
        const AN5D_TYPE __side0Len = 1;
        const AN5D_TYPE __side1Len = 128;
        const AN5D_TYPE __side2Len = 120;
        const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
        const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
        const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
        const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
        const AN5D_TYPE __blockSize = 1 * __side2LenOl;
        assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
        dim3 k0_dimBlock(__blockSize, 1, 1);
        dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
        AN5D_TYPE __c0Padr = (__c0Len % 2) != (((__c0Len + __side0Len - 1) / __side0Len) % 2) && __c0Len % __side0Len < 2 ? 1 : 0;
        __side0LenMax = __side0Len;
        for (c0 = __c0Pad; c0 < __c0Pad + __c0Len / __side0Len - __c0Padr; c0 += 1)
        {
          kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_A, dimsize, timestep, c0);
        }
      }
      if ((__c0Len % 2) != (((__c0Len + __side0LenMax - 1) / __side0LenMax) % 2))
      {
        if (__c0Len % __side0LenMax == 0)
        {
        }
      }
      else if (__c0Len % __side0LenMax)
      {
      }
    }
    cudaCheckKernel();
{
#ifdef STENCILBENCH
hipDeviceSynchronize();
SB_STOP_INSTRUMENTS;
#endif
      cudaCheckReturn(hipMemcpy(A, dev_A, (size_t)(2) * (size_t)(dimsize) * (size_t)(dimsize) * sizeof(double), hipMemcpyDeviceToHost));
}
      cudaCheckReturn(hipFree(dev_A));
    }
  }
  else {
    for (int t = 0; t < timestep; t++)
#pragma omp parallel for
      for (int i = BENCH_RAD; i < dimsize - BENCH_RAD; i++)
        for (int j = BENCH_RAD; j < dimsize - BENCH_RAD; j++)
          A[(t+1)%2][i][j] =
            0.00930f * A[t%2][i-4][j-4] +
            0.00931f * A[t%2][i-4][j-3] +
            0.00932f * A[t%2][i-4][j-2] +
            0.00933f * A[t%2][i-4][j-1] +
            0.00934f * A[t%2][i-4][j] +
            0.00935f * A[t%2][i-4][j+1] +
            0.00936f * A[t%2][i-4][j+2] +
            0.00937f * A[t%2][i-4][j+3] +
            0.00938f * A[t%2][i-4][j+4] +

            0.00939f * A[t%2][i-3][j-4] +
            0.00940f * A[t%2][i-3][j-3] +
            0.00941f * A[t%2][i-3][j-2] +
            0.00942f * A[t%2][i-3][j-1] +
            0.00943f * A[t%2][i-3][j] +
            0.00944f * A[t%2][i-3][j+1] +
            0.00945f * A[t%2][i-3][j+2] +
            0.00946f * A[t%2][i-3][j+3] +
            0.00947f * A[t%2][i-3][j+4] +

            0.00948f * A[t%2][i-2][j-4] +
            0.00949f * A[t%2][i-2][j-3] +
            0.00950f * A[t%2][i-2][j-2] +
            0.00951f * A[t%2][i-2][j-1] +
            0.00952f * A[t%2][i-2][j] +
            0.00953f * A[t%2][i-2][j+1] +
            0.00954f * A[t%2][i-2][j+2] +
            0.00955f * A[t%2][i-2][j+3] +
            0.00956f * A[t%2][i-2][j+4] +

            0.00957f * A[t%2][i-1][j-4] +
            0.00958f * A[t%2][i-1][j-3] +
            0.00959f * A[t%2][i-1][j-2] +
            0.00960f * A[t%2][i-1][j-1] +
            0.00961f * A[t%2][i-1][j] +
            0.00962f * A[t%2][i-1][j+1] +
            0.00963f * A[t%2][i-1][j+2] +
            0.00964f * A[t%2][i-1][j+3] +
            0.00965f * A[t%2][i-1][j+4] +

            0.00966f * A[t%2][i][j-4] +
            0.00967f * A[t%2][i][j-3] +
            0.00968f * A[t%2][i][j-2] +
            0.00969f * A[t%2][i][j-1] +
            0.22400f * A[t%2][i][j] +
            0.00971f * A[t%2][i][j+1] +
            0.00972f * A[t%2][i][j+2] +
            0.00973f * A[t%2][i][j+3] +
            0.00974f * A[t%2][i][j+4] +

            0.00975f * A[t%2][i+1][j-4] +
            0.00976f * A[t%2][i+1][j-3] +
            0.00977f * A[t%2][i+1][j-2] +
            0.00978f * A[t%2][i+1][j-1] +
            0.00979f * A[t%2][i+1][j] +
            0.00980f * A[t%2][i+1][j+1] +
            0.00981f * A[t%2][i+1][j+2] +
            0.00982f * A[t%2][i+1][j+3] +
            0.00983f * A[t%2][i+1][j+4] +

            0.00984f * A[t%2][i+2][j-4] +
            0.00985f * A[t%2][i+2][j-3] +
            0.00986f * A[t%2][i+2][j-2] +
            0.00987f * A[t%2][i+2][j-1] +
            0.00988f * A[t%2][i+2][j] +
            0.00989f * A[t%2][i+2][j+1] +
            0.00990f * A[t%2][i+2][j+2] +
            0.00991f * A[t%2][i+2][j+3] +
            0.00992f * A[t%2][i+2][j+4] +

            0.00993f * A[t%2][i+3][j-4] +
            0.00994f * A[t%2][i+3][j-3] +
            0.00995f * A[t%2][i+3][j-2] +
            0.00996f * A[t%2][i+3][j-1] +
            0.00997f * A[t%2][i+3][j] +
            0.00998f * A[t%2][i+3][j+1] +
            0.00999f * A[t%2][i+3][j+2] +
            0.01000f * A[t%2][i+3][j+3] +
            0.01001f * A[t%2][i+3][j+4] +

            0.01002f * A[t%2][i+4][j-4] +
            0.01003f * A[t%2][i+4][j-3] +
            0.01004f * A[t%2][i+4][j-2] +
            0.01005f * A[t%2][i+4][j-1] +
            0.01006f * A[t%2][i+4][j] +
            0.01007f * A[t%2][i+4][j+1] +
            0.01008f * A[t%2][i+4][j+2] +
            0.01009f * A[t%2][i+4][j+3] +
            0.01010f * A[t%2][i+4][j+4];
  }

  return (((end_time != 0.0) ? end_time : sb_time()) - start_time);
}
