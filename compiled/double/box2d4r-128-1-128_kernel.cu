#include "hip/hip_runtime.h"
#include "box2d4r-128-1-128_kernel.hu"
__device__ double __sbref_wrap(double *sb, size_t index) { return sb[index]; }

__global__ void kernel0_1(double *A, int dimsize, int timestep, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (timestep - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (dimsize - 4 - 4);
    const AN5D_TYPE __c1Pad = (4);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (dimsize - 4 - 4);
    const AN5D_TYPE __c2Pad = (4);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 4;
    const AN5D_TYPE __halo2 = 4;
    const AN5D_TYPE __side0Len = 1;
    const AN5D_TYPE __side1Len = 128;
    const AN5D_TYPE __side2Len = 120;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    double __reg_0;
    double __reg_1_0;
    double __reg_1_1;
    double __reg_1_2;
    double __reg_1_3;
    double __reg_1_4;
    double __reg_1_5;
    double __reg_1_6;
    double __reg_1_7;
    double __reg_1_8;
    __shared__ double __a_sb_double[__blockSize * 2];
    double *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __storeValid = __writeValid1;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = A[((__c0 % 2) * dimsize + __c1) * dimsize + __c2]; }} while (0)
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00930f * (__SBREF(__a_sb, -4))) + (0.00931f * (__SBREF(__a_sb, -3)))) + (0.00932f * (__SBREF(__a_sb, -2)))) + (0.00933f * (__SBREF(__a_sb, -1)))) + (0.00934f * (__REGREF(__a, 0)))) + (0.00935f * (__SBREF(__a_sb, 1)))) + (0.00936f * (__SBREF(__a_sb, 2)))) + (0.00937f * (__SBREF(__a_sb, 3)))) + (0.00938f * (__SBREF(__a_sb, 4)))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00939f * (__SBREF(__a_sb, -4)))) + (0.00940f * (__SBREF(__a_sb, -3)))) + (0.00941f * (__SBREF(__a_sb, -2)))) + (0.00942f * (__SBREF(__a_sb, -1)))) + (0.00943f * (__REGREF(__a, 0)))) + (0.00944f * (__SBREF(__a_sb, 1)))) + (0.00945f * (__SBREF(__a_sb, 2)))) + (0.00946f * (__SBREF(__a_sb, 3)))) + (0.00947f * (__SBREF(__a_sb, 4))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { double etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00948f * (__SBREF(__a_sb, -4)))) + (0.00949f * (__SBREF(__a_sb, -3)))) + (0.00950f * (__SBREF(__a_sb, -2)))) + (0.00951f * (__SBREF(__a_sb, -1)))) + (0.00952f * (__REGREF(__a, 0)))) + (0.00953f * (__SBREF(__a_sb, 1)))) + (0.00954f * (__SBREF(__a_sb, 2)))) + (0.00955f * (__SBREF(__a_sb, 3)))) + (0.00956f * (__SBREF(__a_sb, 4)))))))))))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { double etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_3_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((((((((((((((((((((0.00957f * (__SBREF(__a_sb, -4)))) + (0.00958f * (__SBREF(__a_sb, -3)))) + (0.00959f * (__SBREF(__a_sb, -2)))) + (0.00960f * (__SBREF(__a_sb, -1)))) + (0.00961f * (__REGREF(__a, 0)))) + (0.00962f * (__SBREF(__a_sb, 1)))) + (0.00963f * (__SBREF(__a_sb, 2)))) + (0.00964f * (__SBREF(__a_sb, 3)))) + (0.00965f * (__SBREF(__a_sb, 4))))))))))))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_3(out, a) do { double etmp; __CALCEXPR_3_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_4_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((((((((((((((((((((0.00966f * (__SBREF(__a_sb, -4)))) + (0.00967f * (__SBREF(__a_sb, -3)))) + (0.00968f * (__SBREF(__a_sb, -2)))) + (0.00969f * (__SBREF(__a_sb, -1)))) + (0.22400f * (__REGREF(__a, 0)))) + (0.00971f * (__SBREF(__a_sb, 1)))) + (0.00972f * (__SBREF(__a_sb, 2)))) + (0.00973f * (__SBREF(__a_sb, 3)))) + (0.00974f * (__SBREF(__a_sb, 4)))))))))))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_4(out, a) do { double etmp; __CALCEXPR_4_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_5_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((((((((((((((((((((0.00975f * (__SBREF(__a_sb, -4)))) + (0.00976f * (__SBREF(__a_sb, -3)))) + (0.00977f * (__SBREF(__a_sb, -2)))) + (0.00978f * (__SBREF(__a_sb, -1)))) + (0.00979f * (__REGREF(__a, 0)))) + (0.00980f * (__SBREF(__a_sb, 1)))) + (0.00981f * (__SBREF(__a_sb, 2)))) + (0.00982f * (__SBREF(__a_sb, 3)))) + (0.00983f * (__SBREF(__a_sb, 4))))))))))))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_5(out, a) do { double etmp; __CALCEXPR_5_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_6_wrap(__rn0, __a) do { __rn0 = ((((((((((((((((((((((((((((0.00984f * (__SBREF(__a_sb, -4)))) + (0.00985f * (__SBREF(__a_sb, -3)))) + (0.00986f * (__SBREF(__a_sb, -2)))) + (0.00987f * (__SBREF(__a_sb, -1)))) + (0.00988f * (__REGREF(__a, 0)))) + (0.00989f * (__SBREF(__a_sb, 1)))) + (0.00990f * (__SBREF(__a_sb, 2)))) + (0.00991f * (__SBREF(__a_sb, 3)))) + (0.00992f * (__SBREF(__a_sb, 4)))))))))))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_6(out, a) do { double etmp; __CALCEXPR_6_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_7_wrap(__rn0, __a) do { __rn0 = (((((((((((((((((((0.00993f * (__SBREF(__a_sb, -4)))) + (0.00994f * (__SBREF(__a_sb, -3)))) + (0.00995f * (__SBREF(__a_sb, -2)))) + (0.00996f * (__SBREF(__a_sb, -1)))) + (0.00997f * (__REGREF(__a, 0)))) + (0.00998f * (__SBREF(__a_sb, 1)))) + (0.00999f * (__SBREF(__a_sb, 2)))) + (0.01000f * (__SBREF(__a_sb, 3)))) + (0.01001f * (__SBREF(__a_sb, 4))))))))))))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_7(out, a) do { double etmp; __CALCEXPR_7_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (A[(((c0 + 1) % 2) * dimsize + c1) * dimsize + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_8_wrap(__rn0, __a) do { __rn0 = ((((((((((0.01002f * (__SBREF(__a_sb, -4)))) + (0.01003f * (__SBREF(__a_sb, -3)))) + (0.01004f * (__SBREF(__a_sb, -2)))) + (0.01005f * (__SBREF(__a_sb, -1)))) + (0.01006f * (__REGREF(__a, 0)))) + (0.01007f * (__SBREF(__a_sb, 1)))) + (0.01008f * (__SBREF(__a_sb, 2)))) + (0.01009f * (__SBREF(__a_sb, 3)))) + (0.01010f * (__SBREF(__a_sb, 4)))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_8(out, a) do { double etmp; __CALCEXPR_8_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); __CALCEXPR_3(out3, reg); __CALCEXPR_4(out4, reg); __CALCEXPR_5(out5, reg); __CALCEXPR_6(out6, reg); __CALCEXPR_7(out7, reg); __CALCEXPR_8(out8, reg); } while (0);
    #define __CALC1(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, out3, out4, out5, out6, out7, out8, reg); } else out4 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __STORE(4, __reg_1_4);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __STORE(4, __reg_1_4);
    }
    __a_sb = __a_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 9; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 13;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 4, __reg_1_6);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 4, __reg_1_7);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 4, __reg_1_8);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 4, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 4, __reg_1_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h - 4, __reg_1_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 4, __reg_1_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 4, __reg_1_4);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 4 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
      }
      else if (__h + 5 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
      }
      else if (__h + 6 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
      }
      else if (__h + 7 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_7, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
      }
      else if (__h + 8 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_8, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
      }
      else if (__h + 9 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_0, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_4, __reg_0);
        __STORE(__h + 4, __reg_1_4);
      }
      else if (__h + 10 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_1, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h + 4, __reg_1_4);
        __LOAD(__reg_0, __h + 9);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_5, __reg_0);
        __STORE(__h + 5, __reg_1_5);
      }
      else if (__h + 11 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_2, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h + 4, __reg_1_4);
        __LOAD(__reg_0, __h + 9);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h + 5, __reg_1_5);
        __LOAD(__reg_0, __h + 10);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_6, __reg_0);
        __STORE(__h + 6, __reg_1_6);
      }
      else if (__h + 12 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 3, __reg_1_6);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 2, __reg_1_7);
        __LOAD(__reg_0, __h + 3);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 1, __reg_1_8);
        __LOAD(__reg_0, __h + 4);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 5);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
        __LOAD(__reg_0, __h + 6);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h + 2, __reg_1_2);
        __LOAD(__reg_0, __h + 7);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h + 3, __reg_1_3);
        __LOAD(__reg_0, __h + 8);
        __CALC1(__reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_3, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h + 4, __reg_1_4);
        __LOAD(__reg_0, __h + 9);
        __CALC1(__reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_4, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h + 5, __reg_1_5);
        __LOAD(__reg_0, __h + 10);
        __CALC1(__reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_5, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h + 6, __reg_1_6);
        __LOAD(__reg_0, __h + 11);
        __CALC1(__reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_6, __reg_1_7, __reg_0);
        __STORE(__h + 7, __reg_1_7);
      }
    }
    else
    {
      for (__h = 9; __h <= __side1LenOl - 9;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
        __STORE(__h - 4, __reg_1_5);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
        __STORE(__h - 4, __reg_1_6);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
        __STORE(__h - 4, __reg_1_7);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
        __STORE(__h - 4, __reg_1_8);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 4, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 4, __reg_1_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
        __STORE(__h - 4, __reg_1_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
        __STORE(__h - 4, __reg_1_3);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
        __STORE(__h - 4, __reg_1_4);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_0);
      __STORE(__h - 4, __reg_1_5);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_0);
      __STORE(__h - 4, __reg_1_6);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_0);
      __STORE(__h - 4, __reg_1_7);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_0);
      __STORE(__h - 4, __reg_1_8);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __STORE(__h - 4, __reg_1_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_1_1, __reg_0);
      __STORE(__h - 4, __reg_1_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_1_2, __reg_0);
      __STORE(__h - 4, __reg_1_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_1_3, __reg_0);
      __STORE(__h - 4, __reg_1_3);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_3, __reg_1_2, __reg_1_1, __reg_1_0, __reg_1_8, __reg_1_7, __reg_1_6, __reg_1_5, __reg_1_4, __reg_0);
      __STORE(__h - 4, __reg_1_4);
      __h++;
    }
}
